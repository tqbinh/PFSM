#include "hip/hip_runtime.h"
#include "kernelPrintf.h"


//__device__ void __syncthreads(void);
__global__ void kernelPrintf(int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){			
		printf("[%d]:%d ; ",i,O[i]);
	}

}


hipError_t printInt(int* d_array,int noElem_d_Array){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_Array+block.x-1)/block.x);

	kernelPrintf<<<grid,block>>>(d_array,noElem_d_Array);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintInt failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}

__global__ void kernelprintUnsignedInt(unsigned int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){			
		printf("[%d]:%d ; ",i,O[i]);
	}

}

inline hipError_t printUnsignedInt(unsigned int* d_array,int noElem_d_Array){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_Array+block.x-1)/block.x);

	kernelprintUnsignedInt<<<grid,block>>>(d_array,noElem_d_Array);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintInt failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}



__global__ void kernelPrintFloat(float* A,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){
		printf("[%d]:%.0f ;",i,A[i]);
	}

}

hipError_t printFloat(float* d_array,int numberElementOfArray){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((numberElementOfArray+block.x-1)/block.x);

	kernelPrintFloat<<<grid,block>>>(d_array,numberElementOfArray);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintExtention failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}





