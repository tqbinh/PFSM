#include "hip/hip_runtime.h"
#pragma once
#include "pms.cuh"
#include "moderngpu.cuh"		// Include all MGPU kernels.

using namespace mgpu;
StopWatchWin timer;

#define PMS_PRINT std::printf


float hTime=0.0;
float dTime=0.0;

int nthDFSCODE = 0;

//Ghi DFS_CODE xuống file
void write_minDFS_CODE(DFSCode dfscode)
{
	//Mở file minDFSCODE.txt để ghi thêm vào
	char* dfsfile = "minDFSCODE.txt";
	fstream of;
	of.open(dfsfile,ios::out|ios::app);
	if(!of.is_open()){
		cout<<"Open file minDFSCODE.txt fail"<<endl;
		return;
	}

	//Kiểm tra số cạnh của dfscode
	int no_edges_in_dfscode = dfscode.size();
	//cout<<"so canh cua mindfscode: "<<no_edges_in_dfscode<<endl;


	of<<"d "<<nthDFSCODE<<" "<<dfscode.size()<<endl;
	++nthDFSCODE;
	for(int i=0;i<dfscode.size();++i){
		of<<dfscode.at(i).from<<" "<<dfscode.at(i).to<<" "<<dfscode.at(i).fromlabel<<" "<<dfscode.at(i).elabel<<" "<<dfscode.at(i).tolabel<<endl;
	}
}

//use
void write_array(int *a, int n,char *filename="temp.csv"){
	//Mở file minDFSCODE.txt để ghi thêm vào
	fstream of;
	//Mở file để ghi thêm vào
	of.open(filename,ios::out|ios::app);

	if(!of.is_open()){
		cout<<"Open file: "<<filename<< " fail"<<endl;
		return;
	}

	for(int i=0;i<n;++i){
		of<<a[i]<<endl;
	}
	of.flush();
	of.close();
	std::printf("\nWrite %s successfully",filename);
}

//use
PMS::PMS()
{
	Lv=0;
	Le=0;
	maxOfVer=0;
	numberOfGraph=0;
	minLabel = -1;
	maxId = -1;	
}

//use
PMS::~PMS()
{
	if(hLevelEXT.size()>0){
		hLevelEXT.clear();
	}

	if(hdb.size()!=0){
		for (int i = 0; i < hdb.size(); i++)
		{
			hipFree(hdb.at(i).dO);
			hipFree(hdb.at(i).dLO);
			hipFree(hdb.at(i).dN);
			hipFree(hdb.at(i).dLN);
		}
		hdb.clear();
	}
	if(hExtension.size()!=0){
		for (int i = 0; i < hExtension.size(); i++)
		{
			CUCHECK(hipFree(hExtension.at(i).dExtension));
		}
		hExtension.clear();
	}

	if(hUniEdge.size()!=0){
		for (int i = 0; i < hUniEdge.size(); i++)
		{
			CUCHECK(hipFree(hUniEdge.at(i).dUniEdge));
		}
		hUniEdge.clear();
	}

	//if(hUniEdgeSatisfyMinsup.size()!=0)
	//{
	//	for (int i = 0; i < hUniEdgeSatisfyMinsup.size(); i++)
	//	{			
	//		hipFree(hUniEdgeSatisfyMinsup.at(i).dUniEdge);
	//		free(hUniEdgeSatisfyMinsup.at(i).hArrSup);					
	//	}
	//	hUniEdgeSatisfyMinsup.clear();
	//}
}

//use
bool fexists(const char *filename)
{
  ifstream ifile(filename);
  return ifile;
}

//use
void PMS::prepareDataBase()
{
	//unsigned int minsup = 5000;
	unsigned int minsup = 14;
	unsigned int maxpat = 2;
	//unsigned int maxpat = 0x00000000;
	unsigned int minnodes = 0;
	bool where = true;
	bool enc = false;
	bool directed = false;

	//int opt;
	char* fname;
	//fname = "Klesscus";
	//fname = "Klessorigin";
	//fname = "KlessoriginCust1";
	//fname= "G0G1G2_custom"; //Kết quả giống với gSpan
	//fname= "G0G1G2_custom1"; //Kết quả giống với gSpan
	fname="Chemical_340Origin";
	//fname="dbgraph";


	ofstream fout("result.txt");
	char* minDFSCODE = "minDFSCODE.txt";
	if(fexists(minDFSCODE)==true)
	{
		remove(minDFSCODE);
		cout<<"Xoa file minDFSCODE dang ton tai"<<endl;
	}
	//Chuyển dữ liệu từ fname sang TRANS
	run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);
	maxOfVer=findMaxVertices();
	numberOfGraph=noGraphs();
	int sizeOfarrayO=maxOfVer*numberOfGraph;
	//Tạo mảng arrayO có kích thước D*m
	int* arrayO = new int[sizeOfarrayO];
	if(arrayO==NULL)
	{
		PMS_PRINT("\n!!!Memory Problem ArrayO");
		exit(0);
	}else
	{
		// gán giá trị cho các phần tử mảng bằng -1
		memset(arrayO, -1, sizeOfarrayO*sizeof(int));
	}
	//Tổng bậc của tất cả các đỉnh trong csdl đồ thị TRANS
	unsigned int noDeg;
	noDeg = sumOfDeg();
	unsigned int sizeOfArrayN=noDeg;
	//Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	int* arrayN = new int[sizeOfArrayN];
	if(arrayN==NULL)
	{
		PMS_PRINT("\n!!!Memory Problem ArrayN");
		exit(0);
	}else
	{
		memset(arrayN, -1, noDeg*sizeof(int));
	}
	 //Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	int* arrayLO = new int[sizeOfarrayO];
	if(arrayLO==NULL)
	{
		PMS_PRINT("\n!!!Memory Problem ArrayLO");
		exit(0);
	}else
	{
		memset(arrayLO, -1, sizeOfarrayO*sizeof(int));
	}


	//Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	int* arrayLN = new int[noDeg];
	if(arrayLN==NULL){
		PMS_PRINT("\n!!!Memory Problem ArrayLN");
		exit(0);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}

	importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);

	/*write_array(arrayO,sizeOfarrayO,"arrayO.csv");
	write_array(arrayLO,sizeOfarrayO,"arrayLO.csv");
	write_array(arrayN,noDeg,"arrayN.csv");
	write_array(arrayLN,noDeg,"arrayLN.csv");*/

	//kích thước của dữ liệu
	size_t nBytesO = sizeOfarrayO*sizeof(int);
	size_t nBytesN = noDeg*sizeof(int);

	DB graphdb;
	graphdb.noElemdO = sizeOfarrayO;
	graphdb.noElemdN = noDeg;

	CUCHECK(hipMalloc((void**)&graphdb.dO,nBytesO));
	//Cấp phát bộ nhớ trên GPU được quản lý bởi pointer dLO
	CUCHECK(hipMalloc((void**)&graphdb.dLO,nBytesO));
	CUCHECK(hipMalloc((void**)&graphdb.dN,nBytesN));
	CUCHECK(hipMalloc((void**)&graphdb.dLN,nBytesN));

	//Chép dữ liệu từ mảng arrayO trên CPU sang GPU được quản lý bởi pointer dO
	CUCHECK(hipMemcpy(graphdb.dO,arrayO,nBytesO,hipMemcpyHostToDevice));
	//	delete(arrayO);
	CUCHECK(hipMemcpy(graphdb.dLO,arrayLO,nBytesO,hipMemcpyHostToDevice));
	//delete(arrayLO);
	CUCHECK(hipMemcpy(graphdb.dN,arrayN,nBytesN,hipMemcpyHostToDevice));
	//delete(arrayN);
	CUCHECK(hipMemcpy(graphdb.dLN,arrayLN,nBytesN,hipMemcpyHostToDevice));
	//delete(arrayLN);
	//pms.db.push_back(graphdb); //Đưa cơ sở dữ liệu vào vector db
	//pms.countNumberOfDifferentValue(pms.db.at(0).dLO,pms.db.at(0).noElemdO,pms.Lv);
	//pms.countNumberOfDifferentValue(pms.db.at(0).dLN,pms.db.at(0).noElemdN,pms.Le);
	
	hdb.push_back(graphdb); //Đưa cơ sở dữ liệu vào vector db
	countNumberOfDifferentValue(hdb.at(0).dLO,hdb.at(0).noElemdO,Lv);
	countNumberOfDifferentValue(hdb.at(0).dLN,hdb.at(0).noElemdN,Le);
	//pms.printdb();
	return;
}

//bool PMS::checkArray(int *hostRef, int *gpuRef, const int N) {
//	bool result=true;
//	double epsilon = 1.0E-8;
//	int match = 1;
//	for (int i = 0; i < N; i++) {
//		if ((float)(abs(hostRef[i] - gpuRef[i])) > epsilon) {
//			match = 0;
//			result=false;
//			PMS_PRINT("Arrays do not match!\n");
//			PMS_PRINT("host %5.2f gpu %5.2f at current %d\n",
//				hostRef[i], gpuRef[i], i);
//			break;
//		}
//	}
//	if (match){
//		PMS_PRINT("Arrays match.\n\n");		
//	}
//
//	return result;
//}


//void PMS::displayArray(int *p, const unsigned int pSize=0)
//{
//	for(int i=0;i<pSize;i++){
//		PMS_PRINT("P[%d]:%d ",i,p[i]);
//	}
//	PMS_PRINT("\n");
//	return;
//}

//use
void PMS::displayHostArray(int *&p, const unsigned int pSize=0)
{
	std::printf("\n");
	for(int i=0;i<pSize;i++){
		PMS_PRINT("[%d]:%d ",i,p[i]);
	}
	PMS_PRINT("\n");
	return;
}

//__global__ void kernelPrintdArr(int *dArr,unsigned int noElem){
//	int i = blockDim.x*blockIdx.x + threadIdx.x;
//	if(i<noElem){
//		PMS_PRINT("A[%d]:%d ",i,dArr[i]);
//	}
//}


//void PMS::printdb(){
//	PMS_PRINT("\n *********** Lv, Le **********\n");
//	PMS_PRINT("\n Lv:%d",Lv);
//	PMS_PRINT("\n Le:%d",Le);
//	for (int i = 0; i < hdb.size(); i++)
//	{
//		unsigned int noElem =  hdb.at(i).noElemdO;	
//
//
//		dim3 block(blocksize);
//		dim3 grid((noElem + block.x -1)/block.x);
//		PMS_PRINT("\n ********* dO *********\n");
//		kernelPrintdArr<<<grid,block>>>(hdb.at(i).dO,noElem);
//		hipDeviceSynchronize();
//		PMS_PRINT("\n");
//
//		PMS_PRINT("\n ********* dLO *********\n");
//		kernelPrintdArr<<<grid,block>>>(hdb.at(i).dLO,noElem);
//		hipDeviceSynchronize();
//		PMS_PRINT("\n");
//
//		unsigned int noElemdN = hdb.at(i).noElemdN;
//		dim3 blocka(blocksize);
//		dim3 grida((noElemdN + blocka.x -1)/blocka.x);
//
//		PMS_PRINT("\n ********* dN *********\n");
//		kernelPrintdArr<<<grida,blocka>>>(hdb.at(i).dN,noElemdN);
//		hipDeviceSynchronize();
//		PMS_PRINT("\n");
//
//		PMS_PRINT("\n ********* dLN *********\n");
//		kernelPrintdArr<<<grida,blocka>>>(hdb.at(i).dLN,noElemdN);
//		hipDeviceSynchronize();
//		PMS_PRINT("\n");
//	}
//}


//__global__ void kernelMyScanV(int *dArrInput,int noElem,int *dResult){
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	if(i<noElem){
//		if(i==0){
//			dResult[i]=0;
//		}else
//		{
//			int temp=0;
//			for (int j = 0; j <= (i-1); j++)
//			{
//				temp=temp + dArrInput[j];
//			}
//			dResult[i]=temp;
//		}
//	}
//}

//use
__global__ void kernelCopyDeviceArray(int *dArrInput,int *dResult,int noElem)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<noElem)
	{
		dResult[i]=dArrInput[i];
	}
}

//use
__global__ void kernelCopyDevice(int** dPointerArr,int* dArr,int at)
{
	dPointerArr[at] = dArr;
	//PMS_PRINT("\n dPointerArr:%d, dArr:%d",dPointerArr[at],dArr);
}

//use
__global__ void kernelCopyDeviceEXT(EXT** dPointerArr,EXT* dArr,int at)
{
	dPointerArr[at] = dArr;
}

//use
void myReduce(int *dArrInput,int noElem,int &hResult)
{
	CudaContext& cdactx = *ctx;
	hResult = Reduce(dArrInput,noElem,cdactx);
	/*cout<<"reduce output: "<<hResult<<endl;*/
}

//use
void  myScanV(int *dArrInput,int noElem,int *&dResult)
{
	dim3 block(blocksize);
	dim3 grid((noElem + block.x -1)/block.x);

	CUCHECK(hipMalloc((void**)&dResult,noElem * sizeof(int)));
	//Copy dArrInput to dResult
	kernelCopyDeviceArray<<<grid,block>>>(dArrInput,dResult,noElem);
	CUCHECK(hipDeviceSynchronize());
	CudaContext& cdactx = *ctx;
	mgpu::ScanExc(dResult, noElem,cdactx);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	return;
}

//use
void get_idx(int*& dArrInput,int& noElem,int*& dResult)
{
	dim3 block(blocksize);
	dim3 grid((noElem + block.x -1)/block.x);

	CUCHECK(hipMalloc((void**)&dResult,noElem * sizeof(int)));

	//Copy dArrInput to dResult
	CUCHECK(hipMemcpy(dResult,dArrInput,noElem*sizeof(int),hipMemcpyDeviceToDevice))

	//displayDeviceArr(dResult,noElem);
	CudaContext& cdactx = *ctx;
	mgpu::ScanExc(dResult, noElem,cdactx);
	//displayDeviceArr(dResult,noElem);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}

//use
void  myReduction(int *dArrInput,int noElem,int &hResult){

	CudaContext& cdactx = *ctx;
	int total = Reduce(dArrInput, noElem, cdactx);
	//PMS_PRINT("Reduction total: %d\n\n", total);
	hResult = total;

	return;
}


//use
__global__ void kernelCountNumberOfLabelVertex(int *d_LO,int *d_Lv,unsigned int sizeOfArrayLO){
	int i= blockDim.x*blockIdx.x + threadIdx.x;
	if(i<sizeOfArrayLO){
		if(d_LO[i]!=-1){
			d_Lv[d_LO[i]]=1;
		}
	}
}

//chưa sửa đối tham chiếu

//use
void sumUntilReachZero(int *h_Lv,unsigned int n,int &result)
{
	for(int i=0;i<n && h_Lv[i]!=0;++i)
	{
		++result;
	}
}

//chưa sửa thành đối tham chiếu

//use
void  PMS::countNumberOfDifferentValue(int* d_LO,unsigned int sizeOfArrayLO, unsigned int &numberOfSaperateVertex){
	numberOfSaperateVertex=0;
	size_t nBytesLv = sizeOfArrayLO*sizeof(int);
	//cấp phát mảng d_Lv trên device
	int *d_Lv;
	CUCHECK(hipMalloc((int**)&d_Lv,nBytesLv));
	CUCHECK(hipMemset(d_Lv,0,nBytesLv));
	
	//Cấp phát threads
	dim3 block(blocksize);
	dim3 grid((sizeOfArrayLO+block.x-1)/block.x);
	kernelCountNumberOfLabelVertex<<<grid,block>>>(d_LO,d_Lv,sizeOfArrayLO);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	int* h_Lv=NULL;
	h_Lv=(int*)malloc(nBytesLv);
	if(h_Lv==NULL)
	{
		PMS_PRINT("h_Lv malloc memory fail");
		exit(0);
	}
	CUCHECK(hipMemcpy(h_Lv,d_Lv,nBytesLv,hipMemcpyDeviceToHost));
	
	int result=0;
	sumUntilReachZero(h_Lv,sizeOfArrayLO,result);
	numberOfSaperateVertex=result;

	CUCHECK(hipFree(d_Lv));
	return;
}

//use
__global__ void kernelGetAndStoreExtension(int *d_O,int *d_LO,unsigned int numberOfElementd_O, \
										   int *d_N,int *d_LN,unsigned int numberOfElementd_N,Extension *d_Extension)
{
	//Kernel trích tất cả các mở rộng hợp lệ ban đầu vào mảng d_Extension
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<numberOfElementd_O){
		if (d_O[i]!=-1){
			int j;
			int ek;
			//PMS_PRINT("\nThread:%d",i);	
			for(j=i+1;j<numberOfElementd_O;++j){	
				if(d_O[j]!=-1) {break;}	
			}

			if (j==numberOfElementd_O) {
				ek=numberOfElementd_N;
			}
			else
			{
				ek=d_O[j];
			}
			//PMS_PRINT("\n[%d]:%d",i,ek);
			for(int k=d_O[i];k<ek;k++){
				//do something
				int index= k;
				d_Extension[index].vi=0; //không cần gán nữa vì đã khởi tạo trong định nghĩa cấu trúc Extension
				d_Extension[index].vj=1;
				d_Extension[index].li=d_LO[i];
				d_Extension[index].lij=d_LN[k];
				d_Extension[index].lj=d_LO[d_N[k]];
				d_Extension[index].vgi=i;
				d_Extension[index].vgj=d_N[k];
				//PMS_PRINT("\n[%d]:%d",i,index);
				/*PMS_PRINT("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",k,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,
				d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);*/
			}
		}
	}
}

//use
void PMS::getAndStoreExtension(Extension *&d_Extension)
{
	dim3 block(blocksize);
	unsigned int numberOfElementd_O = hdb.at(0).noElemdO;
	dim3 grid((numberOfElementd_O+block.x-1)/block.x);

	kernelGetAndStoreExtension<<<grid,block>>>(hdb.at(0).dO,hdb.at(0).dLO,numberOfElementd_O,hdb.at(0).dN,hdb.at(0).dLN,hdb.at(0).noElemdN,d_Extension);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}

//use
void PMS::extractAllEdgeInDB()
{
	arrExtension arrE;
	//cấp phát bộ nhớ cho d_Extension
	arrE.noElem =hdb.at(0).noElemdN; //Lấy số lượng cạnh của tất cả các đồ thị
	size_t nBytesOfArrayExtension = arrE.noElem*sizeof(Extension); //Cấp phát bộ nhớ để lưu trữ tất cả các mở rộng ban đầu tương ứng với số lượng cạnh thu được;

	CUCHECK(hipMalloc((Extension**)&arrE.dExtension,nBytesOfArrayExtension));

	//Trích tất cả các cạnh từ database rồi lưu vào d_Extension
	getAndStoreExtension(arrE.dExtension);
	hExtension.push_back(arrE);
	return;
}

//use
__global__ void	kernelValidEdge(Extension *d_Extension,int *dV,unsigned int numberElementd_Extension){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){	
		if(d_Extension[i].li<=d_Extension[i].lj){
			dV[i]=1;
		}
	}
}

//use
void validEdge(Extension *d_Extension,int *&dV,unsigned int numberElementd_Extension)
{
	dim3 block(blocksize);
	dim3 grid(numberElementd_Extension+block.x-1/block.x);
	std::printf("\n gird:%d block:%d");
	kernelValidEdge<<<grid,block>>>(d_Extension,dV,numberElementd_Extension);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}

//use
void PMS::displayDeviceArr(int *dArr,int noElem)
{
	int *temp = (int*)malloc(sizeof(int)*noElem);
	if(temp==NULL)
	{
		PMS_PRINT("\n Malloc temp in displayDeviceArr() failed");
		FCHECK(-1);
	}
	CUCHECK(hipMemcpy(temp,dArr,noElem*sizeof(int),hipMemcpyDeviceToHost));
	for (int i = 0; i < noElem; i++)
	{
		PMS_PRINT(" A[%d]:%d  ",i,temp[i]);
	}
	free(temp);
	return;
}


//use
void PMS::displayDeviceArr(float* &dArr,int &noElem)
{
	try
	{
		float *temp = (float*)malloc(sizeof(float)*noElem);
		if(temp == nullptr){FCHECK(-1);}
		CUCHECK(hipMemcpy(temp,dArr,noElem*sizeof(float),hipMemcpyDeviceToHost));
		cout<<endl;
		for (int i = 0; i < noElem; i++)
		{
			int a = (int)temp[i];
			PMS_PRINT(" A[%d]:%d  ",i,a);
		}
		free(temp);
	}
	catch(...)
	{
		FCHECK(-1);
	}
}

//use
__global__ void kernelGetSize(int *dV,int *dVScanResult,int noElem,int *size)
{
	*size = dVScanResult[noElem-1];
	if(dV[noElem-1]==1)
	{
		*size = *size + 1;
	}
}

//chưa sửa đối tham chiếu

//use
void getSizeBaseOnScanResult(int *dV,int *dVScanResult,int noElem,int &output)
{
	int temp=0;
	int *size=nullptr;
	CUCHECK(hipMalloc((void**)&size,sizeof(int)));
	CUCHECK(hipMemset(size,0,sizeof(int)));
	
	kernelGetSize<<<1,1>>>(dV,dVScanResult,noElem,size);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	CUCHECK(hipMemcpy(&temp,size,sizeof(int),hipMemcpyDeviceToHost));
	output = (int)temp;

	CUCHECK(hipFree(size));
	return;
}

//use
void getSizeBaseOnScanResultv2(int *&dV,int *&dVScanResult,int& noElem,int &output)
{
	int temp=0;
	int *size=nullptr;
	CUCHECK(hipMalloc((void**)&size,sizeof(int)));
	CUCHECK(hipMemset(size,0,sizeof(int)));

	kernelGetSize<<<1,1>>>(dV,dVScanResult,noElem,size);
	CUCHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());

	CUCHECK(hipMemcpy(&temp,size,sizeof(int),hipMemcpyDeviceToHost));
	output = (int)temp;

	CUCHECK(hipFree(size));
	return;
}

//use
void get_noElem_valid(int*& dV,int*& dVScanResult,int& noElem,int &output)
{
	int temp=0;
	int *size=nullptr;
	CUCHECK(hipMalloc((void**)&size,sizeof(int)));
	CUCHECK(hipMemset(size,0,sizeof(int)));

	kernelGetSize<<<1,1>>>(dV,dVScanResult,noElem,size);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	CUCHECK(hipMemcpy(&temp,size,sizeof(int),hipMemcpyDeviceToHost));
	output = (int)temp;
	CUCHECK(hipFree(size));
	return;
}

//use
void allocate_gpu_memory(EXT* &d_array,int noElem)
{
	size_t n_bytes = sizeof(EXT)*noElem;
	CUCHECK(hipMalloc((void**)&d_array,n_bytes));
	return;
}

//__global__ void kernelExtractValidExtension(Extension *d_Extension,int *dV,int *dVScanResult,int numberElementd_Extension,Extension *d_ValidExtension){
//	int i = blockIdx.x*blockDim.x + threadIdx.x;
//	if(i<numberElementd_Extension){
//		if(dV[i]==1){
//			int index = dVScanResult[i];
//			//PMS_PRINT("\nV[%d]:%d, index[%d]:%d,d_Extension[%d], d_Extension[%d]:%d\n",i,V[i],i,index[i],i,i,d_Extension[i].vgi);
//			d_ValidExtension[index].li=d_Extension[i].li;
//			d_ValidExtension[index].lj=d_Extension[i].lj;
//			d_ValidExtension[index].lij=d_Extension[i].lij;
//			d_ValidExtension[index].vgi=d_Extension[i].vgi;
//			d_ValidExtension[index].vgj=d_Extension[i].vgj;
//			d_ValidExtension[index].vi=d_Extension[i].vi;
//			d_ValidExtension[index].vj=d_Extension[i].vj;
//		}
//	}
//}


//use
__global__ void kernelExtractValidExtension_pure(Extension *d_Extension,int *dV,int *dVScanResult, \
												 int numberElementd_Extension,EXT *d_ValidExtension)
{
	//Trích các mở rộng duy nhất ban đầu
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){
		if(dV[i]==1){
			int index = dVScanResult[i];
			//PMS_PRINT("\nV[%d]:%d, index[%d]:%d,d_Extension[%d], d_Extension[%d]:%d\n",i,V[i],i,index[i],i,i,d_Extension[i].vgi);
			d_ValidExtension[index].li=d_Extension[i].li;
			d_ValidExtension[index].lj=d_Extension[i].lj;
			d_ValidExtension[index].lij=d_Extension[i].lij;
			d_ValidExtension[index].vgi=d_Extension[i].vgi;
			d_ValidExtension[index].vgj=d_Extension[i].vgj;
			d_ValidExtension[index].vi=d_Extension[i].vi;
			d_ValidExtension[index].vj=d_Extension[i].vj;
			d_ValidExtension[index].posRow = -1; //posRow ban đầu chưa gắn với bất kỳ embedding column row nào.
		}
	}
}

//hipError_t extractValidExtension(Extension *d_Extension,int *dV,int *dVScanResult, int numberElementd_Extension,Extension *&d_ValidExtension){
//	hipError_t cudaStatus;
//
//	//printfExtension(d_Extension,numberElementd_Extension);
//
//	dim3 block(blocksize);
//	dim3 grid((numberElementd_Extension+block.x)/block.x);
//
//	kernelExtractValidExtension<<<grid,block>>>(d_Extension,dV,dVScanResult,numberElementd_Extension,d_ValidExtension);
//
//	hipDeviceSynchronize();
//	cudaStatus=hipGetLastError();
//	if (cudaStatus != hipSuccess){
//		fprintf(stderr,"\nkernelGetValidExtension failed");
//		goto Error;
//	}
//
//Error:
//	return cudaStatus;
//}

//use
void extractValidExtension_pure(Extension *d_Extension,int *dV,int *dVScanResult, int numberElementd_Extension,EXT *&d_ValidExtension)
{
	dim3 block(blocksize);
	dim3 grid((numberElementd_Extension+block.x)/block.x);
	kernelExtractValidExtension_pure<<<grid,block>>>(d_Extension,dV,dVScanResult,numberElementd_Extension,d_ValidExtension);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}

//use
void PMS::getValidExtension_pure()
{
	//Phase 1: đánh dấu vị trí những cạnh hợp lệ (li<=lj)
	int *dV;
	size_t nBytesdV= hExtension.at(0).noElem *sizeof(int);
	CUCHECK(hipMalloc((void**)&dV,nBytesdV));
	CUCHECK(hipMemset(dV,0,nBytesdV));

	//Đánh dấu các mở rộng hợp lệ trong hExtension.at(0).dExtension
	validEdge(hExtension.at(0).dExtension,dV,hExtension.at(0).noElem);

	int* dVScanResult;
	CUCHECK(hipMalloc((void**)&dVScanResult,hExtension.at(0).noElem*sizeof(int)));
	CUCHECK(hipMemset(dVScanResult,0,hExtension.at(0).noElem*sizeof(int)));
	myScanV(dV,hExtension.at(0).noElem,dVScanResult);

	hLevelEXT.resize(1); 
	hLevelEXT.at(0).noElem=1;
	hLevelEXT.at(0).vE.resize(1);

	myReduction(dV,hExtension.at(0).noElem,hLevelEXT.at(0).vE.at(0).noElem);
	allocate_gpu_memory(hLevelEXT.at(0).vE.at(0).dArrExt,hLevelEXT.at(0).vE.at(0).noElem);
	extractValidExtension_pure(hExtension.at(0).dExtension,dV,dVScanResult,hExtension.at(0).noElem,hLevelEXT.at(0).vE.at(0).dArrExt);
	//free memory
	CUCHECK(hipFree(dV));
	CUCHECK(hipFree(dVScanResult));
	return;
}

//use
__global__ void kernelMarkLabelEdge_pure(EXT *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *d_allPossibleExtension)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		int index=	d_ValidExtension[i].li*Lv*Le + d_ValidExtension[i].lij*Lv + d_ValidExtension[i].lj;
		d_allPossibleExtension[index]=1;
	}
}

//use
void markLabelEdge_pure(EXT *&d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le, \
						int *&d_allPossibleExtension)
{
	//Các cạnh mở rộng hợp lệ có thể giống nhau==> Hàm này sẽ đi ánh xạ chúng vào không gian d_allPossibleExtension.
	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	kernelMarkLabelEdge_pure<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,Lv,Le,d_allPossibleExtension);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	return;
}

//use
__global__ void kernelCalcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult, \
													   unsigned int noElem_allPossibleExtension,UniEdge *d_UniqueExtension, \
													   unsigned int Le,unsigned int Lv)
{
	//Ánh xạ từ vị trí trong d_allPossibleExtension sang cạnh tương ứng trong UniEdge 
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_allPossibleExtension && d_allPossibleExtension[i]==1){
		int li,lj,lij;
		li=i/(Le*Lv);
		lij=(i%(Le*Lv))/Lv;
		lj=(i%(Le*Lv))-((i%(Le*Lv))/Lv)*Lv;
		int index = d_allPossibleExtensionScanResult[i];
		//PMS_PRINT("\n[%d]:%d li:%d lij:%d lj:%d",i,d_allPossibleExtensionScanResult[i],li,lij,lj);
		d_UniqueExtension[index].li=li;
		d_UniqueExtension[index].lij=lij;
		d_UniqueExtension[index].lj=lj;
	}
}


//use
void calcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult, \
									  unsigned int noElem_allPossibleExtension,UniEdge *&d_UniqueExtension, \
									  unsigned int noElem_d_UniqueExtension,unsigned int Le,unsigned int Lv)
{
	//Ánh xạ và lưu cạnh vào dUniEdge từ vị trí có giá trị 1 trong d_allPossibleExtension
	dim3 block(blocksize);
	dim3 grid((noElem_allPossibleExtension+block.x-1)/block.x);
	kernelCalcLabelAndStoreUniqueExtension<<<grid,block>>>(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,Le,Lv);

	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	return;
}


//use
void PMS::extractUniEdge()
{
	//Trích các mở rộng duy nhất
	//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
	unsigned int noElem_dallPossibleExtension=Le*Lv*Lv; //(Mỗi một đỉnh sẽ có thể có Le*Lv mở rộng. Mà chúng ta có Lv đỉnh, nên ta có: Le*Lv*Lv mở rộng có thể có).
	int *d_allPossibleExtension;

	//cấp phát bộ nhớ cho mảng d_allPossibleExtension
	CUCHECK(hipMalloc((void**)&d_allPossibleExtension, noElem_dallPossibleExtension*sizeof(int)));
	CUCHECK(hipMemset(d_allPossibleExtension, 0, noElem_dallPossibleExtension*sizeof(int)));
	
	//Hàm markLabelEdge hoạt động theo nguyên tắc: "Mỗi mở rộng trong dExtension đều có 1 vị trí duy nhất trong d_allPossibleExtension. Và nhiệm vụ của hàm này là bậc giá trị 1 cho vị trí đó"
	//CHECK(cudaStatus=markLabelEdge(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,Lv,Le,d_allPossibleExtension)); 
	markLabelEdge_pure(hLevelEXT.at(0).vE.at(0).dArrExt,hLevelEXT.at(0).vE.at(0).noElem,Lv,Le,d_allPossibleExtension);

	int *d_allPossibleExtensionScanResult;
	CUCHECK(hipMalloc((void**)&d_allPossibleExtensionScanResult,noElem_dallPossibleExtension*sizeof(int)));

	myScanV(d_allPossibleExtension,noElem_dallPossibleExtension,d_allPossibleExtensionScanResult);

	arrUniEdge strUniEdge;
	int noElem_d_UniqueExtension=0;
	//Tính kích thước của mảng d_UniqueExtension dựa vào kết quả exclusive scan
	getSizeBaseOnScanResult(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_dallPossibleExtension,noElem_d_UniqueExtension);

	strUniEdge.noElem = noElem_d_UniqueExtension;
	//Tạo mảng d_UniqueExtension với kích thước mảng vừa tính được
	CUCHECK(hipMalloc((void**)&strUniEdge.dUniEdge,noElem_d_UniqueExtension*sizeof(UniEdge)));
	CUCHECK(hipMemset(strUniEdge.dUniEdge,0,noElem_d_UniqueExtension*sizeof(UniEdge)));
	
	//Ánh xạ ngược lại từ vị trí trong d_allPossibleExtension thành cạnh và lưu kết quả vào d_UniqueExtension
	calcLabelAndStoreUniqueExtension(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_dallPossibleExtension,strUniEdge.dUniEdge,noElem_d_UniqueExtension,Le,Lv);

	hUniEdge.push_back(strUniEdge);
	CUCHECK(hipFree(d_allPossibleExtension));
	CUCHECK(hipFree(d_allPossibleExtensionScanResult));
	return;
}

//use
__global__ void kernelCalcBoundary_pure(EXT *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *dB,unsigned int maxOfVer)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension-1){
		unsigned int graphIdAfter=d_ValidExtension[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=d_ValidExtension[i].vgi/maxOfVer;
		unsigned int resultDiff=graphIdAfter-graphIdCurrent;
		dB[i]=resultDiff;
	}
}


//use
void calcBoundary_pure(EXT *&d_ValidExtension,unsigned int noElem_d_ValidExtension,int *&dB,unsigned int maxOfVer)
{
	//Xây dựng boundary cho các mở rộng hợp lệ trong d_ValidExtension để tính support
	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x)/block.x);

	kernelCalcBoundary_pure<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,dB,maxOfVer);

	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}

//use
__global__ void kernelGetLastElement(int *dScanResult,unsigned int noElem,int *output)
{
	output[0]=dScanResult[noElem-1];
}

//use
void getLastElement(int *dScanResult,unsigned int noElem,int &output)
{
	dim3 block(blocksize);
	dim3 grid((noElem+block.x-1)/block.x);

	int *value=nullptr;
	CUCHECK(hipMalloc((int**)&value,sizeof(int)));

	kernelGetLastElement<<<1,1>>>(dScanResult,noElem,value);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	CUCHECK(hipMemcpy(&output,value,sizeof(int),hipMemcpyDeviceToHost));

	//PMS_PRINT("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);

	CUCHECK(hipFree(value));
	return;
}

//use
__global__ void kernelSetValuedF_pure(UniEdge *dUniEdge,int noElemdUniEdge,EXT *dValidExtension,int noElemdValidExtension,int *dBScanResult,int *dF,int noElemF)
{
	int i = blockDim.x * blockIdx.x +threadIdx.x; //i là các mở rộng hợp lệ
	if(i<noElemdValidExtension){
		for (int j = 0; j < noElemdUniEdge; j++)//j là các mở rộng duy nhất
		{
			if(dUniEdge[j].li==dValidExtension[i].li && dUniEdge[j].lij==dValidExtension[i].lij && dUniEdge[j].lj==dValidExtension[i].lj){ //Nếu mở rộng hợp lệ có trong mở rộng duy nhất (? hình như sai sai)
				dF[dBScanResult[i]+j*noElemF]=1; //Bật 1 tại vị trí tương ứng.
			}
		}
	}
}


//Chưa sửa lại biến là tham chiếu

//use
void calcSupport_pure(UniEdge *dUniEdge,int noElemdUniEdge,EXT *dValidExtension,int noElemdValidExtension,int *dBScanResult,int *dF,int noElemF,int *&hResultSup)
{
	//Tính support cho các mở rộng duy nhất
	//Đánh dấu những đồ thị chứa embedding trong mảng d_F
	dim3 block(blocksize);
	dim3 grid((noElemdValidExtension+block.x - 1)/block.x);
	kernelSetValuedF_pure<<<grid,block>>>(dUniEdge,noElemdUniEdge,dValidExtension,noElemdValidExtension,dBScanResult,dF,noElemF);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	
	hResultSup = (int*)malloc(noElemdUniEdge*sizeof(int));
	if (hResultSup==NULL)
	{
		PMS_PRINT("\n Malloc hResultSup in calcSupport() failed");
		exit(0);
	}

	int *d_supports = nullptr;
	//timer.start();
	int status=0;
	SegReduce(dF,noElemdUniEdge,noElemF,d_supports);
	/*timer.stop();
	std::printf("Time myReduction for Segmented Extension: %f (ms)\n",timer.getTime());
	timer.reset();*/
	//displayDeviceArr(d_supports,noElemdUniEdge);
	CUCHECK(hipMemcpy(hResultSup,d_supports,noElemdUniEdge*sizeof(int),hipMemcpyDeviceToHost));
	CUCHECK(hipFree(d_supports));
	return;
}

//use
__global__ void	kernelMarkUniEdgeSatisfyMinsup(int *dResultSup,int noElemUniEdge,int *dV,unsigned int minsup){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge){
		int temp = dResultSup[i];
		if(temp >= minsup){
			dV[i]=1;
		}
	}
}

//use
__global__ void	kernelExtractUniEdgeSatifyMinsup_pure(UniEdge *dUniEdge,int *dV,int *dVScanResult,int noElemUniEdge,UniEdge *dUniEdgeSatisfyMinsup,int *dSup,int *dResultSup)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge){
		if(dV[i]==1){
			dUniEdgeSatisfyMinsup[dVScanResult[i]].vi = 0;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].vj = 1;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].li = dUniEdge[i].li;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lij = dUniEdge[i].lij;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lj=dUniEdge[i].lj;
			dSup[dVScanResult[i]]=dResultSup[i];
		}
	}
}

//use
__global__ void	kernelExtractUniEdgeSatifyMinsupV3(UniEdge *dUniEdge,int *dV,int *dVScanResult,int noElemUniEdge, \
												   UniEdge *dUniEdgeSatisfyMinsup,int *dSup,int *dResultSup)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge)
	{
		if(dV[i]==1)
		{
			dUniEdgeSatisfyMinsup[dVScanResult[i]].vi = dUniEdge[i].vi;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].vj = dUniEdge[i].vj;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].li = dUniEdge[i].li;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lij = dUniEdge[i].lij;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lj=dUniEdge[i].lj;
			dSup[dVScanResult[i]]=dResultSup[i];
		}
	}
}

//chưa sửa lại đối tham chiếu

//use
void PMS::extractUniEdgeSatisfyMinsup(int *hResultSup,int noElemUniEdge,unsigned int minsup)
{
	//1. Cấp phát mảng trên device có kích thước bằng noElemUniEdge
	int *dResultSup=nullptr;
	CUCHECK(hipMalloc((void**)&dResultSup,noElemUniEdge*sizeof(int)));

	CUCHECK(hipMemcpy(dResultSup,hResultSup,noElemUniEdge*sizeof(int),hipMemcpyHostToDevice));

	//2. Đánh dấu 1 trên dV cho những phần tử thoả minsup
	int *dV=nullptr;
	CUCHECK(hipMalloc((void**)&dV,noElemUniEdge*sizeof(int)));
	
	CUCHECK(hipMemset(dV,0,sizeof(int)*noElemUniEdge));

	dim3 block(blocksize);
	dim3 grid((noElemUniEdge + block.x - 1)/block.x);

	kernelMarkUniEdgeSatisfyMinsup<<<grid,block>>>(dResultSup,noElemUniEdge,dV,minsup);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());


	int *dVScanResult=nullptr;
	CUCHECK(hipMalloc((void**)&dVScanResult,noElemUniEdge*sizeof(int)));
	myScanV(dV,noElemUniEdge,dVScanResult);

	hLevelUniEdgeSatisfyMinsup.resize(1);
	hLevelUniEdgeSatisfyMinsup.at(0).vecUES.resize(1);
	getSizeBaseOnScanResult(dV,dVScanResult,noElemUniEdge, \
		hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).noElem);

	CUCHECK(hipMalloc((void**)&hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).dArrUniEdge, \
		hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).noElem*sizeof(UniEdge)));
	
	hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).hArrSupport = (int*)malloc(sizeof(int)*hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).noElem);
	if (hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).hArrSupport ==NULL)
	{
		PMS_PRINT("\n malloc hArrSup of hUniEdgeSatisfyMinsup failed()");
		exit(0);
	}

	int *dSup=nullptr;
	CUCHECK(hipMalloc((void**)&dSup,hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).noElem*sizeof(int)));

	dim3 blocka(blocksize);
	dim3 grida((noElemUniEdge + blocka.x -1)/blocka.x);
	kernelExtractUniEdgeSatifyMinsup_pure<<<grida,blocka>>>(hUniEdge.at(0).dUniEdge,dV, \
		dVScanResult,noElemUniEdge, \
		hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).dArrUniEdge,dSup,dResultSup);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	CUCHECK(hipMemcpy(hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).hArrSupport,dSup, \
		sizeof(int)*hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0).noElem,hipMemcpyDeviceToHost));
	
	CUCHECK(hipFree(dResultSup));
	CUCHECK(hipFree(dV));
	CUCHECK(hipFree(dVScanResult));
	CUCHECK(hipFree(dSup));
	if(hUniEdge.at(0).noElem>0)
	{
		CUCHECK(hipFree(hUniEdge.at(0).dUniEdge));
		hUniEdge.clear();
	}
	return;
}

//use
void PMS::computeSupport()
{
	/* Xây dựng Boundary cho mảng d_ValidExtension */
	//1. Cấp phát một mảng d_B và gán các giá trị 0 cho mọi phần tử của d_B
	unsigned int noElement_dB=hLevelEXT.at(0).vE.at(0).noElem;
	int* dB = nullptr;
	CUCHECK(hipMalloc((int**)&dB,noElement_dB*sizeof(int)));
	CUCHECK(hipMemset(dB,0,noElement_dB*sizeof(int)));

	
	//Gián giá trị boundary cho d_B
	calcBoundary_pure(hLevelEXT.at(0).vE.at(0).dArrExt,noElement_dB,dB,maxOfVer);

	//2. Exclusive Scan mảng d_B
	int* dBScanResult;
	CUCHECK(hipMalloc((int**)&dBScanResult,noElement_dB*sizeof(int)));
	CUCHECK(hipMemset(dBScanResult,0,noElement_dB*sizeof(int)));
	
	myScanV(dB,noElement_dB,dBScanResult);
	

	//3. Tính độ hỗ trợ cho các mở rộng trong d_UniqueExtension
	//3.1 Tạo mảng d_F có số lượng phần tử bằng với giá trị cuối cùng của mảng d_scanB_Result cộng 1 và gán giá trị 0 cho các phần tử.
	int noElemF=0;
	getLastElement(dBScanResult,noElement_dB,noElemF);
	++noElemF;

	int noElem_d_UniqueExtension= hUniEdge.at(0).noElem;
	int *dF;
	CUCHECK(hipMalloc((int**)&dF,noElem_d_UniqueExtension*noElemF*sizeof(int)));
	CUCHECK(hipMemset(dF,0,noElem_d_UniqueExtension*noElemF*sizeof(int)));
	
	int *hResultSup=nullptr;
	calcSupport_pure(hUniEdge.at(0).dUniEdge, \
		hUniEdge.at(0).noElem,hLevelEXT.at(0).vE.at(0).dArrExt, \
		hLevelEXT.at(0).vE.at(0).noElem,dBScanResult,dF,noElemF,hResultSup);

	extractUniEdgeSatisfyMinsup(hResultSup,noElem_d_UniqueExtension,minsup);
	CUCHECK(hipFree(dBScanResult));
	CUCHECK(hipFree(dB));
	return;
}

//use
__global__ void kernel_generate_segment_index(int* SegmentStarts,\
											  int noElem_segment,\
											  int noElem_of_graph_per_unique_ext)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_segment)
	{
		SegmentStarts[i]=i*noElem_of_graph_per_unique_ext;
	}
}

//use
void generate_segment_index(int noElem_of_graph_per_unique_ext,\
						   int number_unique_extension,\
						   int *&SegmentStarts)
{
	int noElem_segment = number_unique_extension;
	dim3 block(blocksize);
	dim3 grid((noElem_segment + block.x -1)/block.x);
	CUCHECK(hipMalloc((void**)&SegmentStarts,noElem_segment*sizeof(int)));

	kernel_generate_segment_index<<<grid,block>>>(SegmentStarts, noElem_segment, noElem_of_graph_per_unique_ext);
	CUCHECK(hipDeviceSynchronize());
	
	CUCHECK(hipGetLastError());
	
	return;
}

//use
void SegReduce(int* dF,int number_unique_extension,int noElem_of_graph_per_unique_ext,int *&resultsDevice) 
{
	CudaContext& context = *ctx;
	int count = number_unique_extension*noElem_of_graph_per_unique_ext;
	int *SegmentStarts = nullptr;
	generate_segment_index(noElem_of_graph_per_unique_ext,number_unique_extension,SegmentStarts);
	const int NumSegments = number_unique_extension;
	CUCHECK(hipMalloc((void**)&resultsDevice,number_unique_extension*sizeof(int)));
	SegReduceCsr(dF, SegmentStarts, count, number_unique_extension,\
				false, resultsDevice, (int)0, mgpu::plus<int>(), context);

	CUCHECK(hipFree(SegmentStarts));
	return;
}

//use
__global__ void kernelGetGraph(int *dV,int noElemdV,int *d_kq,int *dVScanResult)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdV)
	{
		if(dV[i]!=0){
			d_kq[dVScanResult[i]]=i;
		}
	}
}


//use
__global__ void kernelGetLastElementEXT(EXT *inputArray,int noEleInputArray,int *value,unsigned int maxOfVer)
{
	//Lấy global vertex id chia cho tổng số đỉnh của đồ thị (maxOfVer). 
	//Ở đây các đồ thị luôn có số lượng đỉnh bằng nhau (maxOfVer)
	*value = inputArray[noEleInputArray-1].vgi/maxOfVer; 
}

//chưa sửa đối tham chiếu

//use
void getLastElementEXT(EXT *inputArray,int numberElementOfInputArray,int &outputValue,unsigned int maxOfVer)
{
	int *temp=nullptr;
	CUCHECK(hipMalloc((int**)&temp,sizeof(int)));

	/* Lấy graphId chứa embedding cuối cùng */
	kernelGetLastElementEXT<<<1,1>>>(inputArray,numberElementOfInputArray,temp,maxOfVer);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	CUCHECK(hipMemcpy(&outputValue,temp,sizeof(int),hipMemcpyDeviceToHost));

	if(temp!=nullptr) CUCHECK(hipFree(temp));
	return;
}

//use
__global__ void kernelGetGraphIdContainEmbeddingv2(int vi,int vj,int li,int lij,int lj, \
												   EXT *d_ValidExtension,int noElem_d_ValidExtension, \
												   int *dV,unsigned int maxOfVer)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension)
	{
		if(d_ValidExtension[i].li == li && d_ValidExtension[i].lij == lij && d_ValidExtension[i].lj == lj && \
			d_ValidExtension[i].vi == vi && d_ValidExtension[i].vj == vj)
		{
			int graphid = (d_ValidExtension[i].vgi/maxOfVer);
			dV[graphid]=1;
		}
	}
}

//use
void PMS::get_graphid(UniEdge &edge,int *&hArrGraphId,int &noElemhArrGraphId,EXT *dArrEXT,int noElemdArrEXT)
{
	dim3 block(blocksize);
	dim3 grid((noElemdArrEXT+block.x-1)/block.x);

	int *d_graphid=nullptr;
	int noElem_d_graphid=0;
	//How many graphs contains embeddings of DFS_CODE?
	getLastElementEXT(dArrEXT,noElemdArrEXT,noElem_d_graphid,maxOfVer);
	++noElem_d_graphid;

	CUCHECK(hipMalloc((void**)&d_graphid,noElem_d_graphid*sizeof(int)));
	CUCHECK(hipMemset(d_graphid,0,noElem_d_graphid*sizeof(int)));

	kernelGetGraphIdContainEmbeddingv2<<<grid,block>>>(edge.vi,edge.vj,edge.li,edge.lij,edge.lj,dArrEXT, \
		noElemdArrEXT,d_graphid,maxOfVer);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	int *dVScanResult=nullptr; //2. need hipFree
	CUCHECK(hipMalloc((void**)&dVScanResult,noElem_d_graphid*sizeof(int)));
	CUCHECK(hipMemset(dVScanResult,0,noElem_d_graphid*sizeof(int)));

	myScanV(d_graphid,noElem_d_graphid,dVScanResult);

	int noElem_kq=0;
	getSizeBaseOnScanResultv2(d_graphid,dVScanResult,noElem_d_graphid,noElem_kq);

	int *d_kq;
	CUCHECK(hipMalloc((void**)&d_kq,sizeof(int)*noElem_kq));

	dim3 blocka(blocksize);
	dim3 grida((noElem_d_graphid + blocka.x -1)/blocka.x);

	kernelGetGraph<<<grida,blocka>>>(d_graphid,noElem_d_graphid,d_kq,dVScanResult);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	hArrGraphId=(int*)malloc(sizeof(int)*noElem_kq);
	if(hArrGraphId==nullptr) {FCHECK(-1);}
	noElemhArrGraphId=noElem_kq;

	CUCHECK(hipMemcpy(hArrGraphId,d_kq,sizeof(int)*noElem_kq,hipMemcpyDeviceToHost));
	//free memory
	CUCHECK(hipFree(d_kq));
	CUCHECK(hipFree(d_graphid));
	CUCHECK(hipFree(dVScanResult));
	return;
}

//use
void PMS::MiningDeeper(EXTk &ext,UniEdgeStatisfyMinSup &UES)
{
	try
	{
		if (UES.noElem <= 0) return;
		UES.hArrUniEdge = (UniEdge*)malloc(sizeof(UniEdge)*UES.noElem);

		for(int idx_ues = 0; idx_ues < UES.noElem ; ++idx_ues)
		{
			if(UES.hArrUniEdge==nullptr) {return;}
			CUCHECK(hipMemcpy(UES.hArrUniEdge,UES.dArrUniEdge,sizeof(UniEdge)*UES.noElem,hipMemcpyDeviceToHost));
			DFS_CODE.add(UES.hArrUniEdge[idx_ues].vi,UES.hArrUniEdge[idx_ues].vj, \
				UES.hArrUniEdge[idx_ues].li,UES.hArrUniEdge[idx_ues].lij,UES.hArrUniEdge[idx_ues].lj);
			//Check minDFSCode
			if(is_min())
			{
				write_minDFS_CODE(DFS_CODE);
				//Đánh tất cả các embedding của unique edge trong UES.hArrUniEdge
				//Kết quả đánh dấu của lưu vào dValid
				int *dValid = nullptr;
				//Tại sao mark_edge không truyền vào posRow? Điều này có ảnh hưởng gì không?
				ext.mark_edge(UES.hArrUniEdge[idx_ues].vi,UES.hArrUniEdge[idx_ues].vj, \
					UES.hArrUniEdge[idx_ues].li,UES.hArrUniEdge[idx_ues].lij,UES.hArrUniEdge[idx_ues].lj,dValid);
				//Hiển thị các phần tử trong ext.
				//std::printf("\n********ext.show()************\n");
				//ext.show();

				//Tạo mảng index của các phần tử hợp lệ trong mảng mới.
				int *dIdx = nullptr;
				get_idx(dValid,ext.noElem,dIdx);
				//displayDeviceArr(dIdx,ext.noElem);

				//Tìm số lượng phần tử hợp lệ.
				//Lưu ý, tuỳ vào phần tử cuối của dValid có =1 hay không để tính noElem_valid cho đúng.
				int noElem_valid = 0;
				get_noElem_valid(dValid,dIdx,ext.noElem,noElem_valid);

				//get_graph_id
				int *hArrGraphId=nullptr;
				int noElemhArrGraphId=0;
				get_graphid(UES.hArrUniEdge[idx_ues],hArrGraphId,noElemhArrGraphId,ext.dArrExt,ext.noElem);
				//displayHostArray(hArrGraphId,noElemhArrGraphId);
				report(hArrGraphId,noElemhArrGraphId,UES.hArrSupport[idx_ues]);
				free(hArrGraphId);

				//build embedding for min DFS_CODE
				buildEmbedding(UES.hArrUniEdge[idx_ues],ext,dValid,dIdx);
				//Display Embedding columns.
				//for (int i = 0; i < hEm.size(); i++)
				//{
				//	cout<<endl;
				//	hEm.at(i).show();
				//	if(hEm.at(i).hBackwardEmbedding.size()>0)
				//	{
				//		for (int j = 0; j < hEm.at(i).hBackwardEmbedding.size(); j++)
				//		{
				//			std::printf("\nBackward:(%d;%d)\n:",i,j);
				//			hEm.at(i).hBackwardEmbedding.at(j).show();
				//		}
				//	}
				//}

				vector<EXTk> vecValidEXTk;
				//Find valid extension and return vector EXTk<i>
				findValidExtension(vecValidEXTk);
				//Trích các Unique Forward Extension ở tất cả các EXTk
				for (int idxEXTk = 0; idxEXTk < vecValidEXTk.size(); idxEXTk++)
				{
					if(vecValidEXTk.at(idxEXTk).noElem>0)
					{
						//vecValidEXTk.at(idxEXTk).show();
						vecValidEXTk.at(idxEXTk).extractUniForwardExtension(Lv,Le,DFS_CODE.maxId);
					}
				}
				//Trích các Unique Backward Extension ở EXTk cuối.
				//Các đỉnh trên RMP phải > 2 thì mới có khả năng có mở rộng backward từ đỉnh cuối của RMP.
				if (DFS_CODE.noElemOnRMP >2)
				{
					int lastIdxEXTk = vecValidEXTk.size()-1;
					//Nếu là phần tử cuối thì xét mở rộng backward
					if(vecValidEXTk.at(lastIdxEXTk).noElem > 0)
					{
						//vecValidEXTk.at(lastIdxEXTk).show();
						//Trích các unique edge backward.
						int* dRMP = nullptr;
						int* dRMPLabel = nullptr;
						int noElemMappingVj = 0;
						int vi = 0;
						int li = 0; //Có vẻ như tạo right most path trên device bị fail.
						buildRMPLabel(dRMP,dRMPLabel,noElemMappingVj,vi,li);
						vecValidEXTk.at(lastIdxEXTk).extractUniBackwardExtension( \
							Lv,Le, \
							DFS_CODE.noElemOnRMP, \
							dRMP,dRMPLabel, \
							noElemMappingVj,vi,li);
					}
				}

				//Compute support
				//Duyệt qua các unique trong mảng arrUniEdgea trong vecValidEXTk để tính support cho từng cạnh
				int lastidxEXTk = vecValidEXTk.size()-1;
				for (int idxvecValidEXTk = lastidxEXTk; idxvecValidEXTk >=0; idxvecValidEXTk--)
				{
					if (vecValidEXTk.at(idxvecValidEXTk).noElem>0)
					{
						vecValidEXTk.at(idxvecValidEXTk).findSupport(maxOfVer);
						//Trích các mở rộng forward thoả minsup
						vecValidEXTk.at(idxvecValidEXTk).extractStatisfyMinsup(minsup, \
							vecValidEXTk.at(idxvecValidEXTk).uniFE,vecValidEXTk.at(idxvecValidEXTk).uniFES);
						//Trích các mở rộng backward thoả minsup
						vecValidEXTk.at(idxvecValidEXTk).extractStatisfyMinsup(minsup, \
							vecValidEXTk.at(idxvecValidEXTk).uniBE,vecValidEXTk.at(idxvecValidEXTk).uniBES);
					}
				}

				//Duyet qua các EXTk và gọi MiningDeeper
				for (int idxvecValidEXTk = lastidxEXTk; idxvecValidEXTk >=0; idxvecValidEXTk--)
				{
					if (vecValidEXTk.at(idxvecValidEXTk).noElem>0)
					{
						//MiningDeeper cho backward truoc cho forward sau
						if(vecValidEXTk.at(idxvecValidEXTk).uniBES.noElem>0)
						{
							MiningDeeper(vecValidEXTk.at(idxvecValidEXTk),vecValidEXTk.at(idxvecValidEXTk).uniBES);
							//Đã xử lý xong uniBES có thể giải phóng
							vecValidEXTk.at(idxvecValidEXTk).uniBES.ReleaseMemory();
						}
						if(vecValidEXTk.at(idxvecValidEXTk).uniFES.noElem>0)
						{
							MiningDeeper(vecValidEXTk.at(idxvecValidEXTk),vecValidEXTk.at(idxvecValidEXTk).uniFES);
							//Đã xử lý xong uniFES có thể giải phóng
							vecValidEXTk.at(idxvecValidEXTk).uniFES.ReleaseMemory();
						}
						//Đã xử lý xong một phần tử EXTk
						vecValidEXTk.at(idxvecValidEXTk).ReleaseMemory();
					}
				}



				//Đã xử lý xong tất cả các phần tử trong EXTk
				vecValidEXTk.clear();
				CUCHECK(hipFree(dValid));
				CUCHECK(hipFree(dIdx));
				removeEmbedding();
			}
			//Khi khai thác xong thì gỡ bỏ cạnh vừa thêm ra khỏi DFS_CODE
			DFS_CODE.remove(UES.hArrUniEdge[idx_ues].vi,UES.hArrUniEdge[idx_ues].vj);
		}
		return;
	}
	catch(std::exception &exc)
	{
		cout<<exc.what();
		FCHECK(-1);
	}
}

//use
__global__ void kernelExtractBWEmbeddingRow(Embedding* dArrBWEmbedding,int *dV, \
											int *dVScanResult,int noElemdV,Embedding *dArrEmbedding)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdV)
	{
		if(dV[i]==1)
		{
			dArrBWEmbedding[dVScanResult[i]].idx = dArrEmbedding[i].idx;
			dArrBWEmbedding[dVScanResult[i]].vid = dArrEmbedding[i].vid;
		}
	}
}

//use
__global__ void	kernelExtractRowFromEXT(EXT *dArrExt,int noElemdArrExt,int *dV,int vj)
{
	//Đánh dấu những dòng nào (embeddings nào) trong Embedding column có mở rộng backward.
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdArrExt)
	{
		if(dArrExt[i].vj==vj)
		{
			dV[dArrExt[i].posRow]=1;
			//PMS_PRINT("\n Thread %d: dV[%d]:%d",i,dArrExt[i].posRow,dV[dArrExt[i].posRow]);
		}
	}
}

//use
__global__ void kernelSetValueForFirstTwoEmbeddingColumn(const EXT *d_ValidExtension,int noElem_d_ValidExtension, \
														 Embedding *dQ1,Embedding *dQ2,int *d_scanResult, \
														 int li,int lij,int lj)
{
	int i = blockDim.x *blockIdx.x +threadIdx.x;
	if(i<noElem_d_ValidExtension)
	{
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij == lij && d_ValidExtension[i].lj==lj)
		{
			dQ1[d_scanResult[i]].idx=-1;
			dQ1[d_scanResult[i]].vid=d_ValidExtension[i].vgi;

			dQ2[d_scanResult[i]].idx=d_scanResult[i];
			dQ2[d_scanResult[i]].vid=d_ValidExtension[i].vgj;
		}
	}
}

//use
__global__ void kernelSetValueForEmbeddingColumn(EXT *dArrExt,int noElemInArrExt,Embedding *dArrQ,int *dM, \
												 int *dMScanResult)
{
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElemInArrExt)
	{
		if(dM[i]==1)
		{
			int posRow = dArrExt[i].posRow;
			int vgj =dArrExt[i].vgj;
			dArrQ[dMScanResult[i]].idx=posRow;
			dArrQ[dMScanResult[i]].vid=vgj;
		}
	}
}

//use
void PMS::removeEmbedding()
{
	if(DFS_CODE.size() == 1)
	{
		removeFirstEmbedding();
	}
	else
	{
		//Kiểm tra xem hEm cuối có backward column hay không
		//Nếu có thì giải phóng embedding col backward.
		if(hEm.back().hBackwardEmbedding.size()>0)
		{
			CUCHECK(hipFree(hEm.back().hBackwardEmbedding.back().dArrEmbedding));
			hEm.back().hBackwardEmbedding.pop_back();
		}
		else
		{
			//Ngược lại thì giải phóng hEm
			CUCHECK(hipFree(hEm.back().dArrEmbedding));
			hEm.pop_back();
		}
	}
}

//use
void PMS::removeFirstEmbedding()
{
	CUCHECK(hipFree(hEm.at(1).dArrEmbedding));
	CUCHECK(hipFree(hEm.at(0).dArrEmbedding));
	hEm.pop_back();
	hEm.pop_back();
}

//use
void PMS::buildEmbedding(UniEdge &ue,EXTk &ext,int *&dValid,int *&dIdx)
{
	if(DFS_CODE.size() == 1)
	{
		buildFirstEmbedding(ue,ext,dValid,dIdx);
	}
	else if (ue.vi<ue.vj)
	{
		//Mở rộng embedding column forward.
		buildNewEmbeddingCol(ue,ext,dValid,dIdx);
	}
	else
	{
		//Mở rộng embedding column backward.
		buildBackwardEmbedding(ue,ext,dValid,dIdx);
	}
}

//use
void PMS::buildFirstEmbedding(UniEdge &ue,EXTk &ext,int*&dValid,int*&dIdx)
{
	//Mỗi phần tử của Vector sẽ quản lý 1 dArrEmbedding trên device. 
	//Khi cần thiết có thể tập hợp chúng lại thành 1 mảng trên device.
	hEm.resize(2);
	hEm.at(0).noElem;

	int noElemOfdArEmbedding=0;

	getSizeBaseOnScanResultv2(dValid,dIdx,ext.noElem,noElemOfdArEmbedding);

	hEm.at(0).noElem=hEm.at(1).noElem=noElemOfdArEmbedding;
	//Cấp phát bộ nhớ cho các embedding Columns.
	CUCHECK(hipMalloc((void**)&hEm.at(0).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));
	CUCHECK(hipMalloc((void**)&hEm.at(1).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));

	dim3 block(blocksize);
	dim3 grid((ext.noElem+block.x-1)/block.x);
	kernelSetValueForFirstTwoEmbeddingColumn<<<grid,block>>>(ext.dArrExt,ext.noElem,hEm.at(0).dArrEmbedding, \
		hEm.at(1).dArrEmbedding, dIdx,ue.li,ue.lij,ue.lj);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	//Gán giá trị prevCol cho các embedding columns. 
	hEm.at(0).prevCol=-1;
	hEm.at(1).prevCol=0;

	//show embedding column
	//hEm.at(0).show();
	//cout<<endl;
	//hEm.at(1).show();
	return;
}

//use
void PMS::buildNewEmbeddingCol(UniEdge &ue,EXTk &ext,int*&dValid,int*&dIdx)
{
	int currentSize = hEm.size();
	int newSize = currentSize + 1;
	int lastIdx = currentSize;
	hEm.resize(newSize);

	int noElemOfdArEmbedding=0;
	get_noElem_valid(dValid,dIdx,ext.noElem,noElemOfdArEmbedding);

	hEm.at(lastIdx).noElem = noElemOfdArEmbedding;
	//Cấp phát bộ nhớ cho các embedding Columns.
	CUCHECK(hipMalloc((void**)&hEm.at(lastIdx).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));

	dim3 block(blocksize);
	dim3 grid((ext.noElem+block.x-1)/block.x);
	kernelSetValueForEmbeddingColumn<<<grid,block>>>(ext.dArrExt,ext.noElem,hEm.at(lastIdx).dArrEmbedding,dValid,dIdx);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	//Gán giá trị prevCol cho các embedding columns. 
	hEm.at(lastIdx).prevCol=ue.vi;

	//show embedding column
	//hEm.at(lastIdx).show();
	return;
}

//use
void PMS::buildBackwardEmbedding(UniEdge& ue,EXTk& ext,int*& dValid,int*& dIdx)
{
	try
	{
		//Trích các row trong Embedding column cuối 
		//cout<<endl;
		//ue.print();
		//cout<<endl;
		//ext.show();
		//cout<<endl;
		//displayDeviceArr(dValid,ext.noElem);
		//cout<<endl;
		//displayDeviceArr(dIdx,ext.noElem);
		//cout<<endl;
		//for (int i = 0; i < hEm.size(); i++)
		//{
		//	hEm.at(i).show();
		//}

		//1. Khởi tạo một mảng <int> có số lượng phần tử bằng với số lượng Embedding gọi là dV 
		//	và đánh dấu các posRow chứa backward extension
		int *dV=nullptr;
		int noElemdV = 0;
		//Lấy số lượng Embedding dựa vào Embedding Column cuối.
		//noElemEmbedding tuỳ thuộc vào Embedding Colmn đó đã có backward embedding column nào hay chưa?
		if(hEm.back().hBackwardEmbedding.size()>0)
		{
			noElemdV = hEm.back().hBackwardEmbedding.back().noElem;
		}
		else
		{
			noElemdV = hEm.back().noElem;
		}
		//Cấp phát bộ nhớ cho dV trên device
		size_t nBytedV=noElemdV*sizeof(int);
		CUCHECK(hipMalloc((void**)&dV,nBytedV));
		CUCHECK(hipMemset(dV,0,nBytedV));

		//Trích các dòng từ EXT
		dim3 block(blocksize);
		dim3 grid((ext.noElem+block.x-1)/block.x);
		kernelExtractRowFromEXT<<<grid,block>>>(ext.dArrExt,ext.noElem,dV,ue.vj);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//PMS_PRINT("\n********dV**********\n");
		//FCHECK(displayDeviceArr(dV,noElemdV));

		//1.1 scan dV để biết kích thước của backward column embedding
		int *dVScanResult = nullptr;
		CUCHECK(hipMalloc((void**)&dVScanResult,noElemdV*sizeof(int)));

		get_idx(dV,noElemdV,dVScanResult);

		int noElemBW=0;
		get_noElem_valid(dV,dVScanResult,noElemdV,noElemBW);

		//2.Dựa vào dV để trích các embedding chứa backward extension sang một embedding column mới.
		int currentSize = hEm.back().hBackwardEmbedding.size();
		int newSizeOfBackwardEmCol = currentSize + 1;
		hEm.back().hBackwardEmbedding.resize(newSizeOfBackwardEmCol);

		CUCHECK(hipMalloc((void**)&hEm.back().hBackwardEmbedding.back().dArrEmbedding,noElemBW*sizeof(Embedding)));

		dim3 blocka(blocksize);
		dim3 grida((noElemdV + blocka.x -1)/blocka.x);
		if(hEm.back().hBackwardEmbedding.size()>=2)
		{
			kernelExtractBWEmbeddingRow<<<grida,blocka>>>(hEm.back().hBackwardEmbedding.back().dArrEmbedding, \
				dV,dVScanResult,noElemdV, \
				hEm.back().hBackwardEmbedding.at(currentSize-1).dArrEmbedding);
			CUCHECK(hipDeviceSynchronize());
			CUCHECK(hipGetLastError());
			//Cập nhật số lượng phần tử và preCol cho backward EmCol vừa mới thêm vào.
			hEm.back().hBackwardEmbedding.back().noElem = noElemBW;
			hEm.back().hBackwardEmbedding.back().prevCol = hEm.back().prevCol;
		}
		else
		{
			noElemdV = hEm.back().noElem;
			kernelExtractBWEmbeddingRow<<<grida,blocka>>>(hEm.back().hBackwardEmbedding.back().dArrEmbedding, \
				dV,dVScanResult,noElemdV, \
				hEm.back().dArrEmbedding);
			CUCHECK(hipDeviceSynchronize());
			CUCHECK(hipGetLastError());
			//Cập nhật số lượng phần tử và preCol cho backward EmCol vừa mới thêm vào.
			hEm.back().hBackwardEmbedding.back().noElem = noElemBW;
			hEm.back().hBackwardEmbedding.back().prevCol = hEm.back().prevCol;
		}
		//PMS_PRINT("\n ************* dArrBWEmbeddingCol ***********\n");
		//hEm.back().hBackwardEmbedding.back().show();
		//free memory
		CUCHECK(hipFree(dVScanResult));
		return;
	}
	catch(...)
	{
		FCHECK(-1);
	}
}


//use
__global__ void kernel_mark_edge(int vi,int vj,int li,int lij,int lj,EXT *ext,int *dValid,int noElem)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<noElem)
	{
		if(ext[i].li == li && ext[i].lij == lij && ext[i].lj == lj && ext[i].vi == vi && ext[i].vj == vj)
		{
			dValid[i]=1;
		}
	}
}

//use
void PMS::findValidExtension(vector<EXTk> &vecValidEXTk)
{
	try
	{
		//Create device RMP
		int* dRMP = nullptr; //đây chính là bwInfo.dVj
		int noElemRMP = 0;
		createRMP(dRMP,noElemRMP);

		//Get all Embedding Column of RMP.
		int noElemdEmCol = 0;
		Embedding** dEmCol = nullptr;
		getEmCol(dEmCol,noElemdEmCol);

		//Create mark EmCol RMP
		int* dEmRMP = nullptr;
		createMarkEmColRMP(dRMP,noElemRMP,dEmRMP);
		
		//existBackwardInfo bwInfo;
		int* dValidBackward = nullptr;
		buildExistBackwardInfo(dRMP,noElemRMP, \
			dValidBackward);

		//Tim bac lon nhat
		Embedding** dArrEmbeddingColRMP = nullptr;
		getEmColRMP(dArrEmbeddingColRMP,noElemRMP);

		// Lấy số lượng embedding
		int noElemEmbedding = 0;
		getnoElemEmbedding(noElemEmbedding);
		

		//Tìm bậc của các đỉnh vid của các embeding thuộc RMP
		int noElemVid = noElemRMP*noElemEmbedding;
		float *dArrDegreeOfVid = nullptr;
		int maxDegreeOfVer = 0;
		findMaxDegreeVid(dEmCol,dEmRMP,noElemdEmCol, noElemVid,\
			noElemRMP, noElemEmbedding, \
			dArrDegreeOfVid,maxDegreeOfVer);

		int noPossibleExt = noElemRMP * noElemEmbedding * maxDegreeOfVer;

		//5. Khai thác được các mở rộng
		//vecValid lưu kết quả tìm các mở rộng hợp lệ của kernel. Trong đó,
		//dArrValid <-- đánh dấu các mở rộng hợp lệ
		//dArrBackward <-- đánh dấu các mở rộng là backward
		//dArrEXT <-- thông tin chi tiết của mở rộng (vi,vj,li,lij,lj,vgi,vgj,posRow)
		vector<structValid> vecValid;
		vecValid.resize(noElemRMP);
		for (int i = 0; i < noElemRMP; i++)
		{
			vecValid[i].noElem = noElemEmbedding*maxDegreeOfVer;
			CUCHECK(hipMalloc((void**)&vecValid.at(i).dArrValid,noElemEmbedding*sizeof(int)*maxDegreeOfVer));
			CUCHECK(hipMalloc((void**)&vecValid.at(i).dArrEXT,noElemEmbedding*sizeof(EXT)*maxDegreeOfVer));
			CUCHECK(hipMemset(vecValid.at(i).dArrValid,0,noElemEmbedding*sizeof(int)*maxDegreeOfVer));
		}

		int** dPointerArrValid = nullptr;
		EXT** dPointerArrEXT = nullptr;

		CUCHECK(hipMalloc((void**)&dPointerArrValid,noElemRMP*sizeof(int**)));
		CUCHECK(hipMalloc((void**)&dPointerArrEXT,noElemRMP*sizeof(EXT**)));

		for (int i = 0; i < noElemRMP; i++)
		{
			kernelCopyDevice<<<1,1>>> (dPointerArrValid,vecValid.at(i).dArrValid, i);
			kernelCopyDeviceEXT<<<1,1>>>(dPointerArrEXT, vecValid.at(i).dArrEXT, i);
		}
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());
		dim3 block(blocksize);
		dim3 grid((noElemVid + block.x -1)/block.x);
		//displayDeviceArr(dEmRMP,noElemdEmCol);
		kernelFindValidExtension1<<<grid,block>>>( \
			dEmCol, \
			dEmRMP, \
			noElemdEmCol, \
			dRMP, \
			noElemRMP, \
			noElemEmbedding, \
			hdb.at(0).dO,hdb.at(0).dLO,hdb.at(0).dN,hdb.at(0).dLN, \
			dArrDegreeOfVid, \
			maxDegreeOfVer, \
			dPointerArrValid,dPointerArrEXT, \
			DFS_CODE.minLabel,DFS_CODE.maxId, \
			dValidBackward,dRMP);

		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//Hiển thị thông tin của vecValid
		//for (int i = 0; i < noElemRMP; i++)
		//{
		//	vecValid.at(i).show();
		//}
		//6. Trích được các mở rộng FW/BW từ vecValid lưu vào vecValidEXTk
		//Khởi tạo số lượng phần tử của vecValidEXTk bằng số lượng phần tử của vecValid
		vecValidEXTk.resize(vecValid.size());
		//Duyệt qua các vecValid và trích các mở rộng hợp lệ sang vecValidEXTk tương ứng.
		for (int idxVecValid = 0; idxVecValid < vecValid.size(); idxVecValid++)
		{
			//std::printf("\nMark the valid backward/forward extentions:");
			//vecValid.at(idxVecValid).show();
			vecValid.at(idxVecValid).extractValid(vecValidEXTk.at(idxVecValid));
			vecValid.at(idxVecValid).ReleaseMemory();
			//std::printf("\nShow the valid backward/forward extentions:");
			//vecValidEXTk.at(idxVecValid).show();
		}
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());
		//Chưa dùng dArrBackward của vecValid.
		vecValid.clear();
		//7. Release memory
		CUCHECK(hipFree(dArrEmbeddingColRMP));
		CUCHECK(hipFree(dArrDegreeOfVid));
		CUCHECK(hipFree(dEmRMP));
		CUCHECK(hipFree(dPointerArrValid));
		CUCHECK(hipFree(dPointerArrEXT));
		CUCHECK(hipFree(dValidBackward));
	} catch (std::exception &exc)
	{
		cout<<endl<<exc.what()<<endl;
		FCHECK(-1);
	}
}

//use
__global__ void kernelFillValidBackward(int* dValidBackward,int* dRMP,int noElem, int* dVjBackward,int noElemdVjBackward)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem)
	{
		int vj = dRMP[i];
		for (int k = 0; k < noElemdVjBackward; k++)
		{
			if(vj == dVjBackward[k])
			{
				dValidBackward[i] = 1;
				break;
			}
		}
	}
}

//use
__device__ void deviceGetVid(Embedding** &dEmCol, int* &dEmRMP,int &noElemdEmCol, \
						int &noElemEmbedding,int &idxCol, int &idxRow, int &noElemOnRMP, \
						int &vid,int &idxOnRMP)
{
	int idxLastdEmCol = noElemdEmCol - 1;
	int idxEmColVidOfThread = idxLastdEmCol;
	int idxEmRowVidOfThread = idxRow;
	
	idxOnRMP = noElemOnRMP;
	int countCol=-1;
	{
		do
		{
			if(dEmRMP[idxEmColVidOfThread] == 1)
			{
				--idxOnRMP;
				++countCol;
			}
			if(countCol == idxCol)
			{
				vid = dEmCol[idxEmColVidOfThread][idxEmRowVidOfThread].vid;
				return;
			}
			idxEmRowVidOfThread = dEmCol[idxEmColVidOfThread][idxEmRowVidOfThread].idx;
			--idxEmColVidOfThread;
		}while(idxEmRowVidOfThread != -1);
	}
}

//use
__device__ void deviceFindVid(int &thread , Embedding** &dEmCol, int* &dEmRMP,int &noElemdEmCol, \
						int &noElemEmbedding,int &noElemOnRMP, \
						 int &idxCol, int &idxRow,int& vid,int &idxOnRMP)
{
	//Get idxCol and idxRow of thread base on noElemEmbedding
	idxCol = thread / noElemEmbedding;
	idxRow = thread % noElemEmbedding;

	//initialize vid equal to minus one.
	vid = -1;
	deviceGetVid(dEmCol,dEmRMP,noElemdEmCol,noElemEmbedding,idxCol,idxRow, noElemOnRMP, \
		vid,idxOnRMP);

	//PMS_PRINT("\nthread:%d; idxCol:%d; idxRow:%d; vid:%d idxOnRMP:%d", \
	//	thread,idxCol,idxRow,vid, idxOnRMP);

}

//use
__device__ void deviceIsVidOnEm(int &toVid,Embedding** &dEmCol,int* &dEmRMP,int &noElemdEmCol, int &idxRow,int &noElemRMP, \
				int &onEm, int &onRMP,int &idxOnRMPtovid)
{
	int idxLastdEmCol = noElemdEmCol - 1;
	int idxEmCol = idxLastdEmCol;
	int idxEmRow = idxRow;

	idxOnRMPtovid = noElemRMP; //Dùng để lấy giá trị cho Vj
	do
	{
		if(dEmRMP[idxEmCol] == 1)
		{
			--idxOnRMPtovid;
		}
		if(toVid == dEmCol[idxEmCol][idxEmRow].vid)
		{
			onEm = 1;
			if (dEmRMP[idxEmCol] == 1) { onRMP = 1;}
			return;
		}
		idxEmRow = dEmCol[idxEmCol][idxEmRow].idx;
		--idxEmCol;
	}while(idxEmRow != -1);
}

//use
__global__ void kernelFindValidExtension1(Embedding **dEmCol,int* dEmRMP,int noElemdEmCol,int* dArrRMP, int noElemRMP, \
										 int noElemEmbedding, \
										 int *dO,int *dLO,int *dN,int *dLN, float *dArrDegreeOfVid, \
										 int maxDegreeOfVer,int** dPointerArrValid, \
										 EXT** dPointerArrEXT, int minLabel,int maxId, int* dValidBackward,int* dVj)
{
	//Mỗi Thread sẽ tìm mở rộng cho một vid trên RMP của embedding tương ứng.
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int noElemVid = noElemEmbedding*noElemRMP;
	if(i<noElemVid)
	{
		//get idxCol, idxRow and vid in which current thread needed processing.
		int idxLastCol = noElemRMP-1;
		int idxRow,idxCol,vid;
		int idxOnRMPvid; //biết vid mà thread đang xử lý thuộc RMP nào. Giúp lấy được Vj từ existbackard
		deviceFindVid(i,dEmCol, dEmRMP, noElemdEmCol,\
						noElemEmbedding,noElemRMP, \
						idxCol, idxRow, vid,idxOnRMPvid);

		int degreeOfVid = __float2int_rn(dArrDegreeOfVid[i]); //Lấy bậc của vid
		for (int idxToVid = 0; idxToVid < degreeOfVid; idxToVid++)
		{
			//int idxRMP = noElemRMP;
			int indexToVidIndN=dO[vid]+idxToVid; //Lấy index trong mảng nhãn cạnh.
			int labelFromVid = dLO[vid]; //Lấy nhãn của đỉnh được mở rộng.
			int toVid=dN[indexToVidIndN]; //vid của đỉnh kề.
			int labelToVid = dLO[toVid]; //nhãn của đỉnh kề
			
			int onEm = -1;
			int onRMP = -1;
			int idxOnRMPtovid = -1;
			deviceIsVidOnEm(toVid,dEmCol, dEmRMP, noElemdEmCol,idxRow,noElemRMP, \
				onEm, onRMP, idxOnRMPtovid);


			int idxColEXT,idxRowEXT;
			idxColEXT = (noElemRMP-1) - idxCol;
			//idxColEXT = idxCol;
			idxRowEXT = idxRow*maxDegreeOfVer + idxToVid;
			
			EXT* dArrEXT = dPointerArrEXT[idxColEXT];
			int* dArrValid = dPointerArrValid[idxColEXT];
			int b = -1;
			if(idxCol == 0) //Nếu vid mở rộng thuộc embedding column cuối
			{
				if(onRMP == 1) //Nếu tovid thuộc RMP
				{
					if(idxOnRMPvid - idxOnRMPtovid>=2) //Nếu khoảng cách giữa đỉnh mở rộng và đỉnh kề cách nhau ít nhất 2 EmCols.
					{
						if(dValidBackward[idxOnRMPtovid]==-1) //Nếu mở rộng backward đến tovid đó chưa tồn tại
						{
							//Backward được xem là hợp lệ
							dArrValid[idxRowEXT] = 1;

							dArrEXT[idxRowEXT].vi = maxId;
							dArrEXT[idxRowEXT].vj = dVj[idxOnRMPtovid];
							dArrEXT[idxRowEXT].li = labelFromVid;
							dArrEXT[idxRowEXT].lij = dLN[indexToVidIndN];
							dArrEXT[idxRowEXT].lj = labelToVid;
							dArrEXT[idxRowEXT].vgi = vid; 
							dArrEXT[idxRowEXT].vgj = toVid;
							dArrEXT[idxRowEXT].posRow = idxRow;
							b = 1;
						}
					}
				}
			}
			
			if (onEm==-1 && labelToVid>=minLabel)
			{
				//save valid forward
				dArrValid[idxRowEXT] = 1;
				dArrEXT[idxRowEXT].vi = dVj[idxOnRMPvid];
				dArrEXT[idxRowEXT].vj = maxId + 1;
				dArrEXT[idxRowEXT].li = labelFromVid;
				dArrEXT[idxRowEXT].lij = dLN[indexToVidIndN];
				dArrEXT[idxRowEXT].lj = labelToVid;
				dArrEXT[idxRowEXT].vgi = vid; 
				dArrEXT[idxRowEXT].vgj = toVid;
				dArrEXT[idxRowEXT].posRow = idxRow;
				b=2;
				//return;
			}
			//if (i ==8) std::printf("\n\nThread:%d idxToVid:%d toVid:%d \n labelToVid:%d degreeOfVid:%d \n onEm:%d onRMP:%d idxOnRMPvid:%d \n idxColEXT:%d idxRowEXT:%d minLabel:%d dArrEXT[idxRowEXT].vi:%d dArrValid[idxRowEXT]:%d \n dVj[0]:%d dVj[1]:%d dVj[2]:%d b:%d dArrEXT[idxRowEXT].vj:%d idxRow:%d", \
			//	i,idxToVid,toVid,labelToVid,degreeOfVid,onEm,onRMP,idxOnRMPvid,idxColEXT,idxRowEXT,minLabel,dArrEXT[idxRowEXT].vi,dArrValid[idxRowEXT],dVj[0],dVj[idxColEXT],dVj[2],b,dArrEXT[idxRowEXT].vj,idxRow);
		}



		//Dùng để so sánh với idxCol, nếu bằng nhau thì getVid
		//int countOnedEmRMP =-1;
		////idxRMP dùng để kiểm tra mở rộng backward có hợp lệ không.
		////int idxRMP = noElemRMP;
		////PMS_PRINT("\n Thread: %d idxCol:%d idxRow:%d idxLastCol:%d ",i,idxCol,idxRow,idxLastCol);
		//for(int s = noElemdEmCol-1,int idxRowTemp=idxRow; s>=0; s-- )
		//{
		//	Embedding* dEmTempCol = dEmCol[s];
		//	if(dEmRMP[s]==1)
		//	{
		//		countOnedEmRMP++;
		//		//idxRMP--;
		//	}
		//	if(countOnedEmRMP == idxCol)
		//	{
		//		//get div at current column s
		//		vid = dEmTempCol[idxRowTemp].vid;
		//		//then break
		//		break;
		//	}
		//	//Cập nhật idx row cần truy xuất trong column trước.
		//	idxRowTemp = dEmTempCol[idxRowTemp].idx;
		//}

		////Duyệt qua các đỉnh kề với vid trong dN dựa vào bậc của vid.
		//int degreeOfVid = __float2int_rn(dArrDegreeOfVid[i]); //Lấy bậc của vid
		//for (int idxToVid = 0; idxToVid < degreeOfVid; idxToVid++)
		//{
		//	int idxRMP = noElemRMP;
		//	int indexToVidIndN=dO[vid]+idxToVid; //Lấy index trong mảng nhãn cạnh.
		//	int labelFromVid = dLO[vid]; //Lấy nhãn của đỉnh được mở rộng.
		//	int toVid=dN[indexToVidIndN]; //vid của đỉnh kề.
		//	int labelToVid = dLO[toVid]; //nhãn của đỉnh kề

		//	//Xét đỉnh kề có thoả các điều kiện của mở rộng forward hay không.
		//	//đk1: nếu nhãn đỉnh kề nhỏ hơn minLabel của DFS_CODE thì continue xét đỉnh kề tiếp theo.
		//	if(labelToVid<minLabel) continue;
		//	//đk2: nếu đỉnh kề đã thuộc RMP của embedding rồi thì xét xem nó có là backward hay không.
		//	//Nếu đỉnh kề tồn tại trong lstVidOnRMP thì xem như nó đã thuộc embedding, tiếp tục xét đỉnh kề khác
		//	bool isExist = false;
		//	bool onRMP = false;
		//	for(int s = noElemdEmCol-1,int idxRowTemp=idxRow; s>=0; s-- )
		//	{
		//		Embedding* dEmTempCol = dEmCol[s];
		//		if(dEmRMP[s]==1)
		//		{
		//			idxRMP--;
		//		}
		//		if(toVid == dEmTempCol[idxRowTemp].vid)
		//		{
		//			isExist = true;
		//			//idxRMP = s;
		//			if (dEmRMP[s] == 1) onRMP = true;
		//			PMS_PRINT("\n Thread: %d isExist:%d onRMP:%d\n",i,isExist,onRMP);
		//			break;
		//		}
		//		idxRowTemp = dEmTempCol[idxRowTemp].idx;
		//	}
		//	//PMS_PRINT("\n Thread: %d ,vid: %d toVid:%d isExist:%d \n",i,vid,toVid,isExist);
		//	//vid có tối đa là maxDegreeOfVer mở rộng hợp lệ được lưu trữ trong dPointerArrEXT tương ứng tại idxCol, idxRow.
		//	EXT* dArrEXT = dPointerArrEXT[idxCol];
		//	int* dArrValid = dPointerArrValid[idxCol];
		//	int idxRowEXT = idxRow*maxDegreeOfVer + idxToVid;
		//	if (isExist == true) 
		//	{
		//		//Nếu Thread đang xử lý cho vid thuộc Embedding Column cuối và 
		//		//nhiều hơn 2 đỉnh thuộc RMP thì mới xét mở rộng backard và
		//		//Đỉnh kề phải cách đỉnh cuối ít nhất 1 đỉnh.
		//		if (onRMP == true && (noElemRMP - idxRMP)>=3 && noElemRMP >2) 
		//		{
		//			PMS_PRINT("\nThread:%d YES backward noElemRMP:%d idxRMP:%d\n",i,noElemRMP,idxRMP);
		//			goto considerBackward;
		//		}
		//		continue;
		//	};
		//	//Lưu mở rộng hợp lệ forward
		//	//Lưu mở rộng forward hợp lệ
		//	dArrValid[idxRowEXT] = 1;
		//	//dArrEXT[idxRowEXT].vi = dArrRMP[idxCol]; //<<<<<<<< chú ý lại
		//	dArrEXT[idxRowEXT].vi = dVj[idxCol];
		//	dArrEXT[idxRowEXT].vj = maxId + 1;
		//	dArrEXT[idxRowEXT].li = labelFromVid;
		//	dArrEXT[idxRowEXT].lij = dLN[indexToVidIndN];
		//	dArrEXT[idxRowEXT].lj = labelToVid;
		//	dArrEXT[idxRowEXT].vgi = vid; 
		//	dArrEXT[idxRowEXT].vgj = toVid;
		//	dArrEXT[idxRowEXT].posRow = idxRow;
		//	/*PMS_PRINT("\n Thread: %d, vid:%d,idxCol:%d, dVj: %d, maxId: %d (%d,%d,%d,%d,%d,vgi:%d,vgj:%d,posRow:%d)  \
		//			  idxRowEXT:%d dArrValid[idxRowEXT]:%d", \
		//		i,vid,idxCol, dVj[idxCol],maxId,dArrEXT[idxRowEXT].vi,dArrEXT[idxRowEXT].vj, \
		//		dArrEXT[idxRowEXT].li,dArrEXT[idxRowEXT].lij,dArrEXT[idxRowEXT].lj, \
		//		dArrEXT[idxRowEXT].vgi,dArrEXT[idxRowEXT].vgj,dArrEXT[idxRowEXT].posRow, \
		//		idxRowEXT, dArrValid[idxRowEXT]);*/
		//	//lưu xong thì continue xét đỉnh kề khác
		//	continue;

		//	considerBackward:
		//	//Kiểm tra backward có hợp lệ hay không
		//	//Nếu backward đã tồn tại rồi thì continue xét đỉnh kề khác.
		//	if(1==dValidBackward[idxRMP]) continue;
		//	//Lưu lại mở rộng backward hợp lệ.
		//	dArrValid[idxRowEXT] = 1;
		//	dArrEXT[idxRowEXT].vi = maxId;
		//	dArrEXT[idxRowEXT].vj = dVj[idxRMP];
		//	dArrEXT[idxRowEXT].li = labelFromVid;
		//	dArrEXT[idxRowEXT].lij = dLN[indexToVidIndN];
		//	dArrEXT[idxRowEXT].lj = labelToVid;
		//	dArrEXT[idxRowEXT].vgi = vid; 
		//	dArrEXT[idxRowEXT].vgj = toVid;
		//	dArrEXT[idxRowEXT].posRow = idxRow;
		//	/*PMS_PRINT("\n Thread: %d, vid:%d,idxCol:%d, dVj: %d, maxId: %d (%d,%d,%d,%d,%d,vgi:%d,vgj:%d,posRow:%d)  \
		//			  idxRowEXT:%d dArrValid[idxRowEXT]:%d", \
		//		i,vid,idxCol, dVj[idxRMP],maxId,dArrEXT[idxRowEXT].vi,dArrEXT[idxRowEXT].vj, \
		//		dArrEXT[idxRowEXT].li,dArrEXT[idxRowEXT].lij,dArrEXT[idxRowEXT].lj, \
		//		dArrEXT[idxRowEXT].vgi,dArrEXT[idxRowEXT].vgj,dArrEXT[idxRowEXT].posRow, \
		//		idxRowEXT, dArrValid[idxRowEXT]);*/
		//}
	}
}


//use
__global__ void kernelCreatedEmRMP(int* dArrRMP,int* dEmRMP,int noElemRMP)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElemRMP)
	{
		int idx = dArrRMP[i];
		dEmRMP[idx] = 1;
	}
}

//use
__global__ void	kernelGetPointerdArrEmbedding(Embedding *dArrEmbedding,Embedding **dArrPointerEmbedding,int idx)
{
	//Copy the address of dArrEmbedding into dArrPointerEmbedding
	dArrPointerEmbedding[idx]=dArrEmbedding;
	//PMS_PRINT("\n PointerdArrEmbedding:%p, PointerdArrPointerEmbedding:%p",dArrEmbedding,dArrPointerEmbedding[idx]);
}

//use
__global__ void kernelExtractValidExtensionTodExt(EXT *dArrExtension,int *dArrValid,int *dArrValidScanResult, \
												  int noElem_dArrV,EXT *dExt,int noElem_dExt)
{
	//kernel trích các mở rộng hợp lệ từ mảng dArrExtension sang mảng dExt
	int i =blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dArrV)
	{
		if(dArrValid[i]==1)
		{
			dExt[dArrValidScanResult[i]].vi = dArrExtension[i].vi;
			dExt[dArrValidScanResult[i]].vj = dArrExtension[i].vj;
			dExt[dArrValidScanResult[i]].li = dArrExtension[i].li;
			dExt[dArrValidScanResult[i]].lij = dArrExtension[i].lij;
			dExt[dArrValidScanResult[i]].lj = dArrExtension[i].lj;
			dExt[dArrValidScanResult[i]].vgi = dArrExtension[i].vgi;
			dExt[dArrValidScanResult[i]].vgj = dArrExtension[i].vgj;
			dExt[dArrValidScanResult[i]].posRow = dArrExtension[i].posRow;
		}
	}
}

//use
__global__ void kernelGet_vivjlj(EXT* dArrExt,int* dvi,int* dvj,int* dli,int maxId)
{
	*dvi = dArrExt[0].vi;
	*dvj = maxId+1;
	*dli = dArrExt[0].li;
}

//use
__global__ void kernelExtractUniBE(int* dAllExtension,int noElemdAllExtension, \
									int* dRMP,int* dRMPLabel,int Lv,UniEdge* dUniEdge, \
									int* dAllExtensionIdx,int vi,int li)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElemdAllExtension)
	{
		if(dAllExtension[i] == 1)
		{
			int lij = i / Lv;
			int idxVj = i % Lv;
			int vj = dRMP[idxVj];
			int idxUniEdge = dAllExtensionIdx[i];
			dUniEdge[idxUniEdge].lij = lij;
			dUniEdge[idxUniEdge].vj = vj;
			dUniEdge[idxUniEdge].lj = dRMPLabel[idxVj];
			dUniEdge[idxUniEdge].vi = vi;
			dUniEdge[idxUniEdge].li = li;
		}
	}
}

//use
__global__ void kernelMarkUniBE(int* dMappingVj,int* dAllExtension, int Lv,int noElem,EXT* dArrEXT)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElem)
	{
		int vi = dArrEXT[i].vi;
		int vj = dArrEXT[i].vj;
		if (vi > vj)
		{
			int lij = dArrEXT[i].lij;
			int idxVj = dMappingVj[vj];
			int idxAllExtension = lij*Lv + idxVj;
			dAllExtension[idxAllExtension] = 1;
			//PMS_PRINT("\n Thread: %d, vi:%d vj:%d, lij:%d lv:%d  idxVj:%d idxAllExtension:%d)", \
			//	i,vi,vj,lij,Lv,idxVj,idxAllExtension);
		}
	}
}

//use
__global__ void kernelFilldMappingVj(int noElemBW,int* dMappingVj,int* dRMP)
{
	int i = blockDim.x*blockIdx.x +threadIdx.x;
	if(i<noElemBW)
	{
		int vj = dRMP[i];
		dMappingVj[vj] = i;
	}
}

//use
__global__ void kernelFillUniFE( int *dArrAllPossibleExtension, \
								int *dArrAllPossibleExtensionScanResult, \
								int noElem_dArrAllPossibleExtension, \
								UniEdge *dArrUniEdge, \
								int Lv,int *dvi, \
								int *dvj,int *dli)
{
	//Kernel fill unique forward extension
	int i = blockDim.x*blockIdx.x +threadIdx.x;
	if(i<noElem_dArrAllPossibleExtension)
	{
		if(dArrAllPossibleExtension[i]==1)
		{
			int li,lij,lj;
			li=*dli;
			lij = i/Lv;
			lj=i%Lv;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].vi=*dvi;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].vj=*dvj;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].li=li;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].lij=lij;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].lj=lj;
		}
	}
}



//use
__global__ void kernelmarkValidForwardEdge_LastExt(EXT* dArrExt, int noElemdArrExt,unsigned int Lv, \
												   int *dAllPossibleExtension)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElemdArrExt)
	{
		//Chỉ xét các forward
		if(dArrExt[i].vi < dArrExt[i].vj)
		{
			int index=	dArrExt[i].lij*Lv + dArrExt[i].lj;
			dAllPossibleExtension[index]=1;
		}
	}
}

//use
__global__ void kernelFilldF(UniEdge *dArrUniEdge,int pos,EXT *dArrExt,int noElemdArrExt,int *dArrBoundaryScanResult,int *dF)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElemdArrExt)
	{
		int vi = dArrUniEdge[pos].vi;
		int vj = dArrUniEdge[pos].vj;
		int li = dArrUniEdge[pos].li;
		int lij = dArrUniEdge[pos].lij;
		int lj = dArrUniEdge[pos].lj;
		int Li = dArrExt[i].li;
		int Lij = dArrExt[i].lij;
		int Lj = dArrExt[i].lj;
		int Vi = dArrExt[i].vi;
		int Vj = dArrExt[i].vj;
		if(li==Li && lij==Lij && lj==Lj && vi == Vi && vj == Vj)
		{
			dF[dArrBoundaryScanResult[i]]=1;
		}
		//PMS_PRINT("\nThread %d: UniEdge(li:%d lij:%d lj:%d) (Li:%d Lij:%d Lj:%d idxdF:%d dF:%d)",i,li,lij,lj,Li,Lij,Lj,dArrBoundaryScanResult[i],dF[dArrBoundaryScanResult[i]]);
	}
}

//use
__global__ void kernelfindBoundary(EXT *dArrExt,int noElemdArrExt,int *dArrBoundary,unsigned int maxOfVer)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElemdArrExt-1)
	{
		unsigned int graphIdAfter=dArrExt[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=dArrExt[i].vgi/maxOfVer;
		if(graphIdAfter!=graphIdCurrent)
		{
			dArrBoundary[i]=1;
		}
	}
}

//use
__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cache[256];


	float temp = -1.0;
	while(index + offset < n){
		temp = fmaxf(temp, array[index + offset]);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}


//use
void PMS::findMaxDegreeVid(Embedding** &dEmCol,int* &dEmRMP,int &noElemdEmCol, int &noElemVid, \
			int &noElemRMP, int &noElemEmbedding, \
			float* &dArrDegreeOfVid,int &maxDegreeOfVer)
{
	try
	{
		if(noElemVid<=0) return;
		CUCHECK(hipMalloc((void**)&dArrDegreeOfVid,noElemVid*sizeof(float)));

		dim3 block(blocksize);
		dim3 grid((noElemVid + block.x -1)/block.x);

		kernelCalDegreeOfVidOnEmbeddingColumnv2<<<grid,block>>>( \
			dEmCol,dEmRMP,noElemdEmCol, \
			hdb.at(0).dO,hdb.at(0).noElemdO,noElemRMP, noElemEmbedding, \
			hdb.at(0).noElemdN, maxOfVer,dArrDegreeOfVid);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//displayDeviceArr(dArrDegreeOfVid,noElemVid);
		float* h_max = (float*)malloc(sizeof(float));
		if(h_max==nullptr) FCHECK(-1);

		float *d_max = nullptr;
		int *d_mutex = nullptr;
		CUCHECK(hipMalloc((void**)&d_max,sizeof(float)));
		CUCHECK(hipMemset(d_max,0,sizeof(float)));

		CUCHECK(hipMalloc((void**)&d_mutex,sizeof(int)));
		CUCHECK(hipMemset(d_mutex,0,sizeof(int)));

		dim3 gridSize = 256;
		dim3 blockSize = 256;
		find_maximum_kernel<<<gridSize, blockSize>>>(dArrDegreeOfVid, d_max, d_mutex, noElemVid);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		CUCHECK(hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost));
		maxDegreeOfVer = (int)*h_max;

		free(h_max);
		CUCHECK(hipFree(d_max));
		CUCHECK(hipFree(d_mutex));
	}
	catch(const std::exception &exc)
	{
		std::cerr<<exc.what();
		FCHECK(-1);
	}
}


//use
__global__ void kernelCalDegreeOfVidOnEmbeddingColumnv2(Embedding** dEmCol,int* dEmRMP,int noElemdEmCol, \
									 int *d_O, int numberOfElementd_O,int noElemRMP, int noElemEmbedding, \
									 int numberOfElementd_N,unsigned int maxOfVer,float *dArrDegreeOfVid)
{
	//Mỗi Thread sẽ tìm mở rộng cho một vid trên RMP của embedding tương ứng.
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int noElemVid = noElemEmbedding*noElemRMP;
	if(i<noElemVid)
	{
		int idxLastCol = noElemRMP-1;
		int idxRow = i % noElemEmbedding;
		int idxCol = i / noElemEmbedding;
		//vid mà thread i cần xử lý
		int vid=-1;
		//Dùng để so sánh với idxCol, nếu bằng nhau thì getVid
		int countOnedEmRMP =-1;
		for(int s = noElemdEmCol-1,int idxRowTemp=idxRow; s>=0; s-- )
		{
			Embedding* dEmTempCol = dEmCol[s];
			if(dEmRMP[s]==1)
			{
				countOnedEmRMP++;
			}
			if(countOnedEmRMP == idxCol) 
			{
				//get div at current column s
				vid = dEmTempCol[idxRowTemp].vid;
				break;
			}
			//Cập nhật idx row cần truy xuất trong column trước.
			idxRowTemp = dEmTempCol[idxRowTemp].idx;
		}
		//PMS_PRINT("\nThread %d proccess vid: %d",i,vid);
		float degreeOfV =0;
		int nextVid=-1;
		int graphid=-1;
		int lastGraphId=(numberOfElementd_O-1)/maxOfVer;
		if (vid==numberOfElementd_O-1)
		{ 
			//nếu như đây là đỉnh cuối cùng trong d_O
			//thì bậc của đỉnh vid chính bằng tổng số cạnh trừ cho giá trị của d_O[vid].
			degreeOfV=numberOfElementd_N-d_O[vid]; 
		}
		else
		{
			nextVid = vid+1; //xét đỉnh phía sau có khác 1 hay không?
			graphid=vid/maxOfVer;
			if(d_O[nextVid]==-1 && graphid==lastGraphId)
			{
				degreeOfV=numberOfElementd_N-d_O[vid];
			}
			else if(d_O[nextVid]==-1 && graphid!=lastGraphId)
			{
				nextVid=(graphid+1)*maxOfVer;
				degreeOfV=d_O[nextVid]-d_O[vid];
			}
			else
			{
				degreeOfV=d_O[nextVid]-d_O[vid];
			}
		}
		dArrDegreeOfVid[i]=degreeOfV;
	}
}


//use
void EXTk::mark_edge(int vi,int vj,int li,int lij,int lj,int *&dValid)
{
	CUCHECK(hipMalloc((void**)&dValid,sizeof(int)*noElem));
	CUCHECK(hipMemset(dValid,0,sizeof(int)*noElem));

	dim3 block(blocksize);
	dim3 grid((noElem+block.x-1)/block.x);
	kernel_mark_edge<<<grid,block>>>(vi,vj,li,lij,lj,dArrExt,dValid,noElem);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}

//use
void structValid::extractValid(EXTk &outputEXT)
{
	//doing somethings here
	//1. Scan on dArrValid to get index
	int *dArrValidScanResult=nullptr;
	CUCHECK(hipMalloc((void**)&dArrValidScanResult,sizeof(int)*noElem));
	CUCHECK(hipMemset(dArrValidScanResult,0,sizeof(int)*noElem));

	myScanV(dArrValid,noElem,dArrValidScanResult);
	int noElem_dExt=0;
	get_noElem_valid(dArrValid,dArrValidScanResult,noElem,noElem_dExt);
	if (noElem_dExt == 0) 
	{
		CUCHECK(hipFree(dArrValidScanResult));
		return;
	}
	outputEXT.noElem = noElem_dExt;

	CUCHECK(hipMalloc((void**)&outputEXT.dArrExt,sizeof(EXT)*outputEXT.noElem));
	dim3 block(blocksize);
	dim3 grid((noElem+block.x -1)/block.x);
	kernelExtractValidExtensionTodExt<<<grid,block>>>(dArrEXT,dArrValid,dArrValidScanResult,noElem,outputEXT.dArrExt,noElem_dExt);
	CUCHECK(hipFree(dArrValidScanResult));
}

//use
void EXTk::extractUniForwardExtension(unsigned int& Lv,unsigned int& Le,int& maxId)
{
	//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
	int noElem_dallPossibleExtension=Le*Lv;

	int *d_allPossibleExtension=nullptr;
	int *d_allPossibleExtensionScanResult=nullptr;

	CUCHECK(hipMalloc((void**)&d_allPossibleExtension,noElem_dallPossibleExtension*sizeof(int)));
	CUCHECK(hipMemset(d_allPossibleExtension,0,noElem_dallPossibleExtension*sizeof(int)));
	CUCHECK(hipMalloc((void**)&d_allPossibleExtensionScanResult,noElem_dallPossibleExtension*sizeof(int)));

	dim3 block(blocksize);
	dim3 grid((noElem+block.x-1)/block.x);
	//Đánh dấu vị trí các mở rộng forward hợp lệ là 1 tại vị trí d_allPossibleExtension tương ứng
	kernelmarkValidForwardEdge_LastExt<<<grid,block>>>(dArrExt,noElem,Lv,d_allPossibleExtension);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	get_idx(d_allPossibleExtension,noElem_dallPossibleExtension,d_allPossibleExtensionScanResult);

	int noElem_UniEdge=0;
	get_noElem_valid(d_allPossibleExtension,d_allPossibleExtensionScanResult, \
		noElem_dallPossibleExtension,noElem_UniEdge);

	if(noElem_UniEdge!=0)
	{
		uniFE.noElem = noElem_UniEdge;
		CUCHECK(hipMalloc((void**)&uniFE.dUniEdge,uniFE.noElem*sizeof(UniEdge)));
		int* dvi = nullptr;
		int* dvj = nullptr;
		int* dli = nullptr;
		CUCHECK(hipMalloc((void**)&dvi,sizeof(int)*1));
		CUCHECK(hipMalloc((void**)&dvj,sizeof(int)*1));
		CUCHECK(hipMalloc((void**)&dli,sizeof(int)*1));
		
		kernelGet_vivjlj<<<1,1>>>(dArrExt,dvi,dvj,dli,maxId);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//displayDeviceArr(dvi,1);
		//displayDeviceArr(dvj,1);
		//displayDeviceArr(dli,1);

		dim3 block1(blocksize);
		dim3 grid1((noElem_dallPossibleExtension + block1.x -1)/block1.x);

		kernelFillUniFE<<<grid1,block1>>>( d_allPossibleExtension, \
								d_allPossibleExtensionScanResult, \
								noElem_dallPossibleExtension, \
								uniFE.dUniEdge, \
								Lv, dvi, dvj, dli);

		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//uniFE.show();
	}

	if (d_allPossibleExtension != nullptr) CUCHECK(hipFree(d_allPossibleExtension));
	if (d_allPossibleExtensionScanResult != nullptr) CUCHECK(hipFree(d_allPossibleExtensionScanResult));
	return;
}

//use
void EXTk::extractUniBackwardExtension(unsigned int& Lv,unsigned int& Le,int& noElemRMP, \
									   int*& dRMP,int*& dRMPLabel, int& noElemMappingVj,int& vi,int& li)
{
	try
	{
		int noElemdAllExtension = Le * (noElemRMP-2);
		int tempLv = noElemRMP-2;
		//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
		int noElemBW = noElemRMP -2;
		//cout<<endl<<"dRMP: "<<noElemBW<<endl;
		//displayDeviceArr(dRMP,noElemBW);
		//cout<<endl<<"dRMPLabel: "<<noElemBW<<endl;
		//displayDeviceArr(dRMPLabel,noElemBW);
		//Chứa kết quả đánh dấu các mở rộng backward có thể có.
		int *dAllExtension=nullptr;
		int *dAllExtensionIdx=nullptr;
		int *dMappingVj = nullptr;

		CUCHECK(hipMalloc((void**)&dAllExtension,noElemdAllExtension*sizeof(int)));
		CUCHECK(hipMemset(dAllExtension,0,noElemdAllExtension*sizeof(int)));
		CUCHECK(hipMalloc((void**)&dAllExtensionIdx,noElemdAllExtension*sizeof(int)));
		CUCHECK(hipMalloc((void**)&dMappingVj, noElemMappingVj * sizeof(int)));
		CUCHECK(hipMemset(dMappingVj,-1, noElemMappingVj * sizeof(int)));
		//Xây dựng dMappingVj để ánh xạ vj trong EXT sang idxVj
		dim3 block(blocksize);
		dim3 grid((noElemBW + block.x - 1)/block.x);
		//Mỗi thread sẽ đọc 1 phần tử (lij,vj) từ EXT
		//Đọc mảng dRMP[vj]
		kernelFilldMappingVj<<<grid,block>>>(noElemBW,dMappingVj,dRMP);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());
		//cout<<endl<<"dMappingVj:"<<endl;
		//displayDeviceArr(dMappingVj,noElemMappingVj);
		//Bật 1 cho các unique backward extension trong dAllExtension
		dim3 block1(blocksize);
		dim3 grid1((noElem + block.x - 1)/block.x);

		kernelMarkUniBE<<<grid1,block1>>>(dMappingVj,dAllExtension,tempLv,noElem,dArrExt);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//cout<<endl<<"dAllExtension:"<<endl;
		//displayDeviceArr(dAllExtension,noElemdAllExtension);

		//Scan on dAllExtension to get index, noElem and cudamalloc
		get_idx(dAllExtension,noElemdAllExtension,dAllExtensionIdx);
		get_noElem_valid(dAllExtension,dAllExtensionIdx,noElemdAllExtension,uniBE.noElem);
		CUCHECK(hipMalloc((void**)&uniBE.dUniEdge,sizeof(UniEdge)*uniBE.noElem));
		//Check again
		//Ánh xạ ngược từ dAllExtension sang UniEdge Backward
		dim3 block2(blocksize);
		dim3 grid2((noElemdAllExtension + block2.x -1)/block2.x);
		kernelExtractUniBE<<<grid2,block2>>>(dAllExtension,noElemdAllExtension, \
			dRMP,dRMPLabel,tempLv,uniBE.dUniEdge,dAllExtensionIdx,vi,li);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//uniBE.show();


		if (dAllExtension != nullptr) CUCHECK(hipFree(dAllExtension));
		if (dAllExtensionIdx != nullptr) CUCHECK(hipFree(dAllExtensionIdx));
		if (dMappingVj != nullptr) CUCHECK(hipFree(dMappingVj));
		if (dRMPLabel != nullptr) CUCHECK(hipFree(dRMPLabel));
		if (dRMP != nullptr) CUCHECK(hipFree(dRMP));
		return;
	}
	catch(std::exception &exc)
	{
		cout<<endl<<exc.what()<<endl;
		FCHECK(-1);
	}
}

//use
void arrUniEdge::copyDTH()
{
	try
	{
		if(noElem < 0) return;
		hUniEdge = nullptr;
		hUniEdge = (UniEdge*)malloc(sizeof(UniEdge)*noElem);
		if(hUniEdge == nullptr) {FCHECK(-1);}
		CUCHECK(hipMemcpy(hUniEdge,dUniEdge,sizeof(UniEdge)*noElem,hipMemcpyDeviceToHost));
	}
	catch (...)
	{
		FCHECK(-1)
	}
}

//use
void EXTk::findBoundary(unsigned int& maxOfVer,int*& dArrBoundaryScanResult)
{
	int *dArrBoundary=nullptr; 
	CUCHECK(hipMalloc((void**)&dArrBoundary,sizeof(int)*noElem));
	CUCHECK(hipMemset(dArrBoundary,0,sizeof(int)*noElem));

	dArrBoundaryScanResult=nullptr;
	CUCHECK(hipMalloc((void**)&dArrBoundaryScanResult,sizeof(int)*noElem));
	CUCHECK(hipMemset(dArrBoundaryScanResult,0,sizeof(int)*noElem));

	dim3 block(blocksize);
	dim3 grid((noElem+block.x-1)/block.x);
	

	kernelfindBoundary<<<grid,block>>>(dArrExt,noElem,dArrBoundary,maxOfVer);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());

	get_idx(dArrBoundary,noElem,dArrBoundaryScanResult);

	CUCHECK(hipFree(dArrBoundary));
}

//use
void EXTk::findSupport(unsigned int& maxOfVer)
{
	try
	{
		int* dArrBoundaryIndex = nullptr;
		findBoundary(maxOfVer,dArrBoundaryIndex);

		int *dF=nullptr;
		int noElemdF = 0;
		CUCHECK(hipMemcpy(&noElemdF,&dArrBoundaryIndex[noElem-1],sizeof(int),hipMemcpyDeviceToHost));
		++noElemdF;

		CUCHECK(hipMalloc((void**)&dF,sizeof(int)*noElemdF));
		CUCHECK(hipMemset(dF,0,sizeof(int)*noElemdF));

		//Tính Support cho các mở rộng backward
		if(uniBE.noElem>0)
		{
			uniBE.copyDTH();
			uniBE.hSupport = nullptr;
			uniBE.hSupport = (int*)malloc(sizeof(int)*uniBE.noElem);
			if(uniBE.hSupport==nullptr) FCHECK(-1);
			memset(uniBE.hSupport,0,sizeof(int)*uniBE.noElem);
			for (int i = 0; i < uniBE.noElem; i++)
			{
				findSupportFW(dArrBoundaryIndex,uniBE.dUniEdge,i,dF,noElemdF,uniBE.hSupport[i]);
				//Mỗi lần lặp thì reset lại zerocho dF 
				CUCHECK(hipMemset(dF,0,sizeof(int)*noElemdF));
			}
			//uniBE.showSupport();
		}

		//Tính Support cho các mở rộng forward
		if(uniFE.noElem>0)
		{
			uniFE.copyDTH();
			uniFE.hSupport = nullptr;
			uniFE.hSupport = (int*)malloc(sizeof(int)*uniFE.noElem);
			if(uniFE.hSupport==nullptr) FCHECK(-1);
			memset(uniFE.hSupport,0,sizeof(int)*uniFE.noElem);
			//Duyệt qua các phần tử duy nhất, tính support của chúng và lưu lại trong hSupport tại index tương ứng.
			for (int i = 0; i < uniFE.noElem; i++)
			{
				findSupportFW(dArrBoundaryIndex,uniFE.dUniEdge,i,dF,noElemdF,uniFE.hSupport[i]);
				//Mỗi lần lặp thì reset lại zerocho dF 
				CUCHECK(hipMemset(dF,0,sizeof(int)*noElemdF));
			}
			//uniFE.showSupport();
		}

		CUCHECK(hipFree(dArrBoundaryIndex));
		CUCHECK(hipFree(dF));
	}
	catch(...)
	{
		FCHECK(-1);
	}
}

//use
void EXTk::findSupportFW(int*& dArrBoundaryIndex,UniEdge*& dArrUniEdge,int& idxUniEdge, int*& dF,int& noElemdF,int& support)
{
	try
	{
		int hSupport=0;
		dim3 block(blocksize);
		dim3 grid((noElem + block.x - 1)/block.x);
		kernelFilldF<<<grid,block>>>(dArrUniEdge,idxUniEdge,dArrExt,noElem,dArrBoundaryIndex,dF);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//PMS_PRINT("\n**********dF****************\n");
		//FCHECK(displayDeviceArr(dF,noElemdF));

		myReduce(dF,noElemdF,hSupport);


		//PMS_PRINT("\n******support********");
		//PMS_PRINT("\n Support:%d",hSupport);

		support=hSupport;
	}
	catch(...)
	{
		FCHECK(-1);
	}
}

//use
void EXTk::extractStatisfyMinsup(unsigned int& minsup,arrUniEdge& uniEdge,UniEdgeStatisfyMinSup& uniES)
{
	try
	{
		//Trích các mở rộng forward thoả minsup
		if (uniEdge.noElem<=0) return;
		//1. Cấp phát mảng trên device có kích thước bằng noElemUniEdge
		int *dResultSup=nullptr; //cần được giải phóng ở cuối hàm
		CUCHECK(hipMalloc((void**)&dResultSup,uniEdge.noElem*sizeof(int)));
		//Chép độ hỗ trợ từ host qua device để lọc song song 
		CUCHECK(hipMemcpy(dResultSup,uniEdge.hSupport,uniEdge.noElem*sizeof(int),hipMemcpyHostToDevice));

		//2. Đánh dấu 1 trên dV cho những phần tử thoả minsup
		int *dV=nullptr; //cần được giải phóng ở cuối hàm
		CUCHECK(hipMalloc((void**)&dV,uniEdge.noElem*sizeof(int)));
		CUCHECK(hipMemset(dV,0,sizeof(int)*uniEdge.noElem));

		dim3 block(blocksize);
		dim3 grid((uniEdge.noElem + block.x - 1)/block.x);
		kernelMarkUniEdgeSatisfyMinsup<<<grid,block>>>(dResultSup,uniEdge.noElem,dV,minsup);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		int *dVScanResult=nullptr; //cần được giải phóng ở cuối hàm
		CUCHECK(hipMalloc((void**)&dVScanResult,uniEdge.noElem*sizeof(int)));

		get_idx(dV,uniEdge.noElem,dVScanResult);

		int noElemUniEdgeSatisfyMinSup = 0;
		get_noElem_valid(dV,dVScanResult,uniEdge.noElem,noElemUniEdgeSatisfyMinSup);
		//Nếu không có phần tử nào thoả minsup thì không khai thác nữa
		if(noElemUniEdgeSatisfyMinSup==0)
		{ 
			CUCHECK(hipFree(dResultSup));
			CUCHECK(hipFree(dV));
			CUCHECK(hipFree(dVScanResult));
			return;
		}

		uniES.noElem = noElemUniEdgeSatisfyMinSup;
		CUCHECK(hipMalloc((void**)&uniES.dArrUniEdge,uniES.noElem*sizeof(UniEdge)));

		uniES.hArrSupport = (int*)malloc(sizeof(int)*uniES.noElem);
		if (uniES.hArrSupport ==nullptr){FCHECK(-1);}

		int *dSup=nullptr; //cần được giải phóng ở cuối hàm
		CUCHECK(hipMalloc((void**)&dSup,uniES.noElem*sizeof(int)));

		dim3 blocka(blocksize);
		dim3 grida((uniEdge.noElem + blocka.x -1)/blocka.x);
		kernelExtractUniEdgeSatifyMinsupV3<<<grida,blocka>>> ( \
			uniEdge.dUniEdge,dV,dVScanResult,uniEdge.noElem,uniES.dArrUniEdge,dSup,dResultSup);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//PMS_PRINT("\n ********hUniEdgeSatisfyMinsup.dSup****************\n");
		//displayDeviceArr(dSup,uniES.noElem);

		CUCHECK(hipMemcpy(uniES.hArrSupport,dSup,sizeof(int)*uniES.noElem,hipMemcpyDeviceToHost));

		//uniES.show();
		CUCHECK(hipFree(dResultSup));
		CUCHECK(hipFree(dV));
		CUCHECK(hipFree(dVScanResult));
		CUCHECK(hipFree(dSup));
		uniEdge.ReleaseMemory();
	}
	catch(std::exception &exc)
	{
		cout<<endl<<exc.what()<<endl;
		FCHECK(-1);
	}
}

//use
void PMS::buildRMPLabel(int* &dRMP, int* &dRMPLabel,int &noElemMappingVj,int &_vi,int &_li)
{
	try
	{
		vector<int> RMP;
		vector<int> vertexLabel;
		int vi,vj;
		int preVj;
		int idxContinue=0;
		for(int i = DFS_CODE.size() - 1; i>=0;i--)
		{
			vi = DFS_CODE.at(i).from;
			vj = DFS_CODE.at(i).to;
			bool isForward = (vi<vj);
			if(isForward==true)
			{
				_vi = vj;
				_li = DFS_CODE.at(i).tolabel;
				idxContinue = i;
				preVj = vj;
				break;
			}
		}
		int i;
		for (i = idxContinue; i >= 0; i--)
		{
			vi = DFS_CODE.at(i).from;
			vj = DFS_CODE.at(i).to;
			if(vi<vj && preVj==vj)
			{
				RMP.push_back(vj);
				vertexLabel.push_back(DFS_CODE.at(i).tolabel);
				preVj = vi;
			}
		}
		RMP.push_back(vi);
		vertexLabel.push_back(DFS_CODE.at(++i).fromlabel);

		std::reverse(RMP.begin(),RMP.end());
		std::reverse(vertexLabel.begin(),vertexLabel.end());

		DFS_CODE.noElemOnRMP = RMP.size();
		int *hRMPLabel = nullptr;
		int *hRMP = nullptr;

		hRMP = (int*)malloc(sizeof(int)*(DFS_CODE.noElemOnRMP-2));
		if(hRMP == nullptr) {FCHECK(-1);}
		hRMPLabel = (int*)malloc(sizeof(int)*(DFS_CODE.noElemOnRMP-2));
		if(hRMPLabel == nullptr) {FCHECK(-1);}

		dRMPLabel = nullptr;
		dRMP = nullptr;
		CUCHECK(hipMalloc((void**)&dRMPLabel,sizeof(int)*(DFS_CODE.noElemOnRMP-2)));
		CUCHECK(hipMalloc((void**)&dRMP,sizeof(int)*(DFS_CODE.noElemOnRMP-2)));
		int idx;
		for (idx = 0; idx < RMP.size()-2; ++idx)
		{
			//std::printf("V[%d] Li[%d]; ",RMP[idx],vertexLabel[idx]);
			hRMP[idx] = RMP[idx];
			hRMPLabel[idx] = vertexLabel[idx];
		}

		CUCHECK(hipMemcpy(dRMP,hRMP,(DFS_CODE.noElemOnRMP-2)*sizeof(int),hipMemcpyHostToDevice));
		CUCHECK(hipMemcpy(dRMPLabel,hRMPLabel,(DFS_CODE.noElemOnRMP-2)*sizeof(int),hipMemcpyHostToDevice));
		int lastIdx = idx - 1;
		noElemMappingVj = hRMP[lastIdx] + 1;

		free(hRMP);
		free(hRMPLabel);
		RMP.clear();
		vertexLabel.clear();
	}
	catch(std::exception &exc)
	{
		cout<<endl<<exc.what()<<endl;
		FCHECK(-1);
	}
}

//use
void PMS::getVjBackwardDFSCODE(int* &dRMP,int &noElemOnRMP, \
							   int* &dVj,int &noElemdVj)
{
	try
	{
		//Chứa các Vj của các backward extension có thể có từ đỉnh cuối của DFS_CODE.
		vector<int> vertexVj;
		//Nếu cạnh cuối là backward thì xét cạnh kế cuối cho đến khi đó là forward.
		int vi,vj;
		vector<int> vjBackward;
		for(int i = DFS_CODE.size() - 1; i>=0;i--)
		{ 
			vi = DFS_CODE.at(i).from;
			vj = DFS_CODE.at(i).to;

			bool isForward = (vi<vj);
			//Nếu là forward thì không lấy Vj nữa
			if(isForward==true)
			{
				break;
			}
			else
			{
				vjBackward.push_back(vj);
			}
		}
		if (vjBackward.size() <=0) return;
		int* hValidBackward = (int*)malloc(vjBackward.size()*sizeof(int));
		if(hValidBackward == nullptr) {FCHECK(-1);}
		//Copy dữ liệu từ vjBackward sang mảng hValidBackward
		for (int i = 0; i < vjBackward.size(); i++)
		{
			hValidBackward[i] = vjBackward.at(i);
		}

		//update output
		noElemdVj = vjBackward.size();
		CUCHECK(hipMalloc((void**)&dVj,sizeof(int)*noElemdVj));
		//Copy dữ liệu từ hValidBackward sang dVj
		CUCHECK(hipMemcpy(dVj,hValidBackward,sizeof(int)*noElemdVj,hipMemcpyHostToDevice));
		//giải phóng bộ nhớ
		free(hValidBackward);
	}
	catch(std::exception &exc)
	{
		cout<<endl<<exc.what()<<endl;
		FCHECK(-1);
	}
}

//use
void PMS::buildExistBackwardInfo(int* &dRMP,int &noElemOnRMP, \
								 int* &dValidBackward)
{
	try
	{
		if(noElemOnRMP<=0) return;
		CUCHECK(hipMalloc((void**)&dValidBackward,noElemOnRMP*sizeof(int)));
		CUCHECK(hipMemset(dValidBackward,-1,noElemOnRMP*sizeof(int))); //-1 được xem là dVj đó chưa tồn tại Backard link nào.

		if (hEm.back().hBackwardEmbedding.size()<=0) return; //Chưa có backward nào thì return;

		int* dVjBackward = nullptr;
		int noElemdVjBackward = 0;
		getVjBackwardDFSCODE(dRMP,noElemOnRMP, \
								dVjBackward,noElemdVjBackward);
		if (noElemdVjBackward == 0)
		{
			return;
		}
		
		//Thread i mang giá trị dVj[i] quét trong mảng dValidBackward
		//Thread i set 1 tại existBackwardInfo.dValidBackward[i] nếu dV[i] tồn tại trong dValidBackward
		dim3 block(blocksize);
		dim3 grid((DFS_CODE.noElemOnRMP+block.x-1)/block.x);
		kernelFillValidBackward<<<grid,block>>>( \
			dValidBackward,dRMP,noElemOnRMP, \
			dVjBackward,noElemdVjBackward);
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());

		//cout<<endl<<"****dValidBackward*****"<<endl;
		//displayDeviceArr(dValidBackward,noElemOnRMP);
	}
	catch(const std::exception &exc)
	{
		std::cerr << exc.what();
		FCHECK(-1);
	}
}


//use
void PMS::getEmCol(Embedding** &dEmCol,int &noElemdEmCol)
{
	//Duyệt qua embedding columns và trích các Embedding Col từ hEm.
	//Nếu tại hEm.at(i) có backward col thì ưu tiên trích backward
	try
	{
		if (hEm.size()<=0) return;
		noElemdEmCol = hEm.size();
		size_t noBytedEmCol = hEm.size()*sizeof(Embedding**);
		CUCHECK(hipMalloc((void**)&dEmCol,noBytedEmCol));
		for (int i = 0; i < hEm.size(); i++)
		{
			//Mỗi phần tử của mảng dArrPointerEmbedding chứa địa chỉ của dArrEmbedding
			if (hEm.at(i).hBackwardEmbedding.size()>0)
			{
				kernelGetPointerdArrEmbedding<<<1,1>>>(hEm.at(i).hBackwardEmbedding.back().dArrEmbedding, \
					dEmCol, \
					i);
			}
			else
			{
				kernelGetPointerdArrEmbedding<<<1,1>>>(hEm.at(i).dArrEmbedding, dEmCol, i);
			}
		}
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());
		//-------------------------- đã có dEmCol chứa danh sách các pointer dArrEmbedding trên device.
	}
	catch(const std::exception &exc)
	{
		std::cerr << exc.what();
		FCHECK(-1);
	}
}

//use
void PMS::getEmColRMP(Embedding** &dEmCol,const int &noElemRMP)
{
	try
	{
		if (noElemRMP<=0) return;
		size_t noBytedEmCol = noElemRMP*sizeof(Embedding**);
		CUCHECK(hipMalloc((void**)&dEmCol,noBytedEmCol));

		for (int prevCol = hEm.size()-1; prevCol != -1; prevCol = hEm.at(prevCol).prevCol)
		{
			kernelGetPointerdArrEmbedding<<<1,1>>>(hEm.at(prevCol).dArrEmbedding, dEmCol, prevCol);
		}
		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());
	}
	catch(const std::exception &exc)
	{
		std::cerr << exc.what();
		FCHECK(-1);
	}
}

//use
void PMS::createMarkEmColRMP(int* &dRMP,int &noElemdRMP,int* &dEmColRMP)
{
	try
	{
		if (hEm.size()<=0) return;

		size_t noBytesdEmRMP = hEm.size()*sizeof(int);
		CUCHECK(hipMalloc((void**)&dEmColRMP,noBytesdEmRMP));
		CUCHECK(hipMemset(dEmColRMP,0,noBytesdEmRMP));
		
		//kernel update dEmColRMP
		dim3 block(blocksize);
		dim3 grid((noElemdRMP + block.x -1)/block.x);
		kernelCreatedEmRMP<<<grid,block>>>(dRMP, dEmColRMP,noElemdRMP);

		CUCHECK(hipDeviceSynchronize());
		CUCHECK(hipGetLastError());
	}
	catch(const std::exception &exc)
	{
		std::cerr << exc.what();
		FCHECK(-1);
	}
}

//use
void PMS::createRMP(int* &dRMP,int &noElem)
{
	//example: v0-- v1--v5--v7 is on RMP; noElem =4 and dRMP=[0,1,5,7]
	try
	{
		if(hEm.size()<=0) return;
		//Build list RMP
		list<int> lstRMP;
		lstRMP.push_front(hEm.size() -1);
		int nextPrevCol = hEm.back().prevCol;
		while(nextPrevCol != -1 )
		{
			lstRMP.push_front(nextPrevCol);
			nextPrevCol = hEm.at(nextPrevCol).prevCol;
		}
		//update noElem
		noElem = lstRMP.size();
		DFS_CODE.noElemOnRMP = noElem;
		//copy lstRMP to host temp memory
		int *hArrRMP = nullptr;
		hArrRMP = (int*)malloc(sizeof(int)*noElem);
		if(hArrRMP == nullptr) FCHECK(-1);

		for (int idxLstRMP = 0; idxLstRMP < noElem; idxLstRMP++)
		{
			hArrRMP[idxLstRMP] = lstRMP.front();
			lstRMP.pop_front();
		}
		//clear lstRMP
		lstRMP.clear();

		//update dRMP base on host temp memory.
		CUCHECK(hipMalloc((void**)&dRMP,sizeof(int)*noElem));
		CUCHECK(hipMemcpy(dRMP,hArrRMP,sizeof(int)*noElem,hipMemcpyHostToDevice));

		free(hArrRMP);
	}
	catch(const std::exception &exc)
	{
		std::cerr << exc.what();
		FCHECK(-1);
	}
}

//use
void PMS::getnoElemEmbedding(int &noElemEmbedding)
{
	try
	{
		noElemEmbedding = 0;
		if (hEm.size() <= 0) return;
		if(hEm.back().hBackwardEmbedding.size()>0)
		{
			noElemEmbedding = hEm.back().hBackwardEmbedding.back().noElem;
		}
		else
		{
			noElemEmbedding = hEm.back().noElem;
		}
	}
	catch(const std::exception &exc)
	{
		std::cerr << exc.what();
		FCHECK(-1);
	}
}