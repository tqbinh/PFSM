#include "hip/hip_runtime.h"
#pragma once
#include "pms.cuh"


float hTime=0.0;
float dTime=0.0;

PMS::PMS(){
	Level=0;
	idxLevel=0;
	Lv=0;
	Le=0;
	maxOfVer=0;
	numberOfGraph=0;
	minLabel = -1;
	maxId = -1;
	//std::cout<<" PMS initialized " << std::endl;
	//char* outfile;
	//outfile = "/result.graph";
	//fos.open(outfile);	
}
PMS::~PMS(){
	//std::cout<<" PMS terminated " << std::endl;
	//fos.close();

	if(hLevelUniEdge.size()>0){
		hLevelUniEdge.clear();
	}

	if(hLevelEXT.size()>0){
		hLevelEXT.clear();
	}

	if(hdb.size()!=0){
		for (int i = 0; i < hdb.size(); i++)
		{
			hipFree(hdb.at(i).dO);
			hipFree(hdb.at(i).dLO);
			hipFree(hdb.at(i).dN);
			hipFree(hdb.at(i).dLN);
		}
		hdb.clear();
	}
	if(hExtension.size()!=0){
		for (int i = 0; i < hExtension.size(); i++)
		{
			hipFree(hExtension.at(i).dExtension);
		}
		hExtension.clear();
	}

	if(hUniEdge.size()!=0){
		for (int i = 0; i < hUniEdge.size(); i++)
		{
			hipFree(hUniEdge.at(i).dUniEdge);
		}
		hUniEdge.clear();
	}

	if(hUniEdgeSatisfyMinsup.size()!=0){
		for (int i = 0; i < hUniEdgeSatisfyMinsup.size(); i++)
		{			
			hipFree(hUniEdgeSatisfyMinsup.at(i).dUniEdge);
			free(hUniEdgeSatisfyMinsup.at(i).hArrSup);					
		}
		hUniEdgeSatisfyMinsup.clear();
	}
	hipDeviceReset();
}

void PMS::prepareDataBase(){
	//unsigned int minsup = 34;
	unsigned int minsup = 2;
	unsigned int maxpat = 2;
	//unsigned int maxpat = 0x00000000;
	unsigned int minnodes = 0;
	bool where = true;
	bool enc = false;
	bool directed = false;

	//int opt;
	char* fname;
	//fname = "Klesscus";
	//fname = "Klessorigin";
	//fname = "KlessoriginCust1";
	fname= "G0G1G2_custom";
	//fname="Chemical_340Origin";

	////PMS pms;	
	ofstream fout("result.txt");

	//Chuyển dữ liệu từ fname sang TRANS
	//pms.run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);
	run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);
	//maxOfVer=pms.findMaxVertices();
	maxOfVer=findMaxVertices();	
	numberOfGraph=noGraphs();
	int sizeOfarrayO=maxOfVer*numberOfGraph;
	int* arrayO = new int[sizeOfarrayO]; //Tạo mảng arrayO có kích thước D*m
	if(arrayO==NULL){
		printf("\n!!!Memory Problem ArrayO");
		exit(1);
	}else{
		memset(arrayO, -1, sizeOfarrayO*sizeof(int)); // gán giá trị cho các phần tử mảng bằng -1
	}
	unsigned int noDeg; //Tổng bậc của tất cả các đỉnh trong csdl đồ thị TRANS
	//noDeg = pms.sumOfDeg();
	noDeg = sumOfDeg();
	//cout<<noDeg;
	unsigned int sizeOfArrayN=noDeg;
	int* arrayN = new int[sizeOfArrayN]; //Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	if(arrayN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayN");
		exit(1);
	}else
	{
		memset(arrayN, -1, noDeg*sizeof(int));
	}

	//
	int* arrayLO = new int[sizeOfarrayO]; //Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	if(arrayLO==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLO");
		exit(1);
	}else
	{
		memset(arrayLO, -1, sizeOfarrayO*sizeof(int));
	}



	int* arrayLN = new int[noDeg]; //Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	if(arrayLN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLN");
		exit(1);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}


	//pms.importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);
	importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);
	cout<<"ArrayO:";
	displayArray(arrayO,sizeOfarrayO);
	cout<<"\nArrayLO:";
	displayArray(arrayLO,sizeOfarrayO);
	cout<<"\nArrayN:";
	displayArray(arrayN,noDeg);
	cout<<"\nArrayLN:";
	displayArray(arrayLN,noDeg);
	//kích thước của dữ liệu
	size_t nBytesO = sizeOfarrayO*sizeof(int);
	size_t nBytesN = noDeg*sizeof(int);

	DB graphdb;
	graphdb.noElemdO = sizeOfarrayO;
	graphdb.noElemdN = noDeg;

	CHECK(hipMalloc((void**)&graphdb.dO,nBytesO));
	CHECK(hipMalloc((void**)&graphdb.dLO,nBytesO));
	CHECK(hipMalloc((void**)&graphdb.dN,nBytesN));
	CHECK(hipMalloc((void**)&graphdb.dLN,nBytesN));

	CHECK(hipMemcpy(graphdb.dO,arrayO,nBytesO,hipMemcpyHostToDevice));
	CHECK(hipMemcpy(graphdb.dLO,arrayLO,nBytesO,hipMemcpyHostToDevice));
	CHECK(hipMemcpy(graphdb.dN,arrayN,nBytesN,hipMemcpyHostToDevice));
	CHECK(hipMemcpy(graphdb.dLN,arrayLN,nBytesN,hipMemcpyHostToDevice));

	//pms.db.push_back(graphdb); //Đưa cơ sở dữ liệu vào vector db
	//pms.countNumberOfDifferentValue(pms.db.at(0).dLO,pms.db.at(0).noElemdO,pms.Lv);
	//pms.countNumberOfDifferentValue(pms.db.at(0).dLN,pms.db.at(0).noElemdN,pms.Le);
	hdb.push_back(graphdb); //Đưa cơ sở dữ liệu vào vector db
	countNumberOfDifferentValue(hdb.at(0).dLO,hdb.at(0).noElemdO,Lv);
	countNumberOfDifferentValue(hdb.at(0).dLN,hdb.at(0).noElemdN,Le);
	//pms.printdb();
}

bool PMS::checkArray(int *hostRef, int *gpuRef, const int N) {
	bool result=true;
	double epsilon = 1.0E-8;
	int match = 1;
	for (int i = 0; i < N; i++) {
		if ((float)(abs(hostRef[i] - gpuRef[i])) > epsilon) {
			match = 0;
			result=false;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n",
				hostRef[i], gpuRef[i], i);
			break;
		}
	}
	if (match){
		printf("Arrays match.\n\n");		
	}

	return result;
}


void PMS::displayArray(int *p, const unsigned int pSize=0)
{
	for(int i=0;i<pSize;i++){
		printf("P[%d]:%d ",i,p[i]);
	}
	printf("\n");
	return;
}

__global__ void kernelPrintdArr(int *dArr,unsigned int noElem){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("A[%d]:%d ",i,dArr[i]);
	}
}


void PMS::printdb(){
	printf("\n *********** Lv, Le **********\n");
	printf("\n Lv:%d",Lv);
	printf("\n Le:%d",Le);
	for (int i = 0; i < hdb.size(); i++)
	{
		unsigned int noElem =  hdb.at(i).noElemdO;	


		dim3 block(blocksize);
		dim3 grid((noElem + block.x -1)/block.x);
		printf("\n ********* dO *********\n");
		kernelPrintdArr<<<grid,block>>>(hdb.at(i).dO,noElem);
		hipDeviceSynchronize();
		printf("\n");

		printf("\n ********* dLO *********\n");
		kernelPrintdArr<<<grid,block>>>(hdb.at(i).dLO,noElem);
		hipDeviceSynchronize();
		printf("\n");

		unsigned int noElemdN = hdb.at(i).noElemdN;
		dim3 blocka(blocksize);
		dim3 grida((noElemdN + blocka.x -1)/blocka.x);

		printf("\n ********* dN *********\n");
		kernelPrintdArr<<<grida,blocka>>>(hdb.at(i).dN,noElemdN);
		hipDeviceSynchronize();
		printf("\n");

		printf("\n ********* dLN *********\n");
		kernelPrintdArr<<<grida,blocka>>>(hdb.at(i).dLN,noElemdN);
		hipDeviceSynchronize();
		printf("\n");
	}
}


__global__ void kernelMyScanV(int *dArrInput,int noElem,int *dResult){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		if(i==0){
			dResult[i]=0;
		}else
		{
			int temp=0;
			for (int j = 0; j <= (i-1); j++)
			{
				temp=temp + dArrInput[j];
			}
			dResult[i]=temp;
		}
	}
}


hipError_t  myScanV(int *dArrInput,int noElem,int *&dResult){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem + block.x -1)/block.x);

	CHECK(cudaStatus=hipMalloc((void**)&dResult,noElem * sizeof(int)));
	if(cudaStatus!=hipSuccess){
		goto Error;
	}
	kernelMyScanV<<<grid,block>>>(dArrInput,noElem,dResult);
	hipDeviceSynchronize();
	CHECK(cudaStatus = hipGetLastError());
	if(cudaStatus!=hipSuccess){
		goto Error;
	}
Error:
	return cudaStatus;
}


__global__ void kernelCountNumberOfLabelVertex(int *d_LO,int *d_Lv,unsigned int sizeOfArrayLO){
	int i= blockDim.x*blockIdx.x + threadIdx.x;
	if(i<sizeOfArrayLO){
		if(d_LO[i]!=-1){
			d_Lv[d_LO[i]]=1;
		}
	}
}

void sumUntilReachZero(int *h_Lv,unsigned int n,int &result){
	for(int i=0;i<n && h_Lv[i]!=0;++i){
		++result;
	}
}

int  PMS::countNumberOfDifferentValue(int* d_LO,unsigned int sizeOfArrayLO, unsigned int &numberOfSaperateVertex){
	int status=0;
	hipError_t cudaStatus;
	numberOfSaperateVertex=0;
	size_t nBytesLv = sizeOfArrayLO*sizeof(int);
	//cấp phát mảng d_Lv trên device
	int *d_Lv;
	cudaStatus=hipMalloc((int**)&d_Lv,nBytesLv);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_Lv failed");
		goto Error;
	}
	else
	{
		hipMemset(d_Lv,0,nBytesLv);
	}

	//Cấp phát threads
	dim3 block(blocksize);
	dim3 grid((sizeOfArrayLO+block.x-1)/block.x);
	kernelCountNumberOfLabelVertex<<<grid,block>>>(d_LO,d_Lv,sizeOfArrayLO);

	hipDeviceSynchronize();
	printf("\nElements of d_Lv:");
	kernelPrintdArr<<<grid,block>>>(d_Lv,sizeOfArrayLO);

	int* h_Lv=NULL;
	h_Lv=(int*)malloc(nBytesLv);
	if(h_Lv==NULL){
		printf("h_Lv malloc memory fail");
		exit(1);
	}
	hipMemcpy(h_Lv,d_Lv,nBytesLv,hipMemcpyDeviceToHost);
	cudaStatus=hipDeviceSynchronize();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize fail",cudaStatus);
		status = -1;
		goto Error;
	}
	int result=0;
	sumUntilReachZero(h_Lv,sizeOfArrayLO,result);
	numberOfSaperateVertex=result;	

Error:
	hipFree(d_Lv);	
	return status;
}

__global__ void kernelGetAndStoreExtension(int *d_O,int *d_LO,unsigned int numberOfElementd_O,int *d_N,int *d_LN,unsigned int numberOfElementd_N,Extension *d_Extension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<numberOfElementd_O){
		if (d_O[i]!=-1){
			int j;
			int ek;
			//printf("\nThread:%d",i);	
			for(j=i+1;j<numberOfElementd_O;++j){					
				if(d_O[j]!=-1) {break;}				
			}			

			if (j==numberOfElementd_O) {
				ek=numberOfElementd_N;
			}
			else
			{
				ek=d_O[j];
			}
			//printf("\n[%d]:%d",i,ek);
			for(int k=d_O[i];k<ek;k++){
				//do something
				int index= k;
				d_Extension[index].vi=0;
				d_Extension[index].vj=0;
				d_Extension[index].li=d_LO[i];
				d_Extension[index].lij=d_LN[k];
				d_Extension[index].lj=d_LO[d_N[k]];
				d_Extension[index].vgi=i;
				d_Extension[index].vgj=d_N[k];
				//printf("\n[%d]:%d",i,index);
				/*printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",k,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,
				d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);*/
			}
		}
	}
}


int PMS::getAndStoreExtension(Extension *&d_Extension){
	int status =0;
	hipError_t cudaStatus;
	dim3 block(blocksize);
	unsigned int numberOfElementd_O = hdb.at(0).noElemdO;
	dim3 grid((numberOfElementd_O+block.x-1)/block.x);

	kernelGetAndStoreExtension<<<grid,block>>>(hdb.at(0).dO,hdb.at(0).dLO,numberOfElementd_O,hdb.at(0).dN,hdb.at(0).dLN,hdb.at(0).noElemdN,d_Extension);

	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelGetAndStoreExtension failed",cudaStatus);
		status =-1;
		goto Error;
	}


Error:
	return status;
}


int PMS::extractAllEdgeInDB(){
	int status = 0;
	arrExtension arrE;
	//cấp phát bộ nhớ cho d_Extension
	arrE.noElem =hdb.at(0).noElemdN;
	size_t nBytesOfArrayExtension = arrE.noElem*sizeof(Extension);

	CHECK(hipMalloc((Extension**)&arrE.dExtension,nBytesOfArrayExtension));
	//Trích tất cả các cạnh từ database rồi lưu vào d_Extension

	status  = getAndStoreExtension(arrE.dExtension);
	if(status ==-1){
		printf("\n getAndStoreExtension(arrE.dExtension) in extractAllEdgeInDB() failed");
		goto Error;
	}

	hExtension.push_back(arrE);
Error:
	return status;
}

__global__ void kernelPrintExtention(Extension *d_Extension,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){		
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);
	}
}

int PMS::displayArrExtension(Extension *dExtension,int noElem){
	int status =0;
	hipError_t cudaStatus;
	//dim3 block(blocksize);
	//dim3 grid((noElem + block.x - 1)/block.x);

	//kernelPrintExtention<<<grid,block>>>(dExtension,noElem);
	//hipDeviceSynchronize();
	Extension *hExtension = (Extension*)malloc(sizeof(Extension)*noElem);
	if(hExtension==NULL){
		status=-1;
		printf("\n Malloc hExtension in displayArrExtension() failed");
		goto Error;
	}
	CHECK(cudaStatus=hipMemcpy(hExtension,dExtension,sizeof(Extension)*noElem,hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	for (int i = 0; i < noElem; i++)
	{
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,hExtension[i].vi,hExtension[i].vj,hExtension[i].li,hExtension[i].lij,hExtension[i].lj,hExtension[i].vgi,hExtension[i].vgj);
	}
Error:
	return status;
}

int PMS::displayArrUniEdge(UniEdge* dUniEdge,int noElem){
	hipError_t cudaStatus;
	int status =0;
	UniEdge *hUniEdge = (UniEdge*)malloc(sizeof(UniEdge)*noElem);
	if(hUniEdge==NULL){
		status=-1;
		printf("\n malloc hUniEdge in displayArrUniEdge() failed");
		goto Error;
	}
	CHECK(cudaStatus=hipMemcpy(hUniEdge,dUniEdge,sizeof(UniEdge)*noElem,hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	for (int i = 0; i < noElem; i++)
	{
		printf("\n U[%d]: (li lij lj) = (%d %d %d)",i,hUniEdge[i].li,hUniEdge[i].lij,hUniEdge[i].lj);
	}
	free(hUniEdge);
Error:
	return status;
}


__global__ void	kernelValidEdge(Extension *d_Extension,int *dV,unsigned int numberElementd_Extension){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){	
		if(d_Extension[i].li<=d_Extension[i].lj){
			dV[i]=1;
		}
	}
}


hipError_t validEdge(Extension *d_Extension,int *&dV,unsigned int numberElementd_Extension){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid(numberElementd_Extension+block.x-1/block.x);

	kernelValidEdge<<<grid,block>>>(d_Extension,dV,numberElementd_Extension);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelValidEdge failed",cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

__global__ void kernelPrintdArr(float *dArr,int noElem){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("A[%d]:%d   ",i,dArr[i]);
	}
}
__global__ void kernelPrintdArr(int *dArr,int noElem){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("dArr[%d]:%d ",i,dArr[i]);
	}
}
int PMS::displayDeviceArr(int *dArr,int noElem){
	int status =0;
	//dim3 block(blocksize);
	//dim3 grid((noElem + block.x -1)/block.x);
	//kernelPrintdArr<<<grid,block>>>(dArr,noElem);
	//hipDeviceSynchronize();
	//hipError_t cudaStatus = hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n kernelDisplayDeviceArr() in displayDeviceArr() failed",cudaStatus);
	//	status = -1;
	//	goto Error;
	//}

	int *temp = (int*)malloc(sizeof(int)*noElem);
	if(temp==NULL){
		printf("\n Malloc temp in displayDeviceArr() failed");
		status=-1;
		goto Error;	
	}

	CHECK(hipMemcpy(temp,dArr,noElem*sizeof(int),hipMemcpyDeviceToHost));

	for (int i = 0; i < noElem; i++)
	{
		printf(" A[%d]:%d  ",i,temp[i]);
	}

	free(temp);
Error:

	return 0;
}

int PMS::displayDeviceArr(float *dArr,int noElem){
	int status =0;
	//dim3 block(blocksize);
	//dim3 grid((noElem + block.x -1)/block.x);
	//kernelPrintdArr<<<grid,block>>>(dArr,noElem);
	//hipDeviceSynchronize();
	//hipError_t cudaStatus = hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n kernelDisplayDeviceArr() in displayDeviceArr() failed",cudaStatus);
	//	status = -1;
	//	goto Error;
	//}
	float *temp = (float*)malloc(sizeof(float)*noElem);
	if(temp==NULL){
		printf("\n Malloc temp in displayDeviceArr() failed");
		status=-1;
		goto Error;	
	}

	CHECK(hipMemcpy(temp,dArr,noElem*sizeof(float),hipMemcpyDeviceToHost));

	for (int i = 0; i < noElem; i++)
	{
		int a = (int)temp[i];
		printf(" A[%d]:%d  ",i,a);
	}

	free(temp);
Error:
	return status;
}



__global__ void kernelGetSize(int *dV,int *dVScanResult,int noElem,int *size){
	*size = dVScanResult[noElem-1];
	if(dV[noElem-1]==1){
		*size = *size + 1;
	}
}


hipError_t getSizeBaseOnScanResult(int *dV,int *dVScanResult,int noElem,int &output){
	hipError_t cudaStatus;
	int temp=0;
	int *size=nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&size,sizeof(int)));
	if(cudaStatus!=hipSuccess){
		goto Error;
	}

	CHECK(cudaStatus=hipMemset(size,0,sizeof(int)));
	if(cudaStatus!=hipSuccess){
		goto Error;
	}

	kernelGetSize<<<1,1>>>(dV,dVScanResult,noElem,size);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\n kernelGetSize() in getSizeBaseOnResult() failed",cudaStatus);
		goto Error;
	}

	CHECK(hipMemcpy(&temp,size,sizeof(int),hipMemcpyDeviceToHost));
	output = (int)temp;

	hipFree(size);
Error:

	return cudaStatus;
}

__global__ void kernelExtractValidExtension(Extension *d_Extension,int *dV,int *dVScanResult,int numberElementd_Extension,Extension *d_ValidExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){
		if(dV[i]==1){
			int index = dVScanResult[i];
			//printf("\nV[%d]:%d, index[%d]:%d,d_Extension[%d], d_Extension[%d]:%d\n",i,V[i],i,index[i],i,i,d_Extension[i].vgi);
			d_ValidExtension[index].li=d_Extension[i].li;
			d_ValidExtension[index].lj=d_Extension[i].lj;
			d_ValidExtension[index].lij=d_Extension[i].lij;
			d_ValidExtension[index].vgi=d_Extension[i].vgi;
			d_ValidExtension[index].vgj=d_Extension[i].vgj;
			d_ValidExtension[index].vi=d_Extension[i].vi;
			d_ValidExtension[index].vj=d_Extension[i].vj;
		}
	}
}

hipError_t extractValidExtension(Extension *d_Extension,int *dV,int *dVScanResult, int numberElementd_Extension,Extension *&d_ValidExtension){
	hipError_t cudaStatus;

	//printfExtension(d_Extension,numberElementd_Extension);

	dim3 block(blocksize);
	dim3 grid((numberElementd_Extension+block.x)/block.x);

	kernelExtractValidExtension<<<grid,block>>>(d_Extension,dV,dVScanResult,numberElementd_Extension,d_ValidExtension);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelGetValidExtension failed");
		goto Error;
	}

Error:
	return cudaStatus;
}

int PMS::getValidExtension(){
	int status = 0;


	//Phase 1: đánh dấu vị trí những cạnh hợp lệ (li<=lj)

	int numberElementd_Extension = hExtension.at(0).noElem;
	int *dV;
	size_t nBytesdV= numberElementd_Extension*sizeof(int);

	hipError_t cudaStatus=hipMalloc((void**)&dV,nBytesdV);
	if (cudaStatus!= hipSuccess){
		fprintf(stderr,"hipMalloc array V failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dV,0,nBytesdV));
	}

	cudaStatus=validEdge(hExtension.at(0).dExtension,dV,hExtension.at(0).noElem);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize validEdge failed",cudaStatus);
		status = -1;
		goto Error;
	}
	//
	int *hV = (int*)malloc(sizeof(int)*numberElementd_Extension);
	hipMemcpy(hV,dV,sizeof(int)*numberElementd_Extension,hipMemcpyDeviceToHost);
	printf("\n ************ dV **************\n");
	for (int i = 0; i < numberElementd_Extension; i++)
	{
		int temp = hV[i];
		printf("[%d]:%d ",i,temp);
	}

	int* dVScanResult;
	cudaStatus=hipMalloc((void**)&dVScanResult,numberElementd_Extension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"Cuda Malloc failed",cudaStatus);
		goto Error;
	}	
	else
	{
		hipMemset(dVScanResult,0,numberElementd_Extension*sizeof(int));
	}
	//Exclusive scan mảng V và lưu kết quả scan vào mảng index
	//scanV(dV,numberElementd_Extension,dVScanResult);
	myScanV(dV,numberElementd_Extension,dVScanResult);

	printf("\n ************ dVScanResult **************\n");
	hipMemcpy(hV,dVScanResult,sizeof(int)*numberElementd_Extension,hipMemcpyDeviceToHost);
	for (int i = 0; i < numberElementd_Extension; i++)
	{
		int temp = hV[i]; 
		printf("[%d]:%d ",i,temp);
	}

	//Phase 2: trích những cạnh hợp lệ sang một mảng khác dValidExtension
	//arrExtension arrValidExtension;
	hValidExtension.resize(1);
	getSizeBaseOnScanResult(dV,dVScanResult,numberElementd_Extension,hValidExtension.at(0).noElem);
	printf("\n arrValidExtension.noElem:%d",hValidExtension.at(0).noElem);
	CHECK(hipMalloc((void**)&(hValidExtension.at(0).dExtension),sizeof(Extension)*hValidExtension.at(0).noElem));
	CHECK(extractValidExtension(hExtension.at(0).dExtension,dV,dVScanResult,numberElementd_Extension,hValidExtension.at(0).dExtension));
	//displayArrExtension(arrValidExtension.dExtension,arrValidExtension.noElem);

	//hValidExtension.push_back(arrValidExtension);
	printf("\n************hValidExtension***********\n");
	displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

	hipFree(dV);
	hipFree(dVScanResult);
	free(hV);
Error:	
	return status;
}

__global__ void kernelMarkLabelEdge(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *d_allPossibleExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		int index=	d_ValidExtension[i].li*Lv*Le + d_ValidExtension[i].lij*Lv + d_ValidExtension[i].lj;
		d_allPossibleExtension[index]=1;
	}

}

hipError_t markLabelEdge(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *&d_allPossibleExtension){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	kernelMarkLabelEdge<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,Lv,Le,d_allPossibleExtension);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"kernelMarkLabelEdge in markLabelEdge() failed");
		goto Error;
	}

Error:
	return cudaStatus;
}

__global__ void kernelCalcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,UniEdge *d_UniqueExtension,unsigned int Le,unsigned int Lv){
	int i=blockIdx.x*blockDim.x + threadIdx.x;	
	if(i<noElem_allPossibleExtension && d_allPossibleExtension[i]==1){
		int li,lj,lij;
		li=i/(Le*Lv);
		lij=(i%(Le*Lv))/Lv;
		lj=(i%(Le*Lv))-((i%(Le*Lv))/Lv)*Lv;
		int index = d_allPossibleExtensionScanResult[i];
		//printf("\n[%d]:%d li:%d lij:%d lj:%d",i,d_allPossibleExtensionScanResult[i],li,lij,lj);
		d_UniqueExtension[index].li=li;
		d_UniqueExtension[index].lij=lij;
		d_UniqueExtension[index].lj=lj;
	}
}

__global__ void kernelReverseMappingToUniEdgeLabel(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,UniEdge *d_UniqueExtension,unsigned int Le,unsigned int Lv){
	int i=blockIdx.x*blockDim.x + threadIdx.x;	
	if(i<noElem_allPossibleExtension && d_allPossibleExtension[i]==1){
		int li,lj,lij;
		li=i/(Le*Lv);
		lij=(i%(Le*Lv))/Lv;
		lj=(i%(Le*Lv))-((i%(Le*Lv))/Lv)*Lv;
		int index = d_allPossibleExtensionScanResult[i];
		//printf("\n[%d]:%d li:%d lij:%d lj:%d",i,d_allPossibleExtensionScanResult[i],li,lij,lj);
		d_UniqueExtension[index].li=li;
		d_UniqueExtension[index].lij=lij;
		d_UniqueExtension[index].lj=lj;
	}
}


hipError_t calcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,UniEdge *&d_UniqueExtension,unsigned int noElem_d_UniqueExtension,unsigned int Le,unsigned int Lv){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_allPossibleExtension+block.x-1)/block.x);
	kernelCalcLabelAndStoreUniqueExtension<<<grid,block>>>(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,Le,Lv);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"kernelCalcLabelAndStoreUniqueExtension in CalcLabelAndStoreUniqueExtension() failed");
		goto Error;
	}

Error:
	return cudaStatus;
}


hipError_t reverseMappingToUniEdgeLabel(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,UniEdge *&d_UniqueExtension,unsigned int noElem_d_UniqueExtension,unsigned int Le,unsigned int Lv){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_allPossibleExtension+block.x-1)/block.x);
	kernelReverseMappingToUniEdgeLabel<<<grid,block>>>(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,Le,Lv);
	hipDeviceSynchronize();
	CHECK(cudaStatus=hipGetLastError());
	if(cudaStatus!=hipSuccess){
		goto Error;
	}

Error:
	return cudaStatus;
}


int PMS::extractUniEdge(){
	int status=0;


	//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
	unsigned int noElem_dallPossibleExtension=Le*Lv*Lv;
	int *d_allPossibleExtension;

	//cấp phát bộ nhớ cho mảng d_allPossibleExtension
	hipError_t	cudaStatus=hipMalloc((void**)&d_allPossibleExtension,noElem_dallPossibleExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtension failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(d_allPossibleExtension,0,noElem_dallPossibleExtension*sizeof(int)));
	}

	cudaStatus=markLabelEdge(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,Lv,Le,d_allPossibleExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"getUniqueExtension failed",cudaStatus);
		status=-1;
		goto Error;
	}


	int *d_allPossibleExtensionScanResult;
	cudaStatus=hipMalloc((void**)&d_allPossibleExtensionScanResult,noElem_dallPossibleExtension*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtensionScanResult failed");
		status = -1;
		goto Error;
	}
	// printf("\n **************** hValidExtension ****************\n");
	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

	//Exclusive scan mảng d_allPossibleExtension và lưu kết quả vào mảng d_allPossibleExtensionScanResult
	//cudaStatus = scanV(d_allPossibleExtension,noElem_dallPossibleExtension,d_allPossibleExtensionScanResult);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n ScanV() in computeSupport() failed");
	//	status = -1;
	//	goto Error;
	//}
	myScanV(d_allPossibleExtension,noElem_dallPossibleExtension,d_allPossibleExtensionScanResult);
	//	 printf("\n **************** hValidExtension ****************\n");
	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);


	//printf("\n**********d_allPossibleExtension************\n");
	//displayDeviceArr(d_allPossibleExtension,noElem_dallPossibleExtension);


	arrUniEdge strUniEdge;
	int noElem_d_UniqueExtension=0;
	//Tính kích thước của mảng d_UniqueExtension dựa vào kết quả exclusive scan
	cudaStatus=getSizeBaseOnScanResult(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_dallPossibleExtension,noElem_d_UniqueExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getLastElement() in extractUniEdge() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	//printf("\n\nnoElem_d_UniqueExtension:%d",noElem_d_UniqueExtension);
	strUniEdge.noElem = noElem_d_UniqueExtension;



	//Tạo mảng d_UniqueExtension với kích thước mảng vừa tính được
	cudaStatus=hipMalloc((void**)&strUniEdge.dUniEdge,noElem_d_UniqueExtension*sizeof(UniEdge));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_UniqueExtension in extractUniEdge() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(strUniEdge.dUniEdge,0,noElem_d_UniqueExtension*sizeof(UniEdge)));
	}




	//Ánh xạ ngược lại từ vị trí trong d_allPossibleExtension thành cạnh và lưu kết quả vào d_UniqueExtension
	cudaStatus=calcLabelAndStoreUniqueExtension(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_dallPossibleExtension,strUniEdge.dUniEdge,noElem_d_UniqueExtension,Le,Lv);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n\ncalcLabelAndStoreUniqueExtension() in extractUniEdge() failed",cudaStatus);
		status = -1;
		goto Error;
	}



	hUniEdge.push_back(strUniEdge);
	printf("\n **************** hUniEdge ****************\n");
	displayArrUniEdge(hUniEdge.at(0).dUniEdge,hUniEdge.at(0).noElem);


	hipFree(d_allPossibleExtension);
	hipFree(d_allPossibleExtensionScanResult);
Error:

	return status;
}

__global__ void kernelCalcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *dB,unsigned int maxOfVer){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension-1){
		unsigned int graphIdAfter=d_ValidExtension[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=d_ValidExtension[i].vgi/maxOfVer;
		unsigned int resultDiff=graphIdAfter-graphIdCurrent;
		dB[i]=resultDiff;
	}
}

hipError_t calcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *&dB,unsigned int maxOfVer){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x)/block.x);

	kernelCalcBoundary<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,dB,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\kernelCalcBoundary in calcBoundary() failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
__global__ void kernelGetLastElement(int *dScanResult,unsigned int noElem,int *output){
	output[0]=dScanResult[noElem-1];
}


hipError_t getLastElement(int *dScanResult,unsigned int noElem,int &output){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem+block.x-1)/block.x);

	int *value;
	hipMalloc((int**)&value,sizeof(int));

	kernelGetLastElement<<<1,1>>>(dScanResult,noElem,value);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}

	CHECK(hipMemcpy(&output,value,sizeof(int),hipMemcpyDeviceToHost));
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);

	hipFree(value);
Error:
	return cudaStatus;	
}

__global__ void kernelSetValuedF(UniEdge *dUniEdge,int noElemdUniEdge,Extension *dValidExtension,int noElemdValidExtension,int *dBScanResult,int *dF,int noElemF){
	int i = blockDim.x * blockIdx.x +threadIdx.x;
	if(i<noElemdValidExtension){
		for (int j = 0; j < noElemdUniEdge; j++)
		{
			if(dUniEdge[j].li==dValidExtension[i].li && dUniEdge[j].lij==dValidExtension[i].lij &&	dUniEdge[j].lj==dValidExtension[i].lj){
				dF[dBScanResult[i]+j*noElemF]=1;
			}
		}
	}
}

__global__ void kernelCopyFromdFtoTempF(int *d_F,int *tempF,int from,int noElemNeedToCopy){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemNeedToCopy){
		int index = from*noElemNeedToCopy + i;
		tempF[i]=d_F[index];
	}
}

hipError_t calcSupport(UniEdge *dUniEdge,int noElemdUniEdge,Extension *dValidExtension,int noElemdValidExtension,int *dBScanResult,int *dF,int noElemF,int *&hResultSup){
	hipError_t cudaStatus;

	//Đánh dấu những đồ thị chứa embedding trong mảng d_F
	dim3 block(blocksize);
	dim3 grid((noElemdValidExtension+block.x - 1)/block.x);
	kernelSetValuedF<<<grid,block>>>(dUniEdge,noElemdUniEdge,dValidExtension,noElemdValidExtension,dBScanResult,dF,noElemF);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() of kernelComputeSupport in computeSupport failed",cudaStatus);
		goto Error;
	}



	//Duyệt qua mảng d_UniqueExtension, tính reduction cho mỗi segment i*noElemF, kết quả của reduction là độ support của cạnh i trong d_UniqueExtension
	int *tempF;
	cudaStatus = hipMalloc((void**)&tempF,noElemF*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n CudaMalloc tempF in calcSupport() failed",cudaStatus);
		goto Error;
	}
	else
	{
		CHECK(hipMemset(tempF,0,noElemF*sizeof(int)));
	}

	//float *resultSup; /* Lưu kết quả reduction */
	hResultSup = (int*)malloc(noElemdUniEdge*sizeof(int));
	if (hResultSup==NULL){
		printf("\n Malloc hResultSup in calcSupport() failed");
		exit(1);
	}

	dim3 blocka(blocksize);
	dim3 grida((noElemF+blocka.x-1)/blocka.x);
	/*int from =0;*/	
	for (int i = 0; i < noElemdUniEdge; i++)
	{		
		//chép dữ liệu d_F sang tempF ứng theo các phần tử lần lược là i*noElemF, copy đúng noElemF
		/*from =i;*/				
		kernelCopyFromdFtoTempF<<<grid,block>>>(dF,tempF,i,noElemF);
		hipDeviceSynchronize();
		reduction(tempF,noElemF,hResultSup[i]);		
	}
	////In độ hỗ trợ cho các cạnh tương ứng trong mảng kết quả resultSup
	//for (int i = 0; i < noElemdUniEdge; i++)
	//{
	//	printf("\n resultSup[%d]:%d",i,hResultSup[i]);
	//}

	hipFree(tempF);

Error:
	return cudaStatus;

}

__global__ void	kernelMarkUniEdgeSatisfyMinsup(int *dResultSup,int noElemUniEdge,int *dV,unsigned int minsup){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge){
		int temp = dResultSup[i];
		if(temp >= minsup){
			dV[i]=1;
		}
	}
}

__global__ void	kernelExtractUniEdgeSatifyMinsup(UniEdge *dUniEdge,int *dV,int *dVScanResult,int noElemUniEdge,UniEdge *dUniEdgeSatisfyMinsup,int *dSup,int *dResultSup){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge){
		if(dV[i]==1){
			dUniEdgeSatisfyMinsup[dVScanResult[i]].li = dUniEdge[i].li;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lij = dUniEdge[i].lij;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lj=dUniEdge[i].lj;
			dSup[dVScanResult[i]]=dResultSup[i];
		}
	}
}



int PMS::extractUniEdgeSatisfyMinsup(int *hResultSup,int noElemUniEdge,unsigned int minsup){
	int status=0;
	hipError_t cudaStatus;
	//1. Cấp phát mảng trên device có kích thước bằng noElemUniEdge
	int *dResultSup=nullptr;
	CHECK(hipMalloc((void**)&dResultSup,noElemUniEdge*sizeof(int)));
	CHECK(hipMemcpy(dResultSup,hResultSup,noElemUniEdge*sizeof(int),hipMemcpyHostToDevice));

	printf("\n *******dResultSup********\n");
	displayDeviceArr(dResultSup,noElemUniEdge);

	//2. Đánh dấu 1 trên dV cho những phần tử thoả minsup
	int *dV=nullptr;
	CHECK(hipMalloc((void**)&dV,noElemUniEdge*sizeof(int)));
	CHECK(hipMemset(dV,0,sizeof(int)*noElemUniEdge));

	dim3 block(blocksize);
	dim3 grid((noElemUniEdge + block.x - 1)/block.x);
	kernelMarkUniEdgeSatisfyMinsup<<<grid,block>>>(dResultSup,noElemUniEdge,dV,minsup);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelMarkUniEdgeSatisfyMinsup in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		status = -1;
		goto Error;
	}

	printf("\n ***********dV**********\n");
	displayDeviceArr(dV,noElemUniEdge);

	int *dVScanResult=nullptr;
	CHECK(hipMalloc((void**)&dVScanResult,noElemUniEdge*sizeof(int)));
	//CHECK(scanV(dV,noElemUniEdge,dVScanResult));
	myScanV(dV,noElemUniEdge,dVScanResult);
	printf("\n ***********dVScanResult**********\n");
	displayDeviceArr(dVScanResult,noElemUniEdge);

	hUniEdgeSatisfyMinsup.resize(1);
	CHECK(getSizeBaseOnScanResult(dV,dVScanResult,noElemUniEdge,hUniEdgeSatisfyMinsup.at(0).noElem));
	CHECK(hipMalloc((void**)&hUniEdgeSatisfyMinsup.at(0).dUniEdge,hUniEdgeSatisfyMinsup.at(0).noElem*sizeof(UniEdge)));
	hUniEdgeSatisfyMinsup.at(0).hArrSup = (int*)malloc(sizeof(int)*hUniEdgeSatisfyMinsup.at(0).noElem);
	if (hUniEdgeSatisfyMinsup.at(0).hArrSup ==NULL){
		printf("\n malloc hArrSup of hUniEdgeSatisfyMinsup failed()");
		exit(1);
	}


	int *dSup=nullptr;
	CHECK(hipMalloc((void**)&dSup,hUniEdgeSatisfyMinsup.at(0).noElem*sizeof(int)));


	dim3 blocka(blocksize);
	dim3 grida((noElemUniEdge + blocka.x -1)/blocka.x);
	kernelExtractUniEdgeSatifyMinsup<<<grida,blocka>>>(hUniEdge.at(0).dUniEdge,dV,dVScanResult,noElemUniEdge,hUniEdgeSatisfyMinsup.at(0).dUniEdge,dSup,dResultSup);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelExtractUniEdgeSatisfyMinsup() in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		goto Error;
	}
	printf("\n ********hUniEdgeSatisfyMinsup.dUniEdge****************\n");
	displayArrUniEdge(hUniEdgeSatisfyMinsup.at(0).dUniEdge,hUniEdgeSatisfyMinsup.at(0).noElem);
	printf("\n ********hUniEdgeSatisfyMinsup.dSup****************\n");
	displayDeviceArr(dSup,hUniEdgeSatisfyMinsup.at(0).noElem);

	CHECK(hipMemcpy(hUniEdgeSatisfyMinsup.at(0).hArrSup,dSup,sizeof(int)*hUniEdgeSatisfyMinsup.at(0).noElem,hipMemcpyDeviceToHost));

	for (int i = 0; i < hUniEdgeSatisfyMinsup.at(0).noElem; i++)
	{
		printf("\n hArrSup:%d ",hUniEdgeSatisfyMinsup.at(0).hArrSup[i]);
	}

	hipFree(dResultSup);
	hipFree(dV);
	hipFree(dVScanResult);
	hipFree(dSup);
Error:
	return status;
}


int PMS::computeSupport(){
	int status=0;
	/* Xây dựng Boundary cho mảng d_ValidExtension */
	//1. Cấp phát một mảng d_B và gán các giá trị 0 cho mọi phần tử của d_B
	unsigned int noElement_dB=hValidExtension.at(0).noElem;
	int* dB;
	hipError_t cudaStatus=hipMalloc((int**)&dB,noElement_dB*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc dB in computeSupport() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dB,0,noElement_dB*sizeof(int)));
	}
	//printf("\n**********dValidExtension*************\n");
	//displayArrExtension(hValidExtension.at(0).dExtension,noElement_dB);
	//printf("\n*********dB********\n");
	//displayDeviceArr(dB,noElement_dB);


	//Gián giá trị boundary cho d_B
	cudaStatus=calcBoundary(hValidExtension.at(0).dExtension,noElement_dB,dB,maxOfVer);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"calcBoundary() in computeSupport() failed",cudaStatus);
		return 1;
	}

	printf("\n**********dValidExtension*************\n");
	displayArrExtension(hValidExtension.at(0).dExtension,noElement_dB);
	printf("\n*********dB********\n");
	displayDeviceArr(dB,noElement_dB);


	//2. Exclusive Scan mảng d_B
	int* dBScanResult;
	cudaStatus=hipMalloc((int**)&dBScanResult,noElement_dB*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc dBScanResult in computeSupport() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		hipMemset(dBScanResult,0,noElement_dB*sizeof(int));
	}

	//cudaStatus=scanV(dB,noElement_dB,dBScanResult);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\nscanB function failed",cudaStatus);
	//	status =-1;
	//	goto Error;
	//}
	myScanV(dB,noElement_dB,dBScanResult);
	printf("\n\n*******dBScanResult***********\n");
	displayDeviceArr(dBScanResult,noElement_dB);

	//3. Tính độ hỗ trợ cho các mở rộng trong d_UniqueExtension
	//3.1 Tạo mảng d_F có số lượng phần tử bằng với giá trị cuối cùng của mảng d_scanB_Result cộng 1 và gán giá trị 0 cho các phần tử.
	int noElemF=0;
	cudaStatus=getLastElement(dBScanResult,noElement_dB,noElemF);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ngetLastElement function failed",cudaStatus);
		return 1;
	}

	noElemF++;
	/*noElemGraphInExt=noElemF;*/

	printf("\n\n noElement_F:%d",noElemF);
	int noElem_d_UniqueExtension= hUniEdge.at(0).noElem;
	int *dF;
	cudaStatus=hipMalloc((int**)&dF,noElem_d_UniqueExtension*noElemF*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc dF in computeSupport() failed",cudaStatus);
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dF,0,noElem_d_UniqueExtension*noElemF*sizeof(int)));
	}
	int *hResultSup=nullptr;
	cudaStatus=calcSupport(hUniEdge.at(0).dUniEdge,hUniEdge.at(0).noElem,hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,dBScanResult,dF,noElemF,hResultSup);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n calcSupport() in computeSupport() failed",cudaStatus);
		status =-1;
		goto Error;
	}

	////In độ hỗ trợ cho các cạnh tương ứng trong mảng kết quả h_resultSup
	//for (int i = 0; i < noElem_d_UniqueExtension; i++)
	//{
	//	printf("\n resultSup[%d]:%d",i,hResultSup[i]);
	//}
	//
	extractUniEdgeSatisfyMinsup(hResultSup,noElem_d_UniqueExtension,minsup);

	hipFree(dBScanResult);
	hipFree(dB);
Error:
	return status;
}

__global__ void kernelGetGraphIdContainEmbedding(int li,int lij,int lj,Extension *d_ValidExtension,int noElem_d_ValidExtension,int *dV,unsigned int maxOfVer){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension){
		if(	d_ValidExtension[i].li == li && d_ValidExtension[i].lij == lij && 	d_ValidExtension[i].lj == lj){
			int graphid = (d_ValidExtension[i].vgi/maxOfVer);
			dV[graphid]=1;
		}
	}
}

__global__ void kernelGetLastElementExtension(Extension *inputArray,unsigned int noEleInputArray,int *value,unsigned int maxOfVer){
	value[0] = inputArray[noEleInputArray-1].vgi/maxOfVer; /*Lấy global vertex id chia cho tổng số đỉnh của đồ thị (maxOfVer). Ở đây các đồ thị luôn có số lượng đỉnh bằng nhau (maxOfVer) */
}

hipError_t getLastElementExtension(Extension* inputArray,unsigned int numberElementOfInputArray,int &outputValue,unsigned int maxOfVer){
	hipError_t cudaStatus;

	int *temp=nullptr;
	CHECK(hipMalloc((int**)&temp,sizeof(int)));
	//kernelPrintExtention<<<1,512>>>(inputArray,numberElementOfInputArray);
	//hipDeviceSynchronize();
	//cudaStatus= hipGetLastError();
	//if(cudaStatus != hipSuccess){
	//	fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
	//	goto Error;
	//}

	/* Lấy graphId chứa embedding cuối cùng */
	kernelGetLastElementExtension<<<1,1>>>(inputArray,numberElementOfInputArray,temp,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}

	CHECK(hipMemcpy(&outputValue,temp,sizeof(int),hipMemcpyDeviceToHost));
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);

	hipFree(temp);
Error:	
	return cudaStatus;	
}

__global__ void kernelGetGraph(int *dV,int noElemdV,int *d_kq,int *dVScanResult){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdV){
		if(dV[i]!=0){
			d_kq[dVScanResult[i]]=i;
		}
	}
}

__global__ void kernelGetLastElementEXT(EXT *inputArray,int noEleInputArray,int *value,unsigned int maxOfVer){
	*value = inputArray[noEleInputArray-1].vgi/maxOfVer; /*Lấy global vertex id chia cho tổng số đỉnh của đồ thị (maxOfVer). Ở đây các đồ thị luôn có số lượng đỉnh bằng nhau (maxOfVer) */
}

hipError_t getLastElementEXT(EXT *inputArray,int numberElementOfInputArray,int &outputValue,unsigned int maxOfVer){
	hipError_t cudaStatus;

	int *temp=nullptr;
	CHECK(cudaStatus=hipMalloc((int**)&temp,sizeof(int)));
	if(cudaStatus!=hipSuccess){
		goto Error;
	}

	/* Lấy graphId chứa embedding cuối cùng */
	kernelGetLastElementEXT<<<1,1>>>(inputArray,numberElementOfInputArray,temp,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}

	CHECK(hipMemcpy(&outputValue,temp,sizeof(int),hipMemcpyDeviceToHost));
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);

	hipFree(temp);
Error:	
	return cudaStatus;	
}

__global__ void kernelGetGraphIdContainEmbeddingv2(int li,int lij,int lj,EXT *d_ValidExtension,int noElem_d_ValidExtension,int *dV,unsigned int maxOfVer){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension){
		if(	d_ValidExtension[i].li == li && d_ValidExtension[i].lij == lij && d_ValidExtension[i].lj == lj){
			int graphid = (d_ValidExtension[i].vgi/maxOfVer);
			dV[graphid]=1;
		}
	}
}


int PMS::getGraphIdContainEmbeddingv2(UniEdge edge,int *&hArrGraphId,int &noElemhArrGraphId,EXT *dArrEXT,int noElemdArrEXT){
	int status =0;
	hipError_t cudaStatus;
	int li,lij,lj;
	li = edge.li;
	lij = edge.lij;
	lj = edge.lj;
	dim3 block(blocksize);
	dim3 grid((noElemdArrEXT+block.x-1)/block.x);

	int *dV=nullptr;
	int noElemdV=0;

	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

	CHECK(cudaStatus =getLastElementEXT(dArrEXT,noElemdArrEXT,noElemdV,maxOfVer));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	++noElemdV;

	CHECK(cudaStatus=hipMalloc((void**)&dV,noElemdV*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		fprintf(stderr,"\n hipMalloc dV in getGraphIdContainEmbedding() failed");
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dV,0,noElemdV*sizeof(int)));
	}

	kernelGetGraphIdContainEmbeddingv2<<<grid,block>>>(li,lij,lj,dArrEXT,noElemdArrEXT,dV,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n kernelGetGraphIdContainEmbedding() in getGraphIdContainEmbedding() failed",cudaStatus);
		goto Error;
	}

	int *dVScanResult=nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&dVScanResult,noElemdV*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		fprintf(stderr,"\n hipMalloc dVScanResult in getGraphIdContainEmbedding() failed");
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dVScanResult,0,noElemdV*sizeof(int)));
	}


	//scanV(dV,noElemdV,dVScanResult);
	myScanV(dV,noElemdV,dVScanResult);

	printf("\n ************* dVScanResult *************\n");
	displayDeviceArr(dVScanResult,noElemdV);
	int noElem_kq;	
	CHECK(cudaStatus=getLastElement(dVScanResult,noElemdV,noElem_kq));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	++noElem_kq;

	int *d_kq;
	CHECK(cudaStatus=hipMalloc((void**)&d_kq,sizeof(int)*noElem_kq));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	dim3 blocka(blocksize);
	dim3 grida((noElemdV + blocka.x -1)/blocka.x);

	kernelGetGraph<<<grida,blocka>>>(dV,noElemdV,d_kq,dVScanResult);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}


	printf("\n*********** d_kq ***********\n");
	displayDeviceArr(d_kq,noElem_kq);

	hArrGraphId=(int*)malloc(sizeof(int)*noElem_kq);
	if(hArrGraphId==NULL){
		printf("\nMalloc hArrGraphId in getGraphIdContainEmbedding() failed");
		exit(1);
	}
	noElemhArrGraphId=noElem_kq;

	CHECK(hipMemcpy(hArrGraphId,d_kq,sizeof(int)*noElem_kq,hipMemcpyDeviceToHost));

	hipFree(d_kq);
	hipFree(dV);
	hipFree(dVScanResult);
Error:
	return status;
}


int PMS::getGraphIdContainEmbedding(UniEdge edge,int *&hArrGraphId,int &noElemhArrGraphId){
	int status =0;
	int noElemdValidExtension = hExtension.at(0).noElem;

	int li,lij,lj;
	li = edge.li;
	lij = edge.lij;
	lj = edge.lj;
	dim3 block(blocksize);
	dim3 grid((noElemdValidExtension+block.x-1)/block.x);

	int *dV=nullptr;
	int noElemdV=0;

	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

	CHECK(getLastElementExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,noElemdV,maxOfVer));
	noElemdV++;

	hipError_t cudaStatus=hipMalloc((void**)&dV,noElemdV*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dV in getGraphIdContainEmbedding() failed");
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dV,0,noElemdV*sizeof(int)));
	}

	kernelGetGraphIdContainEmbedding<<<grid,block>>>(li,lij,lj,hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,dV,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n kernelGetGraphIdContainEmbedding() in getGraphIdContainEmbedding() failed",cudaStatus);
		goto Error;
	}

	int *dVScanResult=nullptr;
	cudaStatus=hipMalloc((void**)&dVScanResult,noElemdV*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dVScanResult in getGraphIdContainEmbedding() failed");
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dVScanResult,0,noElemdV*sizeof(int)));
	}


	//scanV(dV,noElemdV,dVScanResult);
	myScanV(dV,noElemdV,dVScanResult);
	printf("\n ************* dVScanResult *************\n");
	displayDeviceArr(dVScanResult,noElemdV);
	int noElem_kq;	
	CHECK(getLastElement(dVScanResult,noElemdV,noElem_kq));
	noElem_kq++;

	int *d_kq;
	hipMalloc((void**)&d_kq,sizeof(int)*noElem_kq);

	dim3 blocka(blocksize);
	dim3 grida((noElemdV + blocka.x -1)/blocka.x);

	kernelGetGraph<<<grida,blocka>>>(dV,noElemdV,d_kq,dVScanResult);
	hipDeviceSynchronize();

	printf("\n*********** d_kq ***********\n");
	displayDeviceArr(d_kq,noElem_kq);

	hArrGraphId=(int*)malloc(sizeof(int)*noElem_kq);
	if(hArrGraphId==NULL){
		printf("\nMalloc hArrGraphId in getGraphIdContainEmbedding() failed");
		exit(1);
	}
	noElemhArrGraphId=noElem_kq;

	CHECK(hipMemcpy(hArrGraphId,d_kq,sizeof(int)*noElem_kq,hipMemcpyDeviceToHost));

	hipFree(d_kq);
	hipFree(dV);
	hipFree(dVScanResult);
Error:
	return status;
}

hipError_t ADM(int *&devicePointer,size_t nBytes){
	hipError_t cudaStatus;
	cudaStatus= hipMalloc((void**)&devicePointer,nBytes);
	return cudaStatus;
}


int PMS::Mining(){
	int status = 0;
	hipError_t cudaStatus;
	int noElemtemp = hUniEdgeSatisfyMinsup.at(0).noElem;
	UniEdge *temp=(UniEdge*)malloc(sizeof(UniEdge)*noElemtemp);
	if(temp==NULL){
		printf("\n malloc temp failed");
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMemcpy(temp,hUniEdgeSatisfyMinsup.at(0).dUniEdge,noElemtemp*sizeof(UniEdge),hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	for (int i = 0; i < noElemtemp; i++) //Duyệt qua các UniEdge thoả minSup để kiểm tra minDFS_CODE, nếu thoả thì ghi kết quả vào result và xây dựng embedding
	{
		int li,lij,lj;
		li = temp[i].li;
		lij= temp[i].lij;
		lj=temp[i].lj;

		DFS_CODE.push(0,1,temp[i].li,temp[i].lij,temp[i].lj);//xây dựng DFS_CODE
		minLabel = temp[i].li;
		maxId = 1;

		if(is_min()){ //Nếu DFS_CODE là min thì tìm các graphid chứa embedding của DFS_CODE
			printf("\n This is minDFSCODE\n");

			int *hArrGraphId; //Mảng chứa các graphID có embedding của DFS_Code.
			int noElemhArrGraphId=0;
			/* Trước khi ghi kết quả thì phải biết đồ thị phổ biến đó tồn tại ở những graphId nào. Hàm getGraphIdContainEmbedding dùng để làm việc này
			* 3 tham số đầu tiên của hàm là nhãn cạnh của phần tử d_UniqueExtension đang xét */
			status =getGraphIdContainEmbedding(temp[i],hArrGraphId,noElemhArrGraphId);
			if (status!=0){
				printf("\n\n getGraphIdContainEmbedding() in Mining() failed");
				goto Error;
			}

			//In nội dung mảng hArrGraphId

			printf("\n ************** hArrGraphId ****************\n");
			for (int j = 0; j < noElemhArrGraphId; j++)
			{
				printf("%d ",hArrGraphId[j]);
			}

			/*	Ghi kết quả DFS_CODE vào file result.txt ************************************************************
			*	Hàm report sẽ chuyển DFS_CODE pattern sang dạng đồ thị, sau đó sẽ ghi đồ thị đó xuống file result.txt
			*	Hàm report gồm 3 tham số:
			*	Tham số thứ 1: mảng chứa danh sách các graphID chứa DFS_CODE pattern
			*	Tham số thứ 2: số lượng mảng
			*	Tham số thứ 3: độ hỗ trợ của DFS_CODE pattern *******************************************************/

			report(hArrGraphId,noElemhArrGraphId,hUniEdgeSatisfyMinsup.at(0).hArrSup[i]);
			//Giải phóng bộ nhớ 
			std::free(hArrGraphId);

			//Xây dựng Embedding cho DFS_Code rồi gọi hàm GraphMining để khai thác
			//Trong GraphMining sẽ gọi GraphMining khác để thực hiện khai thác đệ quy

			FUNCHECK(buildFirstEmbedding(temp[i])); //Xây dựng 2 cột embedding ban đầu.
			FUNCHECK(buildRMP()); //Xây dựng RMP ban đầu
			FUNCHECK(FSMining()); //Gọi FSMining.( Hàm này thực hiện theo tuần tự (1. Find Extension -> 2. Extract UniEdge -> 3.Compute & CHECK Support -> 4. CHECK minDFS_CODE -> 5. BuildEmbedding -> 6.Find RMP -> 1.)
		}
		//Giải phóng bộ nhớ
		if(hRMP.size()>0){
			for (int j = 0; j < hRMP.size(); j++)
			{
				hRMP.at(j).hArrRMP.clear();
			}
			hRMP.clear();
		}

		DFS_CODE.pop();
		if(hEmbedding.size()!=0){
			for (int j = 0; j < hEmbedding.size(); j++)
			{
				hipFree(hEmbedding.at(j).dArrEmbedding);
			}
			hEmbedding.clear();
		}
	}	
	std::free(temp);
Error:
	return status;
}

__global__ void	kernelGetvivj(EXT *dArrEXT,int noElemdArrEXT,int li,int lij,int lj,int *dvi,int *dvj){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdArrEXT){
		if(dArrEXT[i].li == li && dArrEXT[i].lij == lij && dArrEXT[i].lj == lj){
			*dvi=dArrEXT[i].vi;
			*dvj=dArrEXT[i].vj;
			printf("\n Thread:%d (dvi dvj):(%d %d)",i,*dvi,*dvj);
		}
	}
}

int displayDeviceEXT(EXT *dArrEXT,int noElemdArrEXT){
	int status =0;
	hipError_t cudaStatus;

	EXT *hArrEXT = (EXT*)malloc(sizeof(EXT)*noElemdArrEXT);
	if(hArrEXT == NULL){
		printf("\n malloc hArrEXT failed");
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus = hipMemcpy(hArrEXT,dArrEXT,noElemdArrEXT*sizeof(EXT),hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	printf("\n************** EXT **************\n");
	for (int i = 0; i < noElemdArrEXT; i++)
	{
		printf("\n (vi vj):(%d %d) (li lij lj):(%d %d %d) (vgi vgj):(%d %d) (RowPointer:%d)",hArrEXT[i].vi,hArrEXT[i].vj,hArrEXT[i].li,hArrEXT[i].lij,hArrEXT[i].lj,hArrEXT[i].vgi,hArrEXT[i].vgj,hArrEXT[i].posRow);
	}

	std::free(hArrEXT);
Error:
	return status;
}



int PMS::getvivj(EXT *dArrEXT,int noElemdArrEXT,int li,int lij,int lj,int &vi,int &vj){
	int status=0;
	hipError_t cudaStatus;

	int *dvi=nullptr;
	int *dvj=nullptr;
	size_t nBytesvi=sizeof(int);

	CHECK(cudaStatus = ADM(dvi,nBytesvi));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus = ADM(dvj,nBytesvi));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dvi,sizeof(int));
	//CHECK(cudaStatus);
	//if(cudaStatus !=hipSuccess){
	//	status =-1;
	//	goto Error;
	//}

	//cudaStatus = hipMalloc((void**)&dvj,sizeof(int));
	//CHECK(cudaStatus);
	//if(cudaStatus !=hipSuccess){
	//	status =-1;
	//	goto Error;
	//}
	dim3 block(blocksize);
	dim3 grid((noElemdArrEXT+block.x-1)/block.x);

	FUNCHECK(status =displayDeviceEXT(dArrEXT,noElemdArrEXT));
	if(status!=0){
		goto Error;
	}

	kernelGetvivj<<<grid,block>>>(dArrEXT,noElemdArrEXT,li,lij,lj,dvi,dvj);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMemcpy(&vi,dvi,nBytesvi,hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMemcpy(&vj,dvj,nBytesvi,hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}


	CHECK(cudaStatus = hipFree(dvi));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus =hipFree(dvj));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
Error:
	return status;
}

int PMS::updateRMP(){
	int status=0;
	hipError_t cudaStatus;

	//int cSize = hRMP.size();
	//int nSize = cSize +1;
	//hRMP.resize(nSize);
	//int lastIdx = nSize-1;

	hRMPv2.resize(Level);


	RMP *dRMP = nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&dRMP,sizeof(RMP)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	int noEC = hEmbedding.size(); //noEC is number of embedding column
	int lastIdxEC = noEC-1;

	for (int i = lastIdxEC; i != -1;)
	{
		hRMPv2.at(idxLevel).hArrRMP.push_back(i);
		i=hEmbedding.at(i).prevCol;		
	}
	hRMPv2.at(idxLevel).noElem = hRMPv2.at(idxLevel).hArrRMP.size();

	//In RMP
	for (int i = 0; i < hRMPv2.at(idxLevel).noElem; i++)
	{
		printf("\n RMPv2[%d]:%d",i,hRMPv2.at(idxLevel).hArrRMP.at(i));
	}

Error:
	return status;
}

//Đã tính độ hỗ trợ xong. Cần kiểm tra minDFS_code
int PMS::Miningv2(int noElem,UniEdge *dArrUniEdgeSatisfyMinSup,int *hArrSupport,EXT *dArrEXT,int noElemdArrEXT,int idxExt){
	int status = 0;
	hipError_t cudaStatus;
	//1.Tăng Level
	Level++;
	idxLevel=Level-1;

	int vi,vj,backward;
	vi=vj=-1;
	backward=0;
	int noElemtemp = noElem;
	UniEdge *temp=(UniEdge*)malloc(sizeof(UniEdge)*noElemtemp);
	if(temp==NULL){
		printf("\n malloc temp failed");
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMemcpy(temp,dArrUniEdgeSatisfyMinSup,noElemtemp*sizeof(UniEdge),hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	for (int i = 0; i < noElemtemp; i++) //Duyệt qua các UniEdge thoả minSup để kiểm tra minDFS_CODE, nếu thoả thì ghi kết quả vào result và xây dựng embedding
	{
		int li,lij,lj;
		li = temp[i].li;
		lij= temp[i].lij;
		lj=temp[i].lj;

		status = getvivj(dArrEXT,noElemdArrEXT,li,lij,lj,vi,vj);
		FUNCHECK(status);
		if(status!=0){
			goto Error;
		}
		if(vi>vj){
			backward=1;
		}

		//Nếu là mở rộng forward thì cập nhật lại maxId bằng vj;
		if(backward!=1){
			DFS_CODE.push(vi,vj,-1,temp[i].lij,temp[i].lj);//xây dựng DFS_CODE forward
			maxId=vj;
		}
		else
		{
			DFS_CODE.push(vi,vj,-1,temp[i].lij,-1);//xây dựng DFS_CODE backward
		}

		if(is_min()){ //Nếu DFS_CODE là min thì tìm các graphid chứa embedding của DFS_CODE
			printf("\n This is minDFSCODE\n");

			int *hArrGraphId; //Mảng chứa các graphID có embedding của DFS_Code.
			int noElemhArrGraphId=0;
			status =getGraphIdContainEmbeddingv2(temp[i],hArrGraphId,noElemhArrGraphId,dArrEXT,noElemdArrEXT);
			if (status!=0){
				printf("\n\n getGraphIdContainEmbedding() in Mining() failed");
				goto Error;
			}
			////In nội dung mảng hArrGraphId
			printf("\n ************** hArrGraphId ****************\n");
			for (int j = 0; j < noElemhArrGraphId; j++)
			{
				printf("%d ",hArrGraphId[j]);
			}

			report(hArrGraphId,noElemhArrGraphId,hArrSupport[i]);

			//Xây dựng Embedding cho DFS_Code rồi gọi hàm GraphMining để khai thác
			//Trong GraphMining sẽ gọi GraphMining khác để thực hiện khai thác đệ quy

			FUNCHECK(status=extendEmbedding(temp[i],idxExt));
			if(status!=0){
				goto Error;
			}

			//hLevelPtrEmbeddingv2.resize(Level);
			//hLevelPtrEmbeddingv2.at(idxLevel).noElem=hEmbedding.size();
			//int lastCol = hEmbedding.size()-1;
			//hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding=hEmbedding.at(lastCol).noElem;
			//CHECK(cudaStatus = hipMalloc((void**)&hLevelPtrEmbeddingv2.at(idxLevel).dArrPointerEmbedding,hLevelPtrEmbeddingv2.at(idxLevel).noElem*sizeof(Embedding**))); //Cấp phát bộ nhớ cho mảng dArrPointerEmbedding.
			//if(cudaStatus!=hipSuccess){
			//	status = -1;
			//	std::printf("\n hipMalloc dArrPointerEmbedding failed()");
			//	goto Error;
			//}
			//for (int i = 0; i < hEmbedding.size(); i++)
			//{		
			//	kernelGetPointerdArrEmbedding<<<1,1>>>(hEmbedding.at(i).dArrEmbedding,hLevelPtrEmbeddingv2.at(idxLevel).dArrPointerEmbedding,i); //Mỗi phần tử của mảng dArrPointerEmbedding chứa địa chỉ của dArrEmbedding
			//}
			//hipDeviceSynchronize();
			//cudaStatus = hipGetLastError();
			//CHECK(cudaStatus);
			//if(cudaStatus!=hipSuccess){
			//	status = -1;
			//	printf("\n kernelGetPointerdArrEmbedding failed");
			//	goto Error;
			//}

			//FUNCHECK(status = updateRMP());
			//if(status!=0){
			//	goto Error;
			//}

			FUNCHECK(FSMiningv2()); //Gọi FSMining.( Hàm này thực hiện theo tuần tự (1. Find Extension -> 2. Extract UniEdge -> 3.Compute & CHECK Support -> 4. CHECK minDFS_CODE -> 5. BuildEmbedding -> 6.Find RMP -> 1.)

			//Giải phóng bộ nhớ 
			std::free(hArrGraphId);

		}
		DFS_CODE.pop(); //Xoá phần tử cuối của DFS_CODE
		if(backward!=1){ //Nếu pop() một forward thì phải giảm maxId
			--maxId;
		}

		int lastCol = hRMPv2.at(idxLevel).hArrRMP[0];

		hipFree(hEmbedding.at(lastCol).dArrEmbedding); //xoá phần tử cuối của Embedding
		hEmbedding.pop_back();

		hipFree(hLevelPtrEmbeddingv2.at(idxLevel).dArrPointerEmbedding); //xoá dArrPointerEmbedding tại Level đang xét
		hLevelPtrEmbeddingv2.pop_back(); //Xoá phần tử cuối của Level pointerEmbeeding đang xét.

		hRMPv2.at(idxLevel).hArrRMP.clear(); //Xoá  RightMostPath của phần tử Embedding tại Level tương ứng.
		hRMPv2.pop_back();
	}	
	std::free(temp);
Error:
	return status;
}



__global__ void kernelMarkExtension(const Extension *d_ValidExtension,int noElem_d_ValidExtension,int *dV,int li,int lij,int lj){
	int i= blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			dV[i]=1;
		}		
	}
}

__global__ void kernelMarkEXT(const EXT *d_ValidExtension,int noElem_d_ValidExtension,int *dV,int li,int lij,int lj){
	int i= blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			dV[i]=1;
		}		
	}
}


__global__ void kernelSetValueForFirstTwoEmbeddingColumn(const Extension *d_ValidExtension,int noElem_d_ValidExtension,Embedding *dQ1,Embedding *dQ2,int *d_scanResult,int li,int lij,int lj){
	int i = blockDim.x *blockIdx.x +threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij == lij && d_ValidExtension[i].lj==lj){
			dQ1[d_scanResult[i]].idx=-1;
			dQ1[d_scanResult[i]].vid=d_ValidExtension[i].vgi;


			dQ2[d_scanResult[i]].idx=d_scanResult[i];
			dQ2[d_scanResult[i]].vid=d_ValidExtension[i].vgj;
		}
	}
}



__global__ void	kernelPrintEmbedding(Embedding *dArrEmbedding,int noElem){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("\n Thread:%d address:%p (idx vid):(%d %d)",i,dArrEmbedding,dArrEmbedding[i].idx,dArrEmbedding[i].vid);
	}
}

__global__ void kernelSetValueForEmbeddingColumn(EXT *dArrExt,int noElemInArrExt,Embedding *dArrQ,int *dM,int *dMScanResult){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElemInArrExt){		
		if(dM[i]==1){

			int posRow = dArrExt[i].posRow;
			int vgj =dArrExt[i].vgj;
			dArrQ[dMScanResult[i]].idx=posRow;
			dArrQ[dMScanResult[i]].vid=vgj;
		}
	}
}


int PMS::extendEmbedding(UniEdge ue,int idxExt){

	int li,lij,lj;
	li=ue.li;
	lij=ue.lij;
	lj=ue.lj;
	int status =0;
	hipError_t cudaStatus;

	int currentSize= hEmbedding.size();
	int newSize = currentSize+1;
	int lastEC =newSize-1; //lastEC is last Embedding Column or index of last element hEmbedding vector.

	hEmbedding.resize(newSize); //Mỗi phần tử của Vector sẽ quản lý 1 dArrEmbedding trên device. Khi cần thiết có thể tập hợp chúng lại thành 1 mảng trên device.
	//hEmbedding.at(0).noElem;

	int *dV=nullptr;
	int noElemdV = hLevelEXT.at(idxLevel).vE.at(idxExt).noElem;
	CHECK(cudaStatus=hipMalloc((void**)&dV, sizeof(int)*noElemdV));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMemset(dV,0,sizeof(int)*noElemdV));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	displayDeviceEXT( hLevelEXT.at(idxLevel).vE.at(idxExt).dArrExt, hLevelEXT.at(idxLevel).vE.at(idxExt).noElem);

	dim3 block(blocksize);
	dim3 grid((noElemdV+block.x-1)/block.x);


	kernelMarkEXT<<<grid,block>>>(hLevelEXT.at(idxLevel).vE.at(idxExt).dArrExt,noElemdV,dV,li,lij,lj);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelMarkExtension failed",cudaStatus);
		goto Error;
	}

	int* dVScanResult;
	CHECK(cudaStatus=hipMalloc((int**)&dVScanResult,noElemdV*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMemset(dVScanResult,0,noElemdV*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	//CHECK(scanV(dV,noElemdV,dVScanResult));
	myScanV(dV,noElemdV,dVScanResult);


	int noElemOfdArEmbedding=0;
	CHECK(getSizeBaseOnScanResult(dV,dVScanResult,noElemdV,noElemOfdArEmbedding));
	hEmbedding.at(lastEC).noElem=noElemOfdArEmbedding;

	CHECK(cudaStatus=hipMalloc((void**)&hEmbedding.at(lastEC).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	kernelSetValueForEmbeddingColumn<<<grid,block>>>(hLevelEXT.at(idxLevel).vE.at(idxExt).dArrExt,hLevelEXT.at(idxLevel).vE.at(idxExt).noElem,hEmbedding.at(lastEC).dArrEmbedding,dV,dVScanResult);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus !=hipSuccess){
		fprintf(stderr,"\n kernelSetValueForEmbeddingColumn in failed",cudaStatus);
		status = -1;
		goto Error;
	}

	hEmbedding.at(lastEC).prevCol=currentColEmbedding; 

	for (int i = 0; i < hEmbedding.size(); i++)
	{
		printf("\n\n Q[%d] prevCol:%d ",i,hEmbedding.at(i).prevCol);		
		kernelPrintEmbedding<<<1,512>>>(hEmbedding.at(i).dArrEmbedding,hEmbedding.at(i).noElem);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		CHECK(cudaStatus);
		if(cudaStatus!=hipSuccess){
			status =-1;
			printf("kernelPrintEmbedding failed");
			goto Error;
		}
	}
Error:
	return status;
}

//Xây dựng Embedding ban đầu
int PMS::buildFirstEmbedding(UniEdge ue){
	int li,lij,lj;
	li=ue.li;
	lij=ue.lij;
	lj=ue.lj;
	int status =0;
	hipError_t cudaStatus;
	hEmbedding.resize(2); //Mỗi phần tử của Vector sẽ quản lý 1 dArrEmbedding trên device. Khi cần thiết có thể tập hợp chúng lại thành 1 mảng trên device.
	hEmbedding.at(0).noElem;

	int *dV=nullptr;
	int noElemdV = hValidExtension.at(0).noElem;
	CHECK(hipMalloc((void**)&dV, sizeof(int)*noElemdV));
	CHECK(hipMemset(dV,0,sizeof(int)*noElemdV));
	dim3 block(blocksize);
	dim3 grid((noElemdV+block.x-1)/block.x);

	//kernelPrintExtention<<<1,512>>>(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);
	//hipDeviceSynchronize();
	//CHECK(hipGetLastError());
	//if(hipGetLastError() !=hipSuccess){
	//	printf("Error here");
	//	goto Error;
	//}

	kernelMarkExtension<<<grid,block>>>(hValidExtension.at(0).dExtension,noElemdV,dV,li,lij,lj);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelMarkExtension failed",cudaStatus);
		goto Error;
	}

	int* dVScanResult;
	CHECK(hipMalloc((int**)&dVScanResult,noElemdV*sizeof(int)));
	CHECK(hipMemset(dVScanResult,0,noElemdV*sizeof(int)));

	//CHECK(scanV(dV,noElemdV,dVScanResult));
	myScanV(dV,noElemdV,dVScanResult);


	int noElemOfdArEmbedding=0;
	CHECK(getSizeBaseOnScanResult(dV,dVScanResult,noElemdV,noElemOfdArEmbedding));
	hEmbedding.at(0).noElem=hEmbedding.at(1).noElem=noElemOfdArEmbedding;

	CHECK(hipMalloc((void**)&hEmbedding.at(0).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));
	CHECK(hipMalloc((void**)&hEmbedding.at(1).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));


	kernelSetValueForFirstTwoEmbeddingColumn<<<grid,block>>>(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,hEmbedding.at(0).dArrEmbedding,hEmbedding.at(1).dArrEmbedding,dVScanResult,li,lij,lj);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus !=hipSuccess){
		fprintf(stderr,"\n kernelSetValueForFirstTwoEmbeddingColumn in failed",cudaStatus);
		status = -1;
		goto Error;
	}

	hEmbedding.at(0).prevCol=-1;
	hEmbedding.at(1).prevCol=0;

	for (int i = 0; i < hEmbedding.size(); i++)
	{
		printf("\n\n Q[%d] prevCol:%d ",i,hEmbedding.at(i).prevCol);		
		kernelPrintEmbedding<<<1,512>>>(hEmbedding.at(i).dArrEmbedding,hEmbedding.at(i).noElem);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		CHECK(cudaStatus);
		if(cudaStatus!=hipSuccess){
			status =-1;
			printf("kernelPrintEmbedding failed");
			goto Error;
		}
	}


Error:
	return status;
}

//Why do this snippet face the error: Invalid device pointer

//void PMS::displayEmbeddingColumn(EmbeddingColumn ec){
//	printf("\n noElem:%d prevCol:%d",ec.noElem,ec.prevCol);
//	
//	Embedding *hArrEmbeddingt = (Embedding*)malloc(sizeof(Embedding)*ec.noElem);
//	if(hArrEmbeddingt==NULL){
//		printf("\n malloc hArrEmbeddingt in displayEmbeddingColumn() failed");
//		exit(1);
//	}
//
//	CHECK(hipMemcpy(hArrEmbeddingt,ec.dArrEmbedding,sizeof(Embedding)*ec.noElem,hipMemcpyDeviceToHost));
//	for (int i = 0; i < ec.noElem; i++)
//	{
//		printf("\n A[%d]: (idx, vid):(%d, %d)",i,hArrEmbeddingt[i].idx,hArrEmbeddingt[i].vid);
//	}
//
//	hipFree(hArrEmbeddingt);
//}

__global__ void kernelFindVidOnRMP(Embedding **dArrPointerEmbedding,int noElemEmbedding,int *rmp,int noElemVerOnRMP,int *dArrVidOnRMP,int step){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemEmbedding){	
		int index;
		int start=0;
		int prevRow = i;
		int end=0;
		Embedding * dArrEmbedding;
		for (int k = 0; k < noElemVerOnRMP; )
		{
			index = i*step+k;
			int j;
			start = rmp[k];
			k++;
			if(k==noElemVerOnRMP) break;
			end = rmp[k];
			//Từ cột start sẽ trích ra được vid và prevRow;
			for (j = start; j >end; j--)
			{
				dArrEmbedding = dArrPointerEmbedding[j];
				prevRow= dArrEmbedding[prevRow].idx; //update row
			}
			dArrEmbedding = dArrPointerEmbedding[j];
			dArrVidOnRMP[index]=dArrEmbedding[prevRow].vid;
			prevRow= dArrEmbedding[prevRow].idx; //update row
			printf("\n thread:%d start:%d end:%d index:%d vid:%d",i,start,end,index,dArrVidOnRMP[index]);
		}

	}
}


__global__ void kernelFindListVer(Embedding **dArrPointerEmbedding,int noElemEmbedding,int *rmp,int noElemVerOnRMP,int *listVer){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemEmbedding){	
		int j =0;
		int posCol = rmp[j]; //bắt đầu từ cột cuối của Embedding
		int posRow = i;
		Embedding *dArrEmbedding = dArrPointerEmbedding[posCol];
		int idxListVer = j*noElemEmbedding + i;
		listVer[idxListVer] = dArrEmbedding[posRow].vid; //Trích vid lưu vào mảng listVer tại vị trí tương ứng.
loop:
		j=j+1; //tăng chỉ số j của rmp
		if(j==noElemVerOnRMP) return;
		int loopTimes = posCol - rmp[j];
		for (int k = 0; k < loopTimes; k++)
		{
			posRow = dArrEmbedding[posRow].idx;
			posCol = posCol-1;
			dArrEmbedding = dArrPointerEmbedding[posCol];
			//printf("\nThread %d j:%d k:%d posCol:%d posRow:%d",i,j,k,posCol,posRow);
		}
		idxListVer = j*noElemEmbedding + i;
		listVer[idxListVer] = dArrEmbedding[posRow].vid; //Trích vid lưu vào mảng listVer tại vị trí tương ứng.
		//printf("\n Thread %d j:%d vid:%d idxListVer:%d posCol:%e posRow:%d",i,j,dArrEmbedding[posRow].vid, idxListVer,posCol,posRow);
		goto loop;
	}
}

__global__ void kernelDisplaydArrPointerEmbedding(Embedding **dArrPointerEmbedding,int noElemEmbeddingCol,int noElemEmbedding){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemEmbedding){
		int lastCol = noElemEmbeddingCol-1;
		Embedding *dArrEmbedding;
		//printf("\n Last Embedding column:%d Element:%d (idx vid):(%d %d)",lastCol,i,dArrEmbedding[i].idx,dArrEmbedding[i].vid);
		int prevRow=i;
		for (int j = lastCol; j>=0; j--)
		{
			dArrEmbedding= dArrPointerEmbedding[j];
			printf("\n Last Embedding column:%d Element:%d (idx vid):(%d %d)",lastCol,i,dArrEmbedding[prevRow].idx,dArrEmbedding[prevRow].vid);
			prevRow=dArrEmbedding[prevRow].idx;
		}
	}
}


int PMS::displaydArrPointerEmbedding(Embedding **dArrPointerEmbedding,int noElemEmbeddingCol,int noElemEmbedding){
	int status =0;
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElemEmbedding + block.x - 1)/block.x);
	printf("\n************ Embedding dArrPointerEmbedding ************\n");
	kernelDisplaydArrPointerEmbedding<<<grid,block>>>(dArrPointerEmbedding,noElemEmbeddingCol,noElemEmbedding);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
Error:
	return status;
}



int PMS::buildRMP(){
	int status = 0;
	hRMP.resize(1);
	int noElem = hEmbedding.size();
	int index = noElem - 1;

	for (int i = index ; i != -1 ; i = hEmbedding.at(i).prevCol)
	{
		hRMP.at(0).hArrRMP.push_back(i);		
	}
	hRMP.at(0).noElem = 2;
Error:
	return status;
}
//Hàm này thực hiện theo tuần tự (1. Find Extension -> 2. Extract UniEdge -> 3.Compute & CHECK Support -> 4. CHECK minDFS_CODE -> 5. BuildEmbedding -> 6.Find RMP -> Miningv2
int PMS::FSMining()
{
	int status = 0;
	hipError_t cudaStatus;

	//Thiết lập điều kiện dừng (return) khi không tồn tại mở rộng
	/* somethings code here */
	
	//Level++; //Tạm thời chưa cần đến Level
	//idxLevel=Level-1;

	//1. Tìm mở rộng từ các đỉnh thuộc right most path của các embedding
	//2. Trích ra các mở rộng hợp lệ và lưu chúng vào EXTk tương ứng
	//3. Duyệt qua các EXTk trích các mở rộng duy nhất
	//4. Tính độ hỗ trợ cho các mở rộng duy nhất trong EXTk
	//5. Loại bỏ những mở rộng không thoả mãn độ hỗ trợ do người dùng chỉ định
	//6. Kiểm tra minDFS_CODE --> ghi nhận kết quả và tiếp bước 7
	//7. Mở rộng Embedding cho các DFS_CODE thoả minSup
	//8. Lặp lại bước 1.

	//1. Để tìm các mở rộng cho các Embedding từ các đỉnh thuộc RMP, thì chúng ta cần xây dựng Embedding trên device
	//Hàm kernelGetPointerdArrEmbedding giúp chúng ta xây dựng mảng dArrPointerEmbedding chứa các pointer của các mảng dArrEmbedding trên device hiện
	//đang được quản lý bởi các phần tử của vector hEmbedding ở bộ nhớ host.
	//Kernel có thể đọc dữ liệu trực tiếp từ mảng dArrPointerEmbedding.
	hLevelPtrEmbedding.resize(1);
	hLevelPtrEmbedding.at(0).noElem=hEmbedding.size();
#pragma region "build dArrPointerEmbedding on device"
	cudaStatus = hipMalloc((void**)&hLevelPtrEmbedding.at(0).dArrPointerEmbedding,hEmbedding.size()*sizeof(Embedding**)); //Cấp phát bộ nhớ cho mảng dArrPointerEmbedding tại Level tương ứng.
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		std::printf("\n hipMalloc dArrPointerEmbedding failed()");
		goto Error;
	}

	for (int i = 0; i < hEmbedding.size(); i++)
	{		
		kernelGetPointerdArrEmbedding<<<1,1>>>(hEmbedding.at(i).dArrEmbedding,hLevelPtrEmbedding.at(0).dArrPointerEmbedding,i); //Mỗi phần tử của mảng dArrPointerEmbedding chứa địa chỉ của dArrEmbedding
	}
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		printf("\n kernelGetPointerdArrEmbedding failed");
		goto Error;
	}
#pragma endregion 


#pragma region "hipMalloc for listVer to find listVer On All EmbeddingColumn that belong to RMP"

	//Tìm danh sách các đỉnh thuộc right most path của các embedding
	//Kết quả lưu vào các vector tương ứng
	int lastCol = hEmbedding.size() - 1; //cột cuối của embedding
	hLevelPtrEmbedding.at(0).noElemEmbedding= hEmbedding.at(lastCol).noElem; //số lượng embedding	
	int noElemListVer= hRMP.at(0).noElem * hLevelPtrEmbedding.at(0).noElemEmbedding; //số lượng phần tử của listVer bằng số lượng đỉnh trên right most path nhân với số lượng embedding
	hListVer.resize(1);
	hListVer.at(0).noElem=noElemListVer;
	CHECK(cudaStatus = hipMalloc((void**)&hListVer.at(0).dListVer,sizeof(int)*noElemListVer)); //cấp phát bộ nhớ cho listVer
	if(cudaStatus!=hipSuccess){
		printf("\n CudaMalloc dListVer failed");
		status =-1;
		goto Error;
	}

#pragma endregion

	FUNCHECK(status=displaydArrPointerEmbedding(hLevelPtrEmbedding.at(0).dArrPointerEmbedding,hLevelPtrEmbedding.at(0).noElem,hLevelPtrEmbedding.at(0).noElemEmbedding));
	if(status!=0){
		goto Error;
	}


#pragma region "build RMP on device"

	//Xây dựng right most path từ vector<int> hRMP
	int noElemVerOnRMP = hRMP.at(0).noElem; //right most path chứa bao nhiêu đỉnh
	int *rmp = nullptr; //rigt most path trên bộ nhớ device
	CHECK(cudaStatus = hipMalloc((void**)&rmp,noElemVerOnRMP*sizeof(int))); //cấp phát bộ nhớ trên device cho rmp
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	int *temp=(int*)malloc(sizeof(int)*noElemVerOnRMP); //dùng để chứa dữ liệu từ vector hRMP
	if(temp==NULL){
		status =-1;
		FUNCHECK(status);
		goto Error;
	}
	//chép dữ liệu từ hRMP sang bộ nhớ temp
	for (int i = 0; i < noElemVerOnRMP; i++)
	{
		temp[i] = hRMP.at(0).hArrRMP.at(i);
	}
	//Chép dữ liệu từ temp trên host sang rmp trên device
	CHECK(cudaStatus =hipMemcpy(rmp,temp,sizeof(int)*noElemVerOnRMP,hipMemcpyHostToDevice)); //chép dữ liệu từ temp ở host sang rmp trên device
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	std::free(temp);

	printf("\n\n ******* rmp *********\n");
	displayDeviceArr(rmp,noElemVerOnRMP);

#pragma endregion

#pragma region "find listVer from All EmbeddingColumn"

	//Tìm danh sách các đỉnh thuộc right most path ở các cột embedding để thực hiện mở rộng
	dim3 block(blocksize);
	dim3 grid((hLevelPtrEmbedding.at(0).noElemEmbedding + block.x -1)/block.x);

	kernelFindListVer<<<block,grid>>>(hLevelPtrEmbedding.at(0).dArrPointerEmbedding,hLevelPtrEmbedding.at(0).noElemEmbedding,rmp,noElemVerOnRMP,hListVer.at(0).dListVer); //tìm listVer
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		status =-1;
		CHECK(cudaStatus);
		goto Error;
	}
	//hiển thị danh sách đỉnh
	printf("\n\n ********* listVer *********\n");
	displayDeviceArr(hListVer.at(0).dListVer,noElemListVer);

#pragma endregion


	//dArrVidOnRMP: chứa các đỉnh thuộc RMP của mỗi Embedding. Dùng để kiểm tra sự tồn tại của đỉnh trên right most path
	//khi tìm các mở rộng backward. Chỉ dùng tới khi right most path có 3 đỉnh trở lên (chỉ xét trong đơn đồ thị vô hướng).
	//int *dArrVidOnRMP = nullptr; //lưu trữ các đỉnh trên RMP của Embedding, có kích thước nhỏ hơn 2 đỉnh so với RMP
	//int noElemdArrVidOnRMP= hRMP.at(0).noElem - 2;
	//int *fromPosCol=nullptr; //lưu trữ các cột của Embedding mà tại đó thuộc right most path. Thật ra mình có thể suy luận được từ rmp
	//if (noElemdArrVidOnRMP >0){
	//	cudaStatus = hipMalloc((void**)&dArrVidOnRMP,hLevelPtrEmbedding.at(idxLevel).noElemEmbedding*noElemdArrVidOnRMP*sizeof(int));
	//	CHECK(cudaStatus);
	//	if(cudaStatus!=hipSuccess){
	//		status =-1;
	//		goto Error;
	//	}
	//	//cudaStatus = hipMalloc((void**)&fromPosCol,hLevelPtrEmbedding.at(idxLevel).noElemEmbedding*noElemdArrVidOnRMP*sizeof(int));
	//	//CHECK(cudaStatus);
	//	//if(cudaStatus!=hipSuccess){
	//	//	status =-1;
	//	//	goto Error;
	//	//}
	//}
	//if(hRMP.at(0).noElem>2){ //Nếu số lượng đỉnh trên RMP lớn hơn 2 thì mới tồn tại backward. Vì ở đây chỉ xét đơn đồ thị vô hướng
	//	FUNCHECK(status = displaydArrPointerEmbedding(hLevelPtrEmbedding.at(idxLevel).dArrPointerEmbedding,hLevelPtrEmbedding.at(idxLevel).noElem,hLevelPtrEmbedding.at(idxLevel).noElemEmbedding));
	//	if(status!=0){
	//		goto Error;
	//	}
	//	kernelFindVidOnRMP<<<grid,block>>>(hLevelPtrEmbedding.at(idxLevel).dArrPointerEmbedding,hLevelPtrEmbedding.at(idxLevel).noElemEmbedding,rmp,noElemVerOnRMP,dArrVidOnRMP,noElemdArrVidOnRMP);
	//	hipDeviceSynchronize();
	//	cudaStatus = hipGetLastError();
	//	CHECK(cudaStatus);
	//	if(cudaStatus!=hipSuccess){
	//		status =-1;
	//		goto Error;
	//	}
	//	printf("\n ******** dArrVidOnRMP *******\n");
	//	FUNCHECK(status = displayDeviceArr(dArrVidOnRMP,noElemdArrVidOnRMP*hLevelPtrEmbedding.at(idxLevel).noElemEmbedding));
	//	if(status!=0){
	//		goto Error;
	//	}
	//	//printf("\n ******** fromPosCol *******\n");
	//	//displayDeviceArr(fromPosCol,noElemdArrVidOnRMP);
	//}


	int noElemEXTk =noElemVerOnRMP; //Số lượng phần tử EXTk bằng số lượng đỉnh trên right most path
	//hEXTk.resize(noElemEXTk); //Các mở rộng hợp lệ từ đỉnh k sẽ được lưu trữ vào EXTk tương ứng.

	//Quản lý theo Level phục vụ cho khai thác đệ quy
	hLevelEXT.resize(1); //Khởi tạo vector quản lý bộ nhớ cho level
	hLevelEXT.at(0).noElem = noElemVerOnRMP; //Cập nhật số lượng phần tử vector vE bằng số lượng đỉnh trên RMP
	hLevelEXT.at(0).vE.resize(noElemVerOnRMP); //Cấp phát bộ nhớ cho vector vE.

	hLevelUniEdge.resize(1); //Số lượng phần tử UniEdge cũng giống với EXT
	hLevelUniEdge.at(0).noElem=noElemVerOnRMP;
	hLevelUniEdge.at(0).vUE.resize(noElemVerOnRMP);

	hLevelUniEdgeSatisfyMinsup.resize(1);
	hLevelUniEdgeSatisfyMinsup.at(0).noElem= noElemVerOnRMP;
	hLevelUniEdgeSatisfyMinsup.at(0).vecUES.resize(noElemVerOnRMP);

	int *tempListVerCol = nullptr; //chứa danh sách các đỉnh cần mở rộng thuộc một embedding column cụ thể.
	CHECK(cudaStatus = hipMalloc((void**)&tempListVerCol,hLevelPtrEmbedding.at(0).noElemEmbedding * sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	//Nếu số lượng phần tử đỉnh thuộc RMP bằng 2 thì không tồn tại mở rộng backward, nên chúng ta
	if(noElemVerOnRMP == 2){//chỉ tìm các ở rộng forward từ tập đỉnh và lưu kết quả của các mở rộng vào EXTk tương ứng.
		for (int i = 0; i < noElemVerOnRMP ; i++)
		{
			int colEmbedding = hRMP.at(0).hArrRMP.at(i); //Tìm mở rộng cho các đỉnh tại vị trí colEmbedding trong vector hEmbedding
			currentColEmbedding=colEmbedding; //Đang mở rộng từ cột nào của Embedding. Được dùng để cập nhật prevCol, phục vụ cho việc xây dựng Right Most Path
			int k = i; //lưu vào Extk với k = i; K=0 đại diện cho EXT0: là EXT cuối
			kernelExtractFromListVer<<<grid,block>>>(hListVer.at(0).dListVer,i*hLevelPtrEmbedding.at(0).noElemEmbedding,hLevelPtrEmbedding.at(0).noElemEmbedding,tempListVerCol);//trích các đỉnh từ listVer, từ vị trí i*noElemEmbedding,trích noElemEmbedding phần tử, bỏ vào tempListVerCol
			hipDeviceSynchronize();
			cudaStatus = hipGetLastError();
			CHECK(cudaStatus);
			printf("\n ****** tempListVerCol ***********\n");
			displayDeviceArr(tempListVerCol,hLevelPtrEmbedding.at(0).noElemEmbedding);


			//gọi hàm forwardExtension để tìm các mở rộng forward từ cột colEmbedding, lưu kết quả vào hEXTk tại vị trí k, với các đỉnh
			// cần mở rộng là tempListVerCol, thuộc righ most path
			////Hàm này cũng đồng thời trích các mở rộng duy nhất từ các EXT và lưu vào UniEdge
			//Hàm này cũng gọi đệ quy FSMining bên trong
			FUNCHECK(status = forwardExtension(k,tempListVerCol,hLevelPtrEmbedding.at(0).noElemEmbedding,hRMP.at(0).hArrRMP.at(i)));
			if(status ==-1){
				goto Error;
			}


			//FSMining();
			if(hLevelEXT.at(0).vE.at(i).noElem>0){ //Nếu số lượng phần tử của mảng dArrExt = 0 thì chúng ta không giải phóng bộ nhớ dArrExt vì nó chưa được cấp phát.
				hipFree(hLevelEXT.at(0).vE.at(i).dArrExt);
				hipFree(hLevelUniEdge.at(0).vUE.at(i).dArrUniEdge);
			}			
		}
		hLevelEXT.at(0).vE.clear();
		hLevelUniEdge.at(0).vUE.clear();
		
	}
	//Nếu số lượng đỉnh thuộc RMP nhiều hơn 2 thì sẽ tồn tại mở rộng backward
	if (noElemVerOnRMP > 2){
		//1. khai thác backward và forward của đỉnh cuối cùng trước
		for (int i = 1; i < noElemVerOnRMP-1; i++)
		{
			//2. sau đó khai thác forward cho các đỉnh còn lại
			//kernelFindValidFBExtension(dArrPointerEmbedding,hEmbedding.size(),noElemEmbedding,hdb.at(0).dO,hdb.at(0).dLO,hdb.at(0).dN,hdb.at(0).dLN,dArrDegreeOfVid,maxDegreeOfVer,dArrV,dArrExtension,listOfVer,minLabel,maxId,fromRMP,dArrVidOnRMP,noElemdArrVidOnRMP,fromPosCol);
		}

	}

	hipFree(tempListVerCol);
	hipFree(hListVer.at(0).dListVer);
Error:
	return status;
}

int PMS::buildArrPointerEmbedding(vector<EmbeddingColumn> hEmbedding,vector<ptrArrEmbedding>& hLevelPtrEmbedding){
	int status = 0;
	hipError_t cudaStatus;
	//3.1. Cấp phát bộ nhớ cho dArrPointerEmbedding và vector hLevelPtrEmbedding để lưu kết quả
	int lastCol = hEmbedding.size() - 1; //cột cuối của embedding
	hLevelPtrEmbedding.resize(Level);
	hLevelPtrEmbedding.at(idxLevel).noElem=hEmbedding.size(); //số lượng embedding column
	hLevelPtrEmbedding.at(idxLevel).noElemEmbedding= hEmbedding.at(lastCol).noElem; //số lượng embedding
	cudaStatus = hipMalloc((void**)&hLevelPtrEmbedding.at(idxLevel).dArrPointerEmbedding,hEmbedding.size()*sizeof(Embedding**)); //Cấp phát bộ nhớ cho mảng dArrPointerEmbedding tại Level tương ứng.
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		std::printf("\n hipMalloc dArrPointerEmbedding failed()");
		goto Error;
	}
	//3.2 Xây dựng mảng dArrPointerEmbedding chứa địa chỉ của các embedding column trên device
	for (int i = 0; i < hEmbedding.size(); i++)
	{		
		kernelGetPointerdArrEmbedding<<<1,1>>>(hEmbedding.at(i).dArrEmbedding,hLevelPtrEmbedding.at(idxLevel).dArrPointerEmbedding,i); //Mỗi phần tử của mảng dArrPointerEmbedding chứa địa chỉ của dArrEmbedding
	}
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		printf("\n kernelGetPointerdArrEmbedding failed");
		goto Error;
	}

Error:
	return status;
}

int PMS::buildrmpOnDevice(RMP hRMPatALevel,int *&rmp){
	int status = 0;
	hipError_t cudaStatus;
	//cần có rmp trên device
	int noElemVerOnRMP= hRMPatALevel.noElem;
	CHECK(cudaStatus = hipMalloc((void**)&rmp,noElemVerOnRMP*sizeof(int))); //cấp phát bộ nhớ trên device cho rmp
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	int *temp=(int*)malloc(sizeof(int)*noElemVerOnRMP); //dùng để chứa dữ liệu từ vector hRMP
	if(temp==NULL){
		status =-1;
		FUNCHECK(status);
		goto Error;
	}
	//chép dữ liệu từ hRMP sang bộ nhớ temp
	for (int i = 0; i < noElemVerOnRMP; i++)
	{
		temp[i] = hRMPv2.at(idxLevel).hArrRMP.at(i);
	}
	//Chép dữ liệu từ temp trên host sang rmp trên device
	CHECK(cudaStatus =hipMemcpy(rmp,temp,sizeof(int)*noElemVerOnRMP,hipMemcpyHostToDevice)); //chép dữ liệu từ temp ở host sang rmp trên device
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	std::free(temp);

	//printf("\n\n ******* rmp *********\n");
	//displayDeviceArr(rmp,noElemVerOnRMP);
Error:
	return status;
}

int PMS::findListVer(Embedding **dArrPointerEmbedding,int noElemEmbedding,int *rmp,int noElemVerOnRMP,vector<listVer>& hListVer){
	int status = 0;
	hipError_t cudaStatus;

	int noElemListVer= noElemVerOnRMP * noElemEmbedding; //số lượng phần tử của listVer bằng số lượng đỉnh trên right most path nhân với số lượng embedding
	hListVer.resize(Level);
	hListVer.at(idxLevel).noElem=noElemListVer;

	CHECK(cudaStatus = hipMalloc((void**)&hListVer.at(idxLevel).dListVer,sizeof(int)*noElemListVer)); //cấp phát bộ nhớ cho listVer
	if(cudaStatus!=hipSuccess){
		printf("\n CudaMalloc listVer failed");
		status =-1;
		goto Error;
	}
	//cần có danh sách các đỉnh thuộc righ most path để thực hiện mở rộng
	//Tìm danh sách các đỉnh thuộc right most path ở các cột embedding để thực hiện mở rộng
	dim3 block(blocksize);
	dim3 grid((noElemEmbedding + block.x -1)/block.x);
	kernelFindListVer<<<grid,block>>>(dArrPointerEmbedding,noElemEmbedding,rmp,noElemVerOnRMP,hListVer.at(idxLevel).dListVer); //tìm listVer
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	////hiển thị danh sách đỉnh
	//printf("\n\n ********* listVer *********\n");
	//displayDeviceArr(hListVer.at(idxLevel).dListVer,noElemListVer);
Error:
	return status;
}

int PMS::findVerOnRMPForBWCheck(ptrArrEmbedding hLevelPtrEmbeddingatALevel,int* rmp,int noElemVerOnRMP,int *&dArrVidOnRMP){
	int status = 0;
	hipError_t cudaStatus;
		int noElemdArrVidOnRMP= noElemVerOnRMP - 1;
		//int *fromPosCol=nullptr; //lưu trữ các cột của Embedding mà tại đó thuộc right most path. Thật ra mình có thể suy luận được từ rmp

		cudaStatus = hipMalloc((void**)&dArrVidOnRMP,hLevelPtrEmbeddingatALevel.noElemEmbedding*noElemdArrVidOnRMP*sizeof(int));
		CHECK(cudaStatus);
		if(cudaStatus!=hipSuccess){
			status =-1;
			goto Error;
		}

		FUNCHECK(status = displaydArrPointerEmbedding(hLevelPtrEmbeddingatALevel.dArrPointerEmbedding,hLevelPtrEmbeddingatALevel.noElem,hLevelPtrEmbeddingatALevel.noElemEmbedding));
		if(status!=0){
			goto Error;
		}

		dim3 block(blocksize);
		dim3 grid((hLevelPtrEmbeddingatALevel.noElemEmbedding + block.x -1)/block.x);

		//Hàm này tìm các vid thuộc right most path và lưu vào mảng dArrVidOnRMP. Mảng này dùng để tìm các valid backward edge.
		kernelFindVidOnRMP<<<grid,block>>>(hLevelPtrEmbeddingatALevel.dArrPointerEmbedding,hLevelPtrEmbeddingatALevel.noElemEmbedding,rmp,noElemVerOnRMP,dArrVidOnRMP,noElemdArrVidOnRMP);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		CHECK(cudaStatus);
		if(cudaStatus!=hipSuccess){
			status =-1;
			goto Error;
		}

		printf("\n ******** dArrVidOnRMP *******\n");
		displayDeviceArr(dArrVidOnRMP,noElemdArrVidOnRMP*hLevelPtrEmbeddingatALevel.noElemEmbedding);
		//printf("\n ******** fromPosCol *******\n");
		//displayDeviceArr(fromPosCol,noElemdArrVidOnRMP);
Error:
	return status;
}

//Tìm các mở rộng hợp lệ (forward & backward) từ EXT cuối cùng
int PMS::findValidFBExtension(int *listOfVer,ptrArrEmbedding hLevelPtrEmbeddingAtALevel,int k,int fromColumEmbedding,int *dArrVidOnRMP,int *rmp){
	int status =0;
	hipError_t cudaStatus;


	//Tìm bậc lớn nhất của các đỉnh cần mở rộng trong listOfVer
	int maxDegreeOfVer=0;
	float *dArrDegreeOfVid=nullptr; //chứa cậc của các đỉnh trong listOfVer, dùng để duyệt qua các đỉnh lân cận
	//trong csdl
	status=findMaxDegreeOfVer(listOfVer,maxDegreeOfVer,dArrDegreeOfVid,hLevelPtrEmbeddingAtALevel.noElemEmbedding); //tìm bậc lớn nhất
	FUNCHECK(status);
	if(status==-1){
		printf("\n findMaxDegreeOfVer() in forwardExtension() failed");
		goto Error;
	}
	//Tạo mảng dArrV để ghi nhận những mở rộng hợp lệ. 
	V *dArrV=nullptr;
	dArrV = (V*)malloc(sizeof(V));

	dArrV->noElem =maxDegreeOfVer*hLevelPtrEmbeddingAtALevel.noElemEmbedding;
	CHECK(cudaStatus=hipMalloc((void**)&(dArrV->valid),(dArrV->noElem)*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus = hipMemset(dArrV->valid,0,(dArrV->noElem)*sizeof(int)));
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}
	}
	CHECK(cudaStatus=hipMalloc((void**)&(dArrV->backward),(dArrV->noElem)*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(dArrV->backward,0,(dArrV->noElem)*sizeof(int)));
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}
	}


	//////Các mở rộng hợp lệ sẽ được ghi nhận vào mảng dArrV, đồng thời thông tin của cạnh mở rộng gồm dfscode, vgi, vgj và row pointer của nó cũng được xây dựng
	//////và lưu trữ trong mảng EXT *dExtensionTemp, mảng này có số lượng phần tử bằng với mảng dArrV. Sau đó chúng ta sẽ rút trích những mở rộng hợp lệ này và lưu vào dExt. 
	//////Để xây dựng dfscode (vi,vj,li,lij,lj) thì chúng ta cần:
	////// - Dựa vào giá trị của right most path để xác định vi
	////// - Dựa vào maxid để xác định vj
	////// - Dựa vào CSDL để xác định các thành phần còn lại.
	//////Chúng ta có thể giải phóng bộ nhớ của dExtensionTemp sau khi đã trích các mở rộng hợp lệ thành công.


	EXT *dArrExtensionTemp= nullptr; //Nơi lưu trữ tạm thời tất cả các cạnh mở rộng. Sau đó, chúng sẽ được lọc ra các mở rộng hợp lệ sang EXTk tương ứng.
	CHECK(cudaStatus = hipMalloc((void**)&dArrExtensionTemp,(dArrV->noElem)*sizeof(EXT)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(dArrExtensionTemp,0,dArrV->noElem*sizeof(EXT)));
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}
	}

	printf("\n\n noElem_dArrV:%d",dArrV->noElem );



	////Gọi kernel với các đối số: CSDL, bậc của các đỉnh, dArrV, dArrExtension,noElem_Embedding,maxDegreeOfVer,idxQ,dArrPointerEmbedding,minLabel,maxid
	dim3 block(blocksize);
	dim3 grid((hLevelPtrEmbeddingAtALevel.noElemEmbedding+block.x - 1)/block.x);
	//kernelFindValidForwardExtension<<<grid,block>>>(hLevelPtrEmbeddingAtALevel.dArrPointerEmbedding,hLevelPtrEmbeddingAtALevel.noElem,hLevelPtrEmbeddingAtALevel.noElemEmbedding,hdb.at(0).dO,hdb.at(0).dLO,hdb.at(0).dN,hdb.at(0).dLN,dArrDegreeOfVid,maxDegreeOfVer,dArrExtensionTemp,listOfVer,minLabel,maxId,fromColumEmbedding,dArrV->valid,dArrV->backward);
	kernelFindValidFBExtension<<<grid,block>>>(hLevelPtrEmbeddingAtALevel.dArrPointerEmbedding,hLevelPtrEmbeddingAtALevel.noElem,hLevelPtrEmbeddingAtALevel.noElemEmbedding,hdb.at(0).dO,hdb.at(0).dLO,hdb.at(0).dN,hdb.at(0).dLN,dArrDegreeOfVid,maxDegreeOfVer,dArrV->valid,dArrV->backward,dArrExtensionTemp,listOfVer,minLabel,maxId,fromColumEmbedding,dArrVidOnRMP,hRMPv2.at(idxLevel).noElem-1,rmp);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelFindValidForwardExtension in forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}
	////In mảng dArrV để kiểm tra thử
	printf("\n****************dArrV_valid*******************\n");
	displayDeviceArr(dArrV->valid,dArrV->noElem);
	printf("\n****************dArrV_backward*******************\n");
	displayDeviceArr(dArrV->backward,dArrV->noElem);
	
	////Chép kết quả từ dArrExtensionTemp sang dExt
	//chúng ta cần có mảng dArrV để trích các mở rộng duy nhất
	//Hàm này cũng gọi hàm trích các mở rộng duy nhất và tính độ hỗ trợ của chúng
	FUNCHECK(status = displayDeviceEXT(dArrExtensionTemp,dArrV->noElem));
	FUNCHECK(status = extractValidExtensionTodExtv2(dArrExtensionTemp,dArrV,dArrV->noElem,k));
	if(status!=0){
		goto Error;
	}


	CHECK(cudaStatus = hipFree(dArrV->valid));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus = hipFree(dArrV->backward));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus = hipFree(dArrDegreeOfVid));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	free(dArrV);
Error:
	return status;
}



int PMS::FSMiningv2() //đã có Embedding mới và RMP tương ứng với nó. Khai thác các mở rộng
{
	int status = 0;
	hipError_t cudaStatus;
	FUNCHECK(status = updateRMP());
	if(status!=0){
		goto Error;
	}
	//2. Lấy số lượng đỉnh trên right most path
	int noElemVerOnRMP = hRMPv2.at(idxLevel).noElem; //right most path chứa bao nhiêu đỉnh

	//3.Tìm danh sách các đỉnh thuộc right most path của các embedding
	//3.1 xây dựng dArrPointerEmbedding
	FUNCHECK(status = buildArrPointerEmbedding(hEmbedding,hLevelPtrEmbeddingv2));//xây dựng dArrPointerEmbedding dựa vào hEmbedding
	if(status!=0){
		goto Error;
	}
	

	//3.2 xây dựng rmp on device dựa vào hRMPv2
	int *rmp = nullptr; //rigt most path trên bộ nhớ device
	FUNCHECK(status = buildrmpOnDevice(hRMPv2.at(idxLevel),rmp));
	if(status!=0){
		goto Error;
	}
	printf("\n\n ******* rmp *********\n");
	displayDeviceArr(rmp,noElemVerOnRMP);
	//3.3. Dựa vào dArrPointerEmbedding và rmp để tìm các đỉnh cần mở rộng
	FUNCHECK(status = findListVer(hLevelPtrEmbeddingv2.at(idxLevel).dArrPointerEmbedding,hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding,rmp,noElemVerOnRMP,hListVerv2));
	if(status !=0){
		goto Error;
	}
		//hiển thị danh sách đỉnh
	printf("\n\n ********* listVer *********\n");
	displayDeviceArr(hListVerv2.at(idxLevel).dListVer,hListVerv2.at(idxLevel).noElem);


	//Chuẩn bị bộ nhớ ở Level mới
		int noElemEXTk =noElemVerOnRMP; //Số lượng phần tử EXTk bằng số lượng đỉnh trên right most path
	//hEXTk.resize(noElemEXTk); //Các mở rộng hợp lệ từ đỉnh k sẽ được lưu trữ vào EXTk tương ứng.

	//Quản lý theo Level phục vụ cho khai thác đệ quy
	hLevelEXTv2.resize(Level); //Khởi tạo vector quản lý bộ nhớ cho level
	hLevelEXTv2.at(idxLevel).noElem = noElemVerOnRMP; //Cập nhật số lượng phần tử vector vE bằng số lượng đỉnh trên RMP
	hLevelEXTv2.at(idxLevel).vE.resize(noElemVerOnRMP); //Cấp phát bộ nhớ cho vector vE.

	hLevelUniEdgev2.resize(Level); //Số lượng phần tử UniEdge cũng giống với EXT
	hLevelUniEdgev2.at(idxLevel).noElem=noElemVerOnRMP;
	hLevelUniEdgev2.at(idxLevel).vUE.resize(noElemVerOnRMP);

	hLevelUniEdgeSatisfyMinsupv2.resize(Level);
	hLevelUniEdgeSatisfyMinsupv2.at(idxLevel).noElem= noElemVerOnRMP;
	hLevelUniEdgeSatisfyMinsupv2.at(idxLevel).vecUES.resize(noElemVerOnRMP);

	
	int *tempListVerCol = nullptr; //chứa danh sách các đỉnh cần mở rộng thuộc một embedding column cụ thể.
	CHECK(cudaStatus = hipMalloc((void**)&tempListVerCol,hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding * sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}

	//Cần kiểm tra lại hàm tính dArrVidOnRMP
	if(hRMPv2.at(idxLevel).noElem>2){ //Nếu số lượng đỉnh trên RMP lớn hơn 2 thì mới tồn tại backward. Vì ở đây chỉ xét đơn đồ thị vô hướng
		//dArrVidOnRMP: chứa các đỉnh thuộc RMP của mỗi Embedding. Dùng để kiểm tra sự tồn tại của đỉnh trên right most path
		//khi tìm các mở rộng backward. Chỉ dùng tới khi right most path có 3 đỉnh trở lên (chỉ xét trong đơn đồ thị vô hướng).
		int *dArrVidOnRMP = nullptr; //lưu trữ các đỉnh trên RMP của Embedding, có kích thước nhỏ hơn  đỉnh so với RMP
		FUNCHECK(status = findVerOnRMPForBWCheck(hLevelPtrEmbeddingv2.at(idxLevel),rmp,noElemVerOnRMP,dArrVidOnRMP));
		if(status!=0){
			goto Error;
		}

		for (int i = 0; i < noElemVerOnRMP ; i++)
		{
			int colEmbedding = hRMPv2.at(idxLevel).hArrRMP.at(i); //Tìm mở rộng cho các đỉnh tại vị trí colEmbedding trong vector hEmbedding
			currentColEmbedding=colEmbedding; //Đang mở rộng từ cột nào của Embedding. Được dùng để cập nhật prevCol, phục vụ cho việc xây dựng Right Most Path
			int k = i; //lưu vào Extk với k = i; K=0 đại diện cho EXT0: là EXT cuối
			dim3 block(blocksize);
			dim3 grid((hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding + block.x -1)/block.x);

			kernelExtractFromListVer<<<grid,block>>>(hListVerv2.at(idxLevel).dListVer,i*hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding,hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding,tempListVerCol);//trích các đỉnh từ listVer, từ vị trí i*noElemEmbedding,trích noElemEmbedding phần tử, bỏ vào tempListVerCol
			hipDeviceSynchronize();
			cudaStatus = hipGetLastError();
			CHECK(cudaStatus);
			printf("\n ****** tempListVerCol ***********\n");
			displayDeviceArr(tempListVerCol,hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding);


			//Gọi hàm tìm các mở rộng hợp lệ từ đỉnh cuối i=0
			if(i==0){ //khai thác forward lẫn backward. với các tham số:
				//1. Mở rộng từ danh sách đỉnh nào, 2. embedding trên device,3. trên EXTk nào,4.colembedding: để cập nhật vi cho mở rộng mới
				FUNCHECK(status = findValidFBExtension(tempListVerCol,hLevelPtrEmbeddingv2.at(idxLevel),k,colEmbedding,dArrVidOnRMP,rmp));
				if(status!=0){
					goto Error;
				}				
			}
			else
			{
				//chỉ khai thác forward
			}

		}

		//Viết hàm khai thác các valid forward edge và valid backward edge
		//Nhưng lưu kết quả vào đâu?
		//==>Những mở rộng hợp lệ đều được lưu vào EXTk và backward edge chỉ tồn tại ở đỉnh mở rộng cuối cùng.

		//1.Xử lý mở rộng từ đỉnh cuối
		//Giải quyết mọi thứ bằng vấn đề tham số cho các hàm.
	}
	else //Ngược lại, chỉ khai thác forward khi số lượng đỉnh trên RMP chỉ bằng 2.
	{
		for (int i = 0; i < noElemVerOnRMP ; i++)
		{
			int colEmbedding = hRMPv2.at(idxLevel).hArrRMP.at(i); //Tìm mở rộng cho các đỉnh tại vị trí colEmbedding trong vector hEmbedding
			currentColEmbedding=colEmbedding; //Đang mở rộng từ cột nào của Embedding. Được dùng để cập nhật prevCol, phục vụ cho việc xây dựng Right Most Path
			int k = i; //lưu vào Extk với k = i; K=0 đại diện cho EXT0: là EXT cuối
			dim3 block(blocksize);
			dim3 grid((hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding + block.x -1)/block.x);

			kernelExtractFromListVer<<<grid,block>>>(hListVerv2.at(idxLevel).dListVer,i*hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding,hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding,tempListVerCol);//trích các đỉnh từ listVer, từ vị trí i*noElemEmbedding,trích noElemEmbedding phần tử, bỏ vào tempListVerCol
			hipDeviceSynchronize();
			cudaStatus = hipGetLastError();
			CHECK(cudaStatus);
			printf("\n ****** tempListVerCol ***********\n");
			displayDeviceArr(tempListVerCol,hLevelPtrEmbedding.at(idxLevel).noElemEmbedding);


			//gọi hàm forwardExtension để tìm các mở rộng forward từ cột colEmbedding, lưu kết quả vào hEXTk tại vị trí k, với các đỉnh
			// cần mở rộng là tempListVerCol, thuộc righ most path
			////Hàm này cũng đồng thời trích các mở rộng duy nhất từ các EXT và lưu vào UniEdge
			//Hàm này cũng gọi đệ quy FSMining bên trong
			FUNCHECK(status = forwardExtension(k,tempListVerCol,hLevelPtrEmbeddingv2.at(idxLevel).noElemEmbedding,hRMPv2.at(idxLevel).hArrRMP.at(i)));
			if(status ==-1){
				goto Error;
			}

			//FSMining();
			if(hLevelEXTv2.at(idxLevel).vE.at(i).noElem>0){ //Nếu số lượng phần tử của mảng dArrExt = 0 thì chúng ta không giải phóng bộ nhớ dArrExt vì nó chưa được cấp phát.
				hipFree(hLevelEXTv2.at(idxLevel).vE.at(i).dArrExt);
				hipFree(hLevelUniEdgev2.at(idxLevel).vUE.at(i).dArrUniEdge);
			}			
		}
	}
	
	hLevelEXTv2.at(idxLevel).vE.clear();
	hLevelUniEdgev2.at(idxLevel).vUE.clear();

	hipFree(tempListVerCol);
	hipFree(hListVerv2.at(idxLevel).dListVer);
	--Level;
	idxLevel=Level-1;	
Error:
	return status;
}


__global__ void kernelExtractFromListVer(int *listVer,int from,int noElemEmbedding,int *tempListVerCol){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemEmbedding){
		tempListVerCol[i] = listVer[from+i];
	}
}

//kernel tìm các mở rộng hợp lệ và ghi nhận vào mảng dArrV và dArrExtension tương ứng.
__global__ void kernelFindValidForwardExtension(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int noElem_Embedding,int *d_O,int *d_LO,int *d_N,int *d_LN,float *dArrDegreeOfVid,int maxDegreeOfVer,EXT *dArrExtension,int *listOfVer,int minLabel,int maxId,int fromRMP,int *dArrV_valid,int *dArrV_backward){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//Duyệt qua các Embedding và xét các mở rộng cho đỉnh tại vị trí idxQ
	if(i<noElem_Embedding){
		int posColumn =noElem_dArrPointerEmbedding-1;
		int posRow=i;
		int col = posColumn;
		int row = posRow;
		int vid = listOfVer[i];
		int degreeVid=__float2int_rn(dArrDegreeOfVid[i]);
		//Duyệt qua các đỉnh kề với đỉnh vid dựa vào số lần duyệt là bậc
		int indexToVidIndN=d_O[vid];
		int labelFromVid = d_LO[vid];
		int toVid;
		int labelToVid;
		bool b=true;
		for (int j = 0; j < degreeVid; j++,indexToVidIndN++) //Duyệt qua tất cả các đỉnh kề với đỉnh vid, nếu đỉnh không thuộc embedding thì --> cạnh cũng không thuộc embedding vì đây là Q cuối
		{			
			toVid=d_N[indexToVidIndN]; //Lấy vid của đỉnh cần kiểm tra
			labelToVid = d_LO[toVid]; //lấy label của đỉnh cần kiểm tra
			posColumn=col;
			posRow=row;
			Embedding *Q=dArrPointerEmbedding[posColumn];
			//printf("\nThread %d, j: %d has ToVidLabel:%d",i,j,labelToVid);
			//1. Trước tiên kiểm tra nhãn của labelToVid có nhỏ hơn minLabel hay không. Nếu nhỏ hơn thì return
			if(labelToVid<minLabel) continue;
			//2. kiểm tra xem đỉnh toVid có tồn tại trong embedding hay không nếu tồn tại thì return
			//Duyệt qua embedding column từ Q cuối đến Q đầu, lần lượt lấy vid so sánh với toVid

			//printf("\n Q[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);//Q[1][0]
			if(toVid==Q[posRow].vid) continue;
			//printf("\nj:%d toVid:%d Q.vid:%d",j,toVid,Q[posRow].vid);

			while (true)
			{
				posRow = Q[posRow].idx;//0
				posColumn=posColumn-1;		//0
				Q=dArrPointerEmbedding[posColumn];
				//printf("\n posColumn[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);//Q[0][0]
				//printf("\nj:%d toVid:%d Q.vid:%d",j,toVid,Q[posRow].vid);
				if(toVid==Q[posRow].vid) {
					b=false; break;
				}
				posRow=Q[posRow].idx;//-1
				//printf("\nposRow:%d",posRow);
				if(posRow==-1) break;
			}
			if (b==false){b=true; continue;}
			int indexOfd_arr_V=i*maxDegreeOfVer+j;
			//printf("\nThread %d: m:%d",i,maxDegreeOfVer);
			int indexOfd_LN=indexToVidIndN;
			//dArrV[indexOfd_arr_V].valid=1;
			dArrV_valid[indexOfd_arr_V]=1;
			dArrV_backward[indexOfd_arr_V]=0;
			//printf("\ndArrV[%d].valid:%d",indexOfd_arr_V,dArrV[indexOfd_arr_V].valid);
			//cập nhật dữ liệu cho mảng dArrExtension
			dArrExtension[indexOfd_arr_V].vgi=vid;
			dArrExtension[indexOfd_arr_V].vgj=toVid;
			dArrExtension[indexOfd_arr_V].lij=d_LN[indexOfd_LN];
			//printf("\n");
			//printf("d_LN[%d]:%d ",indexOfd_LN,d_LN[indexOfd_LN]);
			dArrExtension[indexOfd_arr_V].li=labelFromVid;
			dArrExtension[indexOfd_arr_V].lj=labelToVid;
			dArrExtension[indexOfd_arr_V].vi=fromRMP;
			dArrExtension[indexOfd_arr_V].vj=maxId+1;
			//dArrExtension[indexOfd_arr_V].posColumn=idxQ;
			dArrExtension[indexOfd_arr_V].posRow=row;
		}
	}
}


//kernel tìm các mở rộng hợp lệ và ghi nhận vào mảng dArrV và dArrExtension tương ứng.
//fromRMP: dùng để cập nhật vi
//rmp: dùng để cập nhật vj
__global__ void kernelFindValidFBExtension(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int noElem_Embedding,int *d_O,int *d_LO,int *d_N,int *d_LN,float *dArrDegreeOfVid,int maxDegreeOfVer,int *dArrV_valid,int *dArrV_backward,EXT *dArrExtension,int *listOfVer,int minLabel,int maxId,int fromRMP, int *dArrVidOnRMP,int segdArrVidOnRMP,int *rmp){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//Duyệt qua các Embedding và xét các mở rộng cho đỉnh tại vị trí idxQ
	if(i<noElem_Embedding){
	//	int fromPosCol;
		//int idxRMP=0;
		//int noELemVerOnRMP = segdArrVidOnRMP +1;
		int posColumn =noElem_dArrPointerEmbedding-1;
		int posRow=i;
		int col = posColumn;
		int row = posRow;
		//Embedding *Q=dArrPointerEmbedding[idxQ];
		int vid = listOfVer[i];
		int degreeVid=__float2int_rn(dArrDegreeOfVid[i]);
		//Duyệt qua các đỉnh kề với đỉnh vid dựa vào số lần duyệt là bậc
		int indexToVidIndN=d_O[vid];
		int labelFromVid = d_LO[vid];
		int toVid;
		int labelToVid;
		bool b=true;
		for (int j = 0; j < degreeVid; j++,indexToVidIndN++) //Duyệt qua tất cả các đỉnh kề với đỉnh vid, nếu đỉnh không thuộc embedding thì --> cạnh cũng không thuộc embedding vì đây là Q cuối, Nếu đỉnh không thuộc Embedding thì nó cũng không phải là backward
		{			
			//1.Kiểm tra forward
			toVid=d_N[indexToVidIndN]; //Lấy vid của đỉnh cần kiểm tra
			labelToVid = d_LO[toVid]; //lấy label của đỉnh cần kiểm tra
			posColumn=col;
			posRow=row;
			Embedding *Q=dArrPointerEmbedding[posColumn];
			printf("\nThread %d, j: %d has ToVidLabel:%d",i,j,labelToVid);
			//1. Trước tiên kiểm tra nhãn của labelToVid có nhỏ hơn minLabel hay không. Nếu nhỏ hơn thì return. Vì nó cũng không có khả năng là backward extension
			if(labelToVid<minLabel) 
					return;
					//goto backward;
			//2. kiểm tra xem đỉnh toVid có tồn tại trong embedding hay không, nếu tồn tại thì nó không là forward extension --> có khả năng nó là backward
			//Duyệt qua embedding column từ Q cuối đến Q đầu, lần lượt lấy vid so sánh với toVid

			//printf("\n Q[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);//Q[1][0]
			if(toVid==Q[posRow].vid) 
					goto backward;

			//printf("\nj:%d toVid:%d Q.vid:%d",j,toVid,Q[posRow].vid);

			while (true)
			{
				posRow = Q[posRow].idx;//0
				posColumn=posColumn-1;		//0
				Q=dArrPointerEmbedding[posColumn];
				//printf("\n posColumn[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);//Q[0][0]
				//printf("\nj:%d toVid:%d Q.vid:%d",j,toVid,Q[posRow].vid);
				if(toVid==Q[posRow].vid) {
					b=false; break;
				}
				posRow=Q[posRow].idx;//-1
				//printf("\nposRow:%d",posRow);
				if(posRow==-1) break;
			}
			if (b==false){
				b=true; 
				goto backward;
			}
			int indexOfd_arr_V=i*maxDegreeOfVer+j;
			//printf("\nThread %d: m:%d",i,maxDegreeOfVer);
			int indexOfd_LN=indexToVidIndN;
			dArrV_valid[indexOfd_arr_V]=1;
			printf("\ndArrV[%d].valid:%d",indexOfd_arr_V,dArrV_valid[indexOfd_arr_V]);
			//cập nhật dữ liệu cho mảng dArrExtension
			dArrExtension[indexOfd_arr_V].vgi=vid;
			dArrExtension[indexOfd_arr_V].vgj=toVid;
			dArrExtension[indexOfd_arr_V].lij=d_LN[indexOfd_LN];
			printf("\n");
			printf("d_LN[%d]:%d ",indexOfd_LN,d_LN[indexOfd_LN]);
			dArrExtension[indexOfd_arr_V].li=labelFromVid;
			dArrExtension[indexOfd_arr_V].lj=labelToVid;
			dArrExtension[indexOfd_arr_V].vi=fromRMP;
			dArrExtension[indexOfd_arr_V].vj=maxId+1;
			//dArrExtension[indexOfd_arr_V].posColumn=idxQ;
			dArrExtension[indexOfd_arr_V].posRow=row;
backward:
			//2. Kiểm tra backward
			for (int k = 1; k < segdArrVidOnRMP; k++)
			{
				if(toVid == dArrVidOnRMP[i*segdArrVidOnRMP+k]){

					int indexOfd_arr_V=i*maxDegreeOfVer+j;
					//printf("\nThread %d: m:%d",i,maxDegreeOfVer);
					int indexOfd_LN=indexToVidIndN;
					dArrV_valid[indexOfd_arr_V] = 1;
					dArrV_backward[indexOfd_arr_V]=1;
					printf("\ndArrV[%d].valid:%d backward:%d",indexOfd_arr_V,dArrV_valid[indexOfd_arr_V],dArrV_backward[indexOfd_arr_V]);
					//cập nhật dữ liệu cho mảng dArrExtension
					dArrExtension[indexOfd_arr_V].vgi=vid;
					dArrExtension[indexOfd_arr_V].vgj=toVid;
					dArrExtension[indexOfd_arr_V].lij=d_LN[indexOfd_LN];
					printf("\n");
					printf("d_LN[%d]:%d ",indexOfd_LN,d_LN[indexOfd_LN]);
					dArrExtension[indexOfd_arr_V].li=labelFromVid;
					dArrExtension[indexOfd_arr_V].lj=labelToVid;
					dArrExtension[indexOfd_arr_V].vi=maxId;
					//dArrExtension[indexOfd_arr_V].vj=fromPosCol[i*segdArrVidOnRMP+k];
					dArrExtension[indexOfd_arr_V].vj=rmp[k+1];
					//dArrExtension[indexOfd_arr_V].posColumn=idxQ;
					dArrExtension[indexOfd_arr_V].posRow=row;
					break; //thoát khỏi vòng lặp hiện tại
				}
			}
		}
	}
}


__global__ void	kernelGetPointerdArrEmbedding(Embedding *dArrEmbedding,Embedding **dArrPointerEmbedding,int idx){
	dArrPointerEmbedding[idx]=dArrEmbedding;
	//printf("\n PointerdArrEmbedding:%p, PointerdArrPointerEmbedding:%p",dArrEmbedding,dArrPointerEmbedding[idx]);
}

__global__ void kernelPrintdArrPointerEmbedding(Embedding **dArrPointerEmbedding,int noElem,int sizeArr){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElem){
		Embedding *E = dArrPointerEmbedding[i];
		for (int j = 0; j < sizeArr; j++)
		{
			printf("\n Thread %d pointer:%p (idx vid):(%d %d)",i,E,E[j].idx,E[j].vid);
		}
	}
}

//kernel in mảng struct_V *dArrV trên device
__global__ void kernelprintdArrV(V *dArrV,int noElem_dArrV,EXT *dArrExtension){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_dArrV){
		int vi = dArrExtension[i].vi;
		int vj = dArrExtension[i].vj;
		int li = dArrExtension[i].li;
		int lij = dArrExtension[i].lij;
		int lj = dArrExtension[i].lj;
		printf("\n dArrV[%d].backward:%d ,dArrV[%d].valid:%d Extension:(vgi:%d,vgj:%d) (vi:%d vj:%d li:%d lij:%d lj:%d)",i,dArrV[i].backward,i,dArrV[i].valid,dArrExtension[i].vgi,dArrExtension[i].vgj,vi,vj,li,lij,lj);
	}

}

hipError_t printdArrV(V *dArrV,int noElem_dArrV,EXT *dArrExtension){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem_dArrV + block.x -1 )/block.x);
	kernelprintdArrV<<<grid,block>>>(dArrV,noElem_dArrV,dArrExtension);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in printdArrV() failed", cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}
//Kernel in nội dung mảng EXT *dExt
__global__ void kernelPrintdExt(EXT *dExt,int noElem_dExt){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_dExt){		
		int vi=dExt[i].vi;
		int vj=dExt[i].vj;
		int li= dExt[i].li;
		int lij=dExt[i].lij;
		int lj=dExt[i].lj;
		int vgi=dExt[i].vgi;
		int vgj=dExt[i].vgj;
		//		int posColumn= dExt[i].posColumn;
		int posRow=dExt[i].posRow;
		printf("\n Thread %d (vi:%d vj:%d li:%d lij:%d lj:%d) (vgi:%d vgj:%d) ( posRow:%d)",i,vi,vj,li,lij,lj,vgi,vgj,posRow);
	}

}

//Hàm in dExt
inline hipError_t printdExt(EXT *dExt,int noElem_dExt){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_dExt+block.x -1)/block.x);
	kernelPrintdExt<<<grid,block>>>(dExt,noElem_dExt);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelPrintdExt in printdExt() failed", cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

//kernel trích các mở rộng hợp lệ từ mảng dArrExtension sang mảng dExt
__global__ void kernelExtractValidExtensionTodExt(EXT *dArrExtension,int *dArrValid,int *dArrValidScanResult,int noElem_dArrV,EXT *dExt,int noElem_dExt){
	int i =blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dArrV){
		if(dArrValid[i]==1){
			dExt[dArrValidScanResult[i]].vi = dArrExtension[i].vi;
			dExt[dArrValidScanResult[i]].vj = dArrExtension[i].vj;
			dExt[dArrValidScanResult[i]].li = dArrExtension[i].li;
			dExt[dArrValidScanResult[i]].lij = dArrExtension[i].lij;
			dExt[dArrValidScanResult[i]].lj = dArrExtension[i].lj;
			dExt[dArrValidScanResult[i]].vgi = dArrExtension[i].vgi;
			dExt[dArrValidScanResult[i]].vgj = dArrExtension[i].vgj;
			//dExt[dArrValidScanResult[i]].posColumn = dArrExtension[i].posColumn;
			dExt[dArrValidScanResult[i]].posRow = dArrExtension[i].posRow;
		}

	}

}

////kernel trích phần tử valid từ mảng dArrV và lưu vào mảng dArrValid
//__global__ void kernelExtractValidFromdArrV(V *dArrV,int noElem_dArrV,int *dArrValid){
//	int i = threadIdx.x + blockDim.x*blockIdx.x;
//	if(i<noElem_dArrV){
//		dArrValid[i]=dArrV[i].valid;
//	}
//}

__global__ void	kernelForwardPossibleExtension_NonLast(EXT *dArrExt,int noElem,int Lv,int *dArrAllPossibleExtension){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElem){
		int lij,lj;
		lij=dArrExt[i].lij;
		lj=dArrExt[i].lj;
		int idx=lij*Lv+lj;
		dArrAllPossibleExtension[idx]=1;
	}
}

//kernel lấy nhãn from Li
__global__ void kernelGetFromLabel(EXT *dArrExt,int *dFromLi){
	*dFromLi	= dArrExt[0].li;
}

__global__ void kernelFilldArrUniEdge(int *dArrAllPossibleExtension,int *dArrAllPossibleExtensionScanResult,int noElem_dArrAllPossibleExtension,UniEdge *dArrUniEdge,int Lv,int *dFromLi){
	int i = blockDim.x*blockIdx.x +threadIdx.x;
	if(i<noElem_dArrAllPossibleExtension){
		if(dArrAllPossibleExtension[i]==1){
			int li,lij,lj;
			li=*dFromLi;
			lij = i/Lv;
			lj=i%Lv;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].li=li;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].lij=lij;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].lj=lj;
		}
	}
}

int displaydArrUniEdge(UniEdge *dArrUniEdge,int noElem_dArrUniEdge){
	hipError_t cudaStatus;
	int status =0;
	UniEdge *hArrUniEdge = (UniEdge*)malloc(sizeof(UniEdge) * noElem_dArrUniEdge);
	if(hArrUniEdge == NULL){
		printf("\n malloc hArrUniEde failed");
		goto Error;
	}

	cudaStatus = hipMemcpy(hArrUniEdge,dArrUniEdge,sizeof(UniEdge)*noElem_dArrUniEdge,hipMemcpyDeviceToHost);
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	for (int i = 0; i < noElem_dArrUniEdge; i++)
	{
		printf("\n U[%d] (li lij lj):(%d %d %d)",i,hArrUniEdge[i].li,hArrUniEdge[i].lij,hArrUniEdge[i].lj);
	}

	std::free(hArrUniEdge);
Error:
	return status;
}



//Hàm trích các mở rộng hợp lệ từ mảng dArrExtension sang mảng dExt
int PMS::extractValidExtensionTodExt(EXT *dArrExtension,V *dArrV,int noElem_dArrV,int idxEXT){
	hipError_t cudaStatus;

	int status =0;
	//2. Scan mảng dArrValid để lấy kích thước của mảng cần tạo
	int *dArrValidScanResult = nullptr;

	cudaStatus = hipMalloc((void**)&dArrValidScanResult,sizeof(int)*noElem_dArrV);
	if (cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n CudaMalloc dArrValidScanResult in extractValidExtensionToExt() failed");
		goto Error;
	}
	else
	{
		hipMemset(dArrValidScanResult,0,sizeof(int)*noElem_dArrV);
	}


	//cudaStatus = scanV(dArrV->valid,noElem_dArrV,dArrValidScanResult); //hàm scan này có vấn đề. Nó làm thay đổi giá trị đầu vào.
	//if (cudaStatus!=hipSuccess){
	//	status = -1;
	//	fprintf(stderr,"\n scanV dArrValid in extractValidExtensionToExt() failed");
	//	goto Error;
	//}
	myScanV(dArrV->valid,noElem_dArrV,dArrValidScanResult);

	////In nội dung kết quả dArrValidScanResult
	printf("\n********dArrValid******\n");
	displayDeviceArr(dArrV->valid,noElem_dArrV);

	printf("\n********dArrValidScanResult******\n");
	displayDeviceArr(dArrValidScanResult,noElem_dArrV);

	//3. Lấy kích thước của mảng dArrExt;
	int noElem_dExt=0;
	cudaStatus=getSizeBaseOnScanResult(dArrV->valid,dArrValidScanResult,noElem_dArrV,noElem_dExt);
	if (cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n getSizeBaseOnScanResult in extractValidExtensionToExt() failed");
		goto Error;
	}

	//In nội dung noElem_dExt
	printf("\n******** noElem In dArrExt ******\n");
	printf("\n noElem_dExt:%d",noElem_dExt);
	hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem = noElem_dExt;
	/**************** Nếu không tìm được mở rộng nào thì return *************/
	if (noElem_dExt == 0) 
	{
		hipFree(dArrValidScanResult);
		return status;
	}
	//Nếu tìm được mở rộng thì xây dựng EXTk, rồi trích các mở rộng duy nhất và tính độ hỗ trợ của chúng. Đồng thời
	//lọc ra các độ hỗ trợ thoả minsup
	//Quản lý theo Level
	//4. Khởi tạo mảng dArrExt có kích thước noElem_dExt rồi trích dữ liệu từ dArrExtension sang dựa vào dArrValid.
	//hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem = noElem_dExt;
	cudaStatus = hipMalloc((void**)&hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt*sizeof(EXT));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n hipMalloc dExt in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,0,sizeof(EXT)*noElem_dExt);
	}
	dim3 blockb(blocksize);
	dim3 gridb((noElem_dArrV+blockb.x -1)/blockb.x);
	kernelExtractValidExtensionTodExt<<<gridb,blockb>>>(dArrExtension,dArrV->valid,dArrValidScanResult,noElem_dArrV,hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n hipDeviceSynchronize() kernelExtractValidExtensionTodExt in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
	//In mảng dExt;
	printf("\n********** dArrExt **********\n");
	cudaStatus =printdExt(hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt);
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n printdExt() in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}

	//kernelGetvivj<<<1,100>>>(hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt);
	/*hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
	status=-1;
	goto Error;
	}*/


	//Dựa vào dArrV để trích các cạnh duy nhất từ dArrExt và lưu vào dArrUniEdge tại vị trí tương ứng
	//Ở đây chỉ trích các mở rộng forward, vì nó chưa tồn tại mở rộng backward.
	int *dArrAllPossibleExtension =nullptr;
	int noElem_dArrAllPossibleExtension = Lv*Le;
	int noElem_dArrUniEdge=0;
	cudaStatus=hipMalloc((void**)&dArrAllPossibleExtension,noElem_dArrAllPossibleExtension*sizeof(int));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n hipMalloc((void**)&dArrAllPossibleExtension in extractUniExtension() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dArrAllPossibleExtension,0,noElem_dArrAllPossibleExtension*sizeof(int));
	}

	dim3 blockc(blocksize);
	dim3 gridc((hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem + blockc.x -1)/blockc.x);
	kernelForwardPossibleExtension_NonLast<<<gridc,blockc>>>(hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem,Lv,dArrAllPossibleExtension);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus !=hipSuccess){
		status =-1;
		fprintf(stderr,"\n kernelForwardPossibleExtension_NonLast  failed",cudaStatus);
		goto Error;
	}

	//Scan mảng dArrAllPossibleExtension để biết kích thước của mảng dArrUniEdge và ánh xạ từ vị trí trong dArrAllPossibleExtension thành nhãn để lưu vào dArrUniEdge được quản lý bởi hLevelUniEdge
	int *dArrAllPossibleExtensionScanResult =nullptr;
	cudaStatus = hipMalloc((void**)&dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension*sizeof(int));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n hipMalloc dArrAllPossibleExtensionScanResult  failed",cudaStatus);
		goto Error;
	}
	//cudaStatus = scanV(dArrAllPossibleExtension,noElem_dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult);
	//CHECK(cudaStatus);
	//if(cudaStatus!=hipSuccess){
	//	status = -1;
	//	fprintf(stderr,"\n scanV dArrAllPossibleExtension failed",cudaStatus);
	//	goto Error;
	//}
	myScanV(dArrAllPossibleExtension,noElem_dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult);
	//Tính kích thước của dArrUniEdge và lưu vào noElem_dArrUniEdge
	cudaStatus =getSizeBaseOnScanResult(dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension,noElem_dArrUniEdge);
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n scanV dArrAllPossibleExtension in extractUniExtension() failed",cudaStatus);
		goto Error;
	}

	//Hiển thị giá trị của noElem_dArrUniEdge
	printf("\n******noElem_dArrUniEdge************\n");
	printf("\n noElem_dArrUniEdge:%d",noElem_dArrUniEdge);

	hLevelUniEdge.at(idxLevel).vUE.at(idxEXT).noElem=noElem_dArrUniEdge;

	//Cấp phát bộ nhớ cho dArrUniEdge
	cudaStatus = hipMalloc((void**)&hLevelUniEdge.at(idxLevel).vUE.at(idxEXT).dArrUniEdge,noElem_dArrUniEdge*sizeof(UniEdge));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n hipMalloc dArrUniEdge  failed",cudaStatus);
		goto Error;
	}

	//lấy nhãn Li lưu vào biến dFromLi	
	int *dFromLi=nullptr;
	cudaStatus = hipMalloc((void**)&dFromLi,sizeof(int));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		printf("\n hipMalloc dFromLi failed");
		goto Error;
	}

	kernelGetFromLabel<<<1,1>>>(hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,dFromLi);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n  kernelGetFromLabel  failed");
		goto Error;
	}



	//Gọi hàm để ánh xạ dữ liệu từ dArrAllPossibleExtension sang mảng dArrUniEdge
	/* Input Data:	dArrAllPossibleExtension, dArrAllPossibleExtensionScanResult,  */
	dim3 blockd(blocksize);
	dim3 gridd((noElem_dArrAllPossibleExtension + blockd.x -1)/blockd.x);
	kernelFilldArrUniEdge<<<gridd,blockd>>>(dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension,hLevelUniEdge.at(idxLevel).vUE.at(idxEXT).dArrUniEdge,Lv,dFromLi);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n kernelFilldArrUniEdge failed",cudaStatus);
		goto Error;
	}

	//In nội dung mảng dArrUniEdge
	printf("\n**********dArrUniEdge************");
	displaydArrUniEdge(hLevelUniEdge.at(idxLevel).vUE.at(idxEXT).dArrUniEdge,noElem_dArrUniEdge);

	//Duyệt qua các cạnh duy nhất tính và lưu trữ độ hỗ trợ của chúng vào một mảng tạm nào đó
	//Sau đó trích những độ hỗ trợ thoả minsup vào lưu vào hLevelUniEdgeSatisfyMinsup
	//Chỉ cần quan tâm kết quả trả về gồm số lượng cạnh thoả minsup, cạnh đó là gì và độ hỗ trợ là bao nhiêu.
	status = computeSupportv2(hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem,hLevelUniEdge.at(idxLevel).vUE.at(idxEXT).dArrUniEdge,hLevelUniEdge.at(idxLevel).vUE.at(idxEXT).noElem,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).dArrUniEdge,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).hArrSupport);
	if(status!=0){
		goto Error;
	}



	//printf("\n************ dArrUniEdgeSatisfyMinSup*********\n");
	//printf("\n noElem:%d",hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem);
	//displaydArrUniEdge(hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).dArrUniEdge,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem);
	//for (int j = 0; j < hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem; j++)
	//{
	//	printf("\n Support: %d ",hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).hArrSupport[j]);
	//}

	status=Miningv2(hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).dArrUniEdge,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).hArrSupport,hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem,idxEXT);
	FUNCHECK(status);
	if(status!=0){
		goto Error;
	}


	CHECK(cudaStatus = hipFree(dFromLi));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus = hipFree(dArrAllPossibleExtension));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus = hipFree(dArrValidScanResult));	
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

Error:
	return cudaStatus;
}
__global__ void kernelmarkValidForwardEdge_LastExt(EXT* dArrExt, int noElemdArrExt,unsigned int Lv,int *dAllPossibleExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElemdArrExt){
		if(dArrExt[i].vi < dArrExt[i].vj){ //Chỉ xét các forward
		int index=	dArrExt[i].lij*Lv + dArrExt[i].lj;
		dAllPossibleExtension[index]=1;
		}
	}
}

__global__ void kernelmarkValidBackwardEdge_LastExt(EXT* dArrExt, int noElemdArrExt,unsigned int Lv,int *dAllPossibleExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElemdArrExt){
		if(dArrExt[i].vi > dArrExt[i].vj){ //Chỉ xét các forward
		int index=	dArrExt[i].lij*Lv + dArrExt[i].lj;
		dAllPossibleExtension[index]=1;
		}
	}
}


int PMS::markValidForwardEdge(EXT* dArrExt,int noElemdArrExt,unsigned int _Lv,int* dAllPossibleExtension){
	hipError_t cudaStatus;
	int status =0;

	dim3 block(blocksize);
	dim3 grid((noElemdArrExt+block.x-1)/block.x);

	kernelmarkValidForwardEdge_LastExt<<<grid,block>>>(dArrExt,noElemdArrExt,_Lv,dAllPossibleExtension);
	hipDeviceSynchronize();
	CHECK(cudaStatus=hipGetLastError());
	if (cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

Error:
	return status;
}

int PMS::markValidBackwardEdge(EXT* dArrExt,int noElemdArrExt,unsigned int _Lv,int* dAllPossibleExtension){
	hipError_t cudaStatus;
	int status =0;

	dim3 block(blocksize);
	dim3 grid((noElemdArrExt+block.x-1)/block.x);

	kernelmarkValidBackwardEdge_LastExt<<<grid,block>>>(dArrExt,noElemdArrExt,_Lv,dAllPossibleExtension);
	hipDeviceSynchronize();
	CHECK(cudaStatus=hipGetLastError());
	if (cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

Error:
	return status;
}


int PMS::extractUniqueForwardBackwardEdge_LastExt(EXTk ext,UniEdgek& ue){
	hipError_t cudaStatus;
	int status =0;

	UniEdgek fwEdgeTemp;
	UniEdgek bwEdgeTemp;
	int *dFromLi=nullptr;
	CHECK(cudaStatus = hipMalloc((void**)&dFromLi,sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	//Cập nhật Nhãn đỉnh được mở rộng
	kernelGetFromLabel<<<1,1>>>(ext.dArrExt,dFromLi);
	hipDeviceSynchronize();
	CHECK(cudaStatus=hipGetLastError());
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus = hipMemcpy(&fwEdgeTemp.Li,dFromLi,sizeof(int),hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus = hipMemcpy(&bwEdgeTemp.Li,dFromLi,sizeof(int),hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}


	//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
	int noElem_dallPossibleExtension=Le*Lv;
	int *d_allPossibleExtensionFW;
	int *d_allPossibleExtensionBW;

	//cấp phát bộ nhớ cho mảng d_allPossibleExtension
	CHECK(cudaStatus=hipMalloc((void**)&d_allPossibleExtensionFW,noElem_dallPossibleExtension*sizeof(int)));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(d_allPossibleExtensionFW,0,noElem_dallPossibleExtension*sizeof(int)));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}
	}

	CHECK(cudaStatus=hipMalloc((void**)&d_allPossibleExtensionBW,noElem_dallPossibleExtension*sizeof(int)));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(d_allPossibleExtensionBW,0,noElem_dallPossibleExtension*sizeof(int)));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}
	}

	int *d_allPossibleExtensionScanResultFW=nullptr;
	int *d_allPossibleExtensionScanResultBW=nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&d_allPossibleExtensionScanResultFW,noElem_dallPossibleExtension*sizeof(int)));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	CHECK(cudaStatus=hipMalloc((void**)&d_allPossibleExtensionScanResultBW,noElem_dallPossibleExtension*sizeof(int)));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}

	//Đánh dấu vị trí các mở rộng forward hợp lệ là 1 tại vị trí d_allPossibleExtension tương ứng
	FUNCHECK(status=markValidForwardEdge(ext.dArrExt,ext.noElem,Lv,d_allPossibleExtensionFW));
	if(status!=0){
		goto Error;
	}

	FUNCHECK(status=markValidBackwardEdge(ext.dArrExt,ext.noElem,Lv,d_allPossibleExtensionBW));
	if(status!=0){
		goto Error;
	}


	printf("\n************* d_AllPossibleExtensionFW ************\n");
	FUNCHECK(status=displayDeviceArr(d_allPossibleExtensionFW,noElem_dallPossibleExtension));
	if(status!=0){
		goto Error;
	}
	printf("\n************* d_AllPossibleExtensionBW ************\n");
	FUNCHECK(status=displayDeviceArr(d_allPossibleExtensionBW,noElem_dallPossibleExtension));
	if(status!=0){
		goto Error;
	}


	////Exclusive scan mảng d_allPossibleExtension và lưu kết quả vào mảng d_allPossibleExtensionScanResult
	////cudaStatus = scanV(d_allPossibleExtension,noElem_dallPossibleExtension,d_allPossibleExtensionScanResult);
	////if(cudaStatus!=hipSuccess){
	////	fprintf(stderr,"\n ScanV() in computeSupport() failed");
	////	status = -1;
	////	goto Error;
	////}

	CHECK(cudaStatus=myScanV(d_allPossibleExtensionFW,noElem_dallPossibleExtension,d_allPossibleExtensionScanResultFW));
	if(cudaStatus!=hipSuccess){
		goto Error;
	}

	CHECK(cudaStatus=myScanV(d_allPossibleExtensionBW,noElem_dallPossibleExtension,d_allPossibleExtensionScanResultBW));
	if(cudaStatus!=hipSuccess){
		goto Error;
	}

	printf("\n************* d_AllPossibleExtensionResultFW ************\n");
	FUNCHECK(status=displayDeviceArr(d_allPossibleExtensionScanResultFW,noElem_dallPossibleExtension));
	if(status!=0){
		goto Error;
	}
	printf("\n************* d_AllPossibleExtensionResultBW ************\n");
	FUNCHECK(status=displayDeviceArr(d_allPossibleExtensionScanResultBW,noElem_dallPossibleExtension));
	if(status!=0){
		goto Error;
	}

	int noElem_d_UniqueExtensionFW=0;
	int noElem_d_UniqueExtensionBW=0;
	////Tính kích thước của mảng d_UniqueExtension dựa vào kết quả exclusive scan
	CHECK(cudaStatus=getSizeBaseOnScanResult(d_allPossibleExtensionFW,d_allPossibleExtensionScanResultFW,noElem_dallPossibleExtension,noElem_d_UniqueExtensionFW));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	CHECK(cudaStatus=getSizeBaseOnScanResult(d_allPossibleExtensionBW,d_allPossibleExtensionScanResultBW,noElem_dallPossibleExtension,noElem_d_UniqueExtensionBW));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}

	printf("\n\nnoElem_d_UniqueExtension:%d",noElem_d_UniqueExtensionFW);
	printf("\n\nnoElem_d_UniqueExtension:%d",noElem_d_UniqueExtensionBW);
	
	fwEdgeTemp.noElem = noElem_d_UniqueExtensionFW;
	bwEdgeTemp.noElem = noElem_d_UniqueExtensionBW;



	//Tạo mảng dArrUniEdge với kích thước mảng vừa tính được
	CHECK(cudaStatus=hipMalloc((void**)&fwEdgeTemp.dArrUniEdge,fwEdgeTemp.noElem*sizeof(UniEdge)));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(fwEdgeTemp.dArrUniEdge,0,fwEdgeTemp.noElem*sizeof(UniEdge)));
		if(cudaStatus!=hipSuccess){
			status = -1;
			goto Error;
		}

	}

	CHECK(cudaStatus=hipMalloc((void**)&bwEdgeTemp.dArrUniEdge,bwEdgeTemp.noElem*sizeof(UniEdge)));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(bwEdgeTemp.dArrUniEdge,0,bwEdgeTemp.noElem*sizeof(UniEdge)));
		if(cudaStatus!=hipSuccess){
			status = -1;
			goto Error;
		}

	}
	////Ánh xạ ngược lại từ vị trí trong d_allPossibleExtension thành cạnh và lưu kết quả vào d_UniqueExtension
	//CHECK(cudaStatus=calcLabelAndStoreUniqueExtension(d_allPossibleExtensionFW,d_allPossibleExtensionScanResultFW,noElem_dallPossibleExtension,fwEdgeTemp.dArrUniEdge,noElem_d_UniqueExtensionFW,Le,Lv));
	//if(cudaStatus!=hipSuccess){
	//	status = -1;
	//	goto Error;
	//}

	dim3 blockd(blocksize);
	dim3 gridd((noElem_dallPossibleExtension + blockd.x -1)/blockd.x);
	kernelFilldArrUniEdge<<<gridd,blockd>>>(d_allPossibleExtensionFW,d_allPossibleExtensionScanResultFW,noElem_dallPossibleExtension,fwEdgeTemp.dArrUniEdge,Lv,dFromLi);
	kernelFilldArrUniEdge<<<gridd,blockd>>>(d_allPossibleExtensionBW,d_allPossibleExtensionScanResultBW,noElem_dallPossibleExtension,bwEdgeTemp.dArrUniEdge,Lv,dFromLi);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n kernelFilldArrUniEdge failed",cudaStatus);
		goto Error;
	}

	printf("\n *********** fwEdgeTemp.dArrUniEdge **********\n");
	FUNCHECK(status=displayArrUniEdge(fwEdgeTemp.dArrUniEdge,fwEdgeTemp.noElem));
	if(status!=0){
		goto Error;
	}

	printf("\n *********** bwEdgeTemp.dArrUniEdge **********\n");
	FUNCHECK(status=displayArrUniEdge(bwEdgeTemp.dArrUniEdge,fwEdgeTemp.noElem));
	if(status!=0){
		goto Error;
	}

	//chép kết quả fwEdgeTemp và bwEdgeTemp sang ue.
	FUNCHECK(status = cpResultToUE(fwEdgeTemp,bwEdgeTemp,dFromLi,ue));
	if(status!=0){
		goto Error;
	}

	ue.firstIndexForwardExtension = bwEdgeTemp.noElem;

	printf("\n *********** ue.dArrUniEdge **********\n");
	FUNCHECK(status=displayArrUniEdge(ue.dArrUniEdge,ue.noElem));
	if(status!=0){
		goto Error;
	}

	//Giai phong bo nho
	CHECK(cudaStatus=hipFree(fwEdgeTemp.dArrUniEdge));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	CHECK(cudaStatus=hipFree(bwEdgeTemp.dArrUniEdge));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}


	CHECK(cudaStatus=hipFree(dFromLi));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}


	CHECK(cudaStatus=hipFree(d_allPossibleExtensionFW));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus=hipFree(d_allPossibleExtensionScanResultFW));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	CHECK(cudaStatus=hipFree(d_allPossibleExtensionBW));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	CHECK(cudaStatus=hipFree(d_allPossibleExtensionScanResultBW));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

Error:
	return status;
}

int PMS::cpResultToUE(UniEdgek fwEdgeTemp,UniEdgek bwEdgeTemp,int *dFromLi,UniEdgek& ue){
	int status = 0;
	hipError_t cudaStatus;

	ue.noElem = fwEdgeTemp.noElem + bwEdgeTemp.noElem;

	CHECK(cudaStatus = hipMalloc((void**)&ue.dArrUniEdge,sizeof(UniEdge)*ue.noElem));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}

	CHECK(cudaStatus = hipMemcpy(&ue.Li,dFromLi,sizeof(int),hipMemcpyDeviceToHost));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	printf("\n *********** fwEdgeTemp.dArrUniEdge **********\n");
	FUNCHECK(status=displayArrUniEdge(fwEdgeTemp.dArrUniEdge,fwEdgeTemp.noElem));
	if(status!=0){
		goto Error;
	}

	printf("\n *********** bwEdgeTemp.dArrUniEdge **********\n");
	FUNCHECK(status=displayArrUniEdge(bwEdgeTemp.dArrUniEdge,fwEdgeTemp.noElem));
	if(status!=0){
		goto Error;
	}



	dim3 block(blocksize);
	dim3 grid((ue.noElem + block.x -1)/block.x);
	kernelCopyResultToUE<<<grid,block>>>(fwEdgeTemp.dArrUniEdge,bwEdgeTemp.dArrUniEdge,bwEdgeTemp.noElem,ue.dArrUniEdge,ue.noElem);
	hipDeviceSynchronize();
	CHECK(cudaStatus = hipGetLastError());
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
Error:
	return status;
}



__global__ void kernelCopyResultToUE(UniEdge *fwdArrUniEdge,UniEdge *bwdArrUniEdge,int bwnoElem,UniEdge *uedArrUniEdge,int uenoElem){
	int i = blockDim.x * blockIdx.x  + threadIdx.x;
	if(i<uenoElem){
		if(i<bwnoElem){
			uedArrUniEdge[i].li=bwdArrUniEdge[i].li;
			uedArrUniEdge[i].lij=bwdArrUniEdge[i].lij;
			uedArrUniEdge[i].lj=bwdArrUniEdge[i].lj;
		}
		else
		{
			uedArrUniEdge[i].li=fwdArrUniEdge[i-bwnoElem].li;
			uedArrUniEdge[i].lij=fwdArrUniEdge[i-bwnoElem].lij;
			uedArrUniEdge[i].lj=fwdArrUniEdge[i-bwnoElem].lj;
		}
	}
}



int PMS::extractValidExtensionTodExtv2(EXT *dArrExtension,V *dArrV,int noElem_dArrV,int idxEXT){
	hipError_t cudaStatus;
	int status =0;
	//2. Scan mảng dArrValid để lấy kích thước của mảng cần tạo
	int *dArrValidScanResult = nullptr;

	CHECK(cudaStatus = hipMalloc((void**)&dArrValidScanResult,sizeof(int)*noElem_dArrV));
	if (cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus = hipMemset(dArrValidScanResult,0,sizeof(int)*noElem_dArrV));
		if (cudaStatus!=hipSuccess){
			status = -1;
			goto Error;
		}
	}

	//cudaStatus = scanV(dArrV->valid,noElem_dArrV,dArrValidScanResult); //hàm scan này có vấn đề. Nó làm thay đổi giá trị đầu vào.
	//if (cudaStatus!=hipSuccess){
	//	status = -1;
	//	fprintf(stderr,"\n scanV dArrValid in extractValidExtensionToExt() failed");
	//	goto Error;
	//}

	CHECK(cudaStatus = myScanV(dArrV->valid,noElem_dArrV,dArrValidScanResult));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	//In nội dung kết quả dArrValidScanResult
	printf("\n********dArrValid******\n");
	displayDeviceArr(dArrV->valid,noElem_dArrV);

	printf("\n********dArrValidScanResult******\n");
	displayDeviceArr(dArrValidScanResult,noElem_dArrV);

	////3. Lấy kích thước của mảng dArrExt;
	int noElem_dExt=0;
	cudaStatus=getSizeBaseOnScanResult(dArrV->valid,dArrValidScanResult,noElem_dArrV,noElem_dExt);
	if (cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n getSizeBaseOnScanResult in extractValidExtensionToExt() failed");
		goto Error;
	}

	//In nội dung noElem_dExt
	printf("\n******** noElem In dArrExt ******\n");
	printf("\n noElem_dExt:%d",noElem_dExt);
	hLevelEXTv2.at(idxLevel).vE.at(idxEXT).noElem = noElem_dExt;
	//**************** Nếu không tìm được mở rộng nào thì return *************/
	if (noElem_dExt == 0) 
	{
		hipFree(dArrValidScanResult);
		return status;
	}
	//Nếu tìm được mở rộng thì xây dựng EXTk, rồi trích các mở rộng duy nhất và tính độ hỗ trợ của chúng. Đồng thời
	//lọc ra các độ hỗ trợ thoả minsup
	//Quản lý theo Level
	//4. Khởi tạo mảng dArrExt có kích thước noElem_dExt rồi trích dữ liệu từ dArrExtension sang dựa vào dArrValid.
	hLevelEXTv2.at(idxLevel).vE.at(idxEXT).noElem = noElem_dExt;
	CHECK(cudaStatus = hipMalloc((void**)&hLevelEXTv2.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt*sizeof(EXT)));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus = hipMemset(hLevelEXTv2.at(idxLevel).vE.at(idxEXT).dArrExt,0,sizeof(EXT)*noElem_dExt));
		if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
		}
	}
	dim3 blockb(blocksize);
	dim3 gridb((noElem_dArrV+blockb.x -1)/blockb.x);
	kernelExtractValidExtensionTodExt<<<gridb,blockb>>>(dArrExtension,dArrV->valid,dArrValidScanResult,noElem_dArrV,hLevelEXTv2.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt);
	hipDeviceSynchronize();
	CHECK(cudaStatus = hipGetLastError());
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	////In mảng dExt;
	printf("\n********** dArrExt **********\n");
	CHECK(cudaStatus =printdExt(hLevelEXTv2.at(idxLevel).vE.at(idxEXT).dArrExt,noElem_dExt));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}

	//Chuẩn bị bộ nhớ để trích các cạnh duy nhất
	//Trích các mở rộng duy nhất forward: lưu chúng vào mảng dUniEdgeForwardTemp
	//Các tham số:1. dArrExt: để trích các forward uniedge extension
	//hLevelUniEdgev2.at(idxLevel).vUE.at(0); //chúng ta lưu vào đây, vì sao? vì đây là hàm khai thác các backward và forward edge ở EXT cuối.
	UniEdgek tempUniEdgeBF; //khai báo cấu trúc chứa các cạnh duy nhất backward và forward
	FUNCHECK(status=extractUniqueForwardBackwardEdge_LastExt(hLevelEXTv2.at(idxLevel).vE.at(idxEXT),hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT)));
	if(status!=0){
		goto Error;
	}

	printf("\n ************ dArrUniEdge from hLevelUniEdgev2 ***************\n");
	printf("\n noElem:%d",hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).noElem);
	printf("\n Li:%d",hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).Li);
	printf("\n firstIndex of forward extension:%d",hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).firstIndexForwardExtension);
	displayArrUniEdge(hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).dArrUniEdge,hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).noElem);

	//Duyệt qua mảng dUniEdge để tính độ hỗ trợ. 
	//Như vậy, trong trường hợp này chúng ta sẽ tính độ hỗ trợ cho các mở rộng backward trước nếu có, rồi mới đến forward.
	//vì backward nằm trước forward extension trong mảng dArrUniEdge
	FUNCHECK(status = computeSupportv2(hLevelEXTv2.at(idxLevel).vE.at(idxEXT).dArrExt,hLevelEXTv2.at(idxLevel).vE.at(idxEXT).noElem,hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).dArrUniEdge,hLevelUniEdgev2.at(idxLevel).vUE.at(idxEXT).noElem,hLevelUniEdgeSatisfyMinsupv2.at(idxLevel).vecUES.at(idxEXT).noElem,hLevelUniEdgeSatisfyMinsupv2.at(idxLevel).vecUES.at(idxEXT).dArrUniEdge,hLevelUniEdgeSatisfyMinsupv2.at(idxLevel).vecUES.at(idxEXT).hArrSupport));
	if(status!=0){
		goto Error;
	}


	////Sau đó trích những độ hỗ trợ thoả minsup vào lưu vào hLevelUniEdgeSatisfyMinsup
	////Chỉ cần quan tâm kết quả trả về gồm số lượng cạnh thoả minsup, cạnh đó là gì và độ hỗ trợ là bao nhiêu.



	////printf("\n************ dArrUniEdgeSatisfyMinSup*********\n");
	////printf("\n noElem:%d",hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem);
	////displaydArrUniEdge(hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).dArrUniEdge,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem);
	////for (int j = 0; j < hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem; j++)
	////{
	////	printf("\n Support: %d ",hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).hArrSupport[j]);
	////}

	////status=Miningv2(hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).noElem,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).dArrUniEdge,hLevelUniEdgeSatisfyMinsup.at(idxLevel).vecUES.at(idxEXT).hArrSupport,hLevelEXT.at(idxLevel).vE.at(idxEXT).dArrExt,hLevelEXT.at(idxLevel).vE.at(idxEXT).noElem,idxEXT);
	////FUNCHECK(status);
	////if(status!=0){
	////	goto Error;
	////}


	////CHECK(cudaStatus = hipFree(dFromLi));
	////if(cudaStatus!=hipSuccess){
	////	status=-1;
	////	goto Error;
	////}

	////CHECK(cudaStatus = hipFree(dArrAllPossibleExtension));
	////if(cudaStatus!=hipSuccess){
	////	status=-1;
	////	goto Error;
	////}

	////CHECK(cudaStatus = hipFree(dArrValidScanResult));	
	////if(cudaStatus!=hipSuccess){
	////	status=-1;
	////	goto Error;
	////}

Error:
	return cudaStatus;
}


int PMS::computeSupportv2(EXT *dArrExt,int noElemdArrExt,UniEdge *dArrUniEdge,int noElemdArrUniEdge,int &noElem,UniEdge *&dArrUniEdgeSup,int *&hArrSupport){
	int status=0;
	hipError_t cudaStatus;

#pragma region "find Boundary and scan Boundary"
	int *dArrBoundary=nullptr;
	int noElemdArrBoundary = noElemdArrExt;
	CHECK(cudaStatus=hipMalloc((void**)&dArrBoundary,sizeof(int)*noElemdArrBoundary));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(dArrBoundary,0,sizeof(int)*noElemdArrBoundary));
		if(cudaStatus!=hipSuccess){
			status =-1;
			goto Error;
		}
	}

	int *dArrBoundaryScanResult=nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&dArrBoundaryScanResult,sizeof(int)*noElemdArrBoundary));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus=hipMemset(dArrBoundaryScanResult,0,sizeof(int)*noElemdArrBoundary));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}
	}

	//Tìm boundary của EXTk và lưu kết quả vào mảng dArrBoundary
	FUNCHECK(status = findBoundary(dArrExt,noElemdArrExt,dArrBoundary));
	if(status!=0){
		goto Error;
	}

	printf("\n ************* dArrBoundary ************\n");
	FUNCHECK(status=displayDeviceArr(dArrBoundary,noElemdArrExt));
	if(status!=0){
		goto Error;
	}


	//Scan dArrBoundary lưu kết quả vào dArrBoundaryScanResult
	//cudaStatus=scanV(dArrBoundary,noElemdArrBoundary,dArrBoundaryScanResult);
	//CHECK(cudaStatus);
	//if(cudaStatus!=hipSuccess){
	//	status = -1;
	//	fprintf(stderr,"\n Exclusive scan dArrBoundary in computeSupportv2() failed",cudaStatus);
	//	goto Error;
	//}

	CHECK(cudaStatus = myScanV(dArrBoundary,noElemdArrBoundary,dArrBoundaryScanResult));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	printf("\n**************dArrBoundaryScanResult****************\n");
	FUNCHECK(status=displayDeviceArr(dArrBoundaryScanResult,noElemdArrBoundary));
	if(status!=0){
		goto Error;
	}


	//Tính support của cạnh duy nhất.
	float *dF=nullptr; //khai báo mảng dF
	int noElemdF = 0; //Số lượng phần tử của mảng dF

	CHECK(cudaStatus = hipMemcpy(&noElemdF,&dArrBoundaryScanResult[noElemdArrBoundary-1],sizeof(int),hipMemcpyDeviceToHost));
	if(cudaStatus !=hipSuccess){
		status =-1;
		goto Error;
	}
	noElemdF++; //Phải tăng lên 1 vì giá trị hiện tại chỉ là chỉ số của mảng
	printf("\n*****noElemdF******\n");
	printf("noElemdF:%d",noElemdF);

	//Cấp phát bộ nhớ trên device cho mảng dF
	CHECK(cudaStatus = hipMalloc((void**)&dF,sizeof(float)*noElemdF));
	if(cudaStatus!=hipSuccess){
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(cudaStatus = hipMemset(dF,0,sizeof(float)*noElemdF));
		if(cudaStatus!=hipSuccess){
			status =-1;
			goto Error;
		}
	}
#pragma endregion "end of finding Boundary"

	//Tạm thời chứa độ hỗ trợ của tất cả các cạnh duy nhất.
	//Sau đó, trích những cạnh và độ hỗ trợ thoả minsup vào hLevelUniEdgeSatisfyMinsup tại level tương ứng
	int *hArrSupportTemp = (int*)malloc(sizeof(int)*noElemdArrUniEdge);
	if(hArrSupportTemp==NULL){
		status =-1;
		printf("\n Malloc hArrSupportTemp in computeSupportv2() failed");
		goto Error;
	}
	else
	{
		memset(hArrSupportTemp,0,sizeof(unsigned int)*noElemdArrUniEdge);
	}
	//		//Duyệt và tính độ hỗ trợ của các cạnh
	dim3 blocke(blocksize);
	dim3 gride((noElemdArrExt+blocke.x-1)/blocke.x);

	//printf("\n**********dArrUniEdge************");				
	//displaydArrUniEdge(dArrUniEdge,noElemdArrUniEdge);

	for (int i = 0; i < noElemdArrUniEdge; i++)
	{					
		float support=0;
		kernelFilldF<<<gride,blocke>>>(dArrUniEdge,i,dArrExt,noElemdArrExt,dArrBoundaryScanResult,dF);

		hipDeviceSynchronize();
		CHECK(cudaStatus = hipGetLastError());
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}				

		printf("\n**********dF****************\n");
		displayDeviceArr(dF,noElemdF);

		CHECK(cudaStatus = reduction(dF,noElemdF,support));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}

		printf("\n******support********");
		printf("\n Support:%f",support);

		CHECK(cudaStatus = hipMemset(dF,0,noElemdF*sizeof(float)));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}

		hArrSupportTemp[i]=support;
	}
	printf("\n************hArrSupportTemp**********\n");
	for (int j = 0; j < noElemdArrUniEdge; j++)
	{
		printf("j[%d]:%d ",j,hArrSupportTemp[j]);
	}

	//Tiếp theo là lọc giữ lại cạnh và độ hỗ trợ thoả minsup
	status = extractUniEdgeSatisfyMinsupV2(hArrSupportTemp,dArrUniEdge,noElemdArrUniEdge,minsup,noElem,dArrUniEdgeSup,hArrSupport);
	FUNCHECK(status)
		if(status!=0){
			goto Error;
		}


		free(hArrSupportTemp);	

		CHECK(cudaStatus =hipFree(dArrBoundary));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}

		CHECK(cudaStatus =hipFree(dArrBoundaryScanResult));
		if(cudaStatus!=hipSuccess){
			status=-1;
			goto Error;
		}

Error:	
		return status;
}

int PMS::extractUniEdgeSatisfyMinsupV2(int *hResultSup,UniEdge *dArrUniEdge,int noElemUniEdge,unsigned int minsup,int &noElem,UniEdge *&dArrUniEdgeSup,int *&hArrSupport){
	int status=0;
	hipError_t cudaStatus;
	//1. Cấp phát mảng trên device có kích thước bằng noElemUniEdge
	int *dResultSup=nullptr;
	CHECK(cudaStatus =hipMalloc((void**)&dResultSup,noElemUniEdge*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	CHECK(cudaStatus = hipMemcpy(dResultSup,hResultSup,noElemUniEdge*sizeof(int),hipMemcpyHostToDevice));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}

	printf("\n *******dResultSup********\n");
	FUNCHECK(status = displayDeviceArr(dResultSup,noElemUniEdge));
	if(status != 0){
		goto Error;
	}


	//2. Đánh dấu 1 trên dV cho những phần tử thoả minsup
	int *dV=nullptr;
	CHECK(cudaStatus =hipMalloc((void**)&dV,noElemUniEdge*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}
	CHECK(cudaStatus =hipMemset(dV,0,sizeof(int)*noElemUniEdge));
	if(cudaStatus!=hipSuccess){
		status = -1;
		goto Error;
	}

	dim3 block(blocksize);
	dim3 grid((noElemUniEdge + block.x - 1)/block.x);
	kernelMarkUniEdgeSatisfyMinsup<<<grid,block>>>(dResultSup,noElemUniEdge,dV,minsup);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelMarkUniEdgeSatisfyMinsup in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		status = -1;
		goto Error;
	}

	printf("\n ***********dV**********\n");
	FUNCHECK(status = displayDeviceArr(dV,noElemUniEdge));
	if(status!=0){
		goto Error;
	}


	int *dVScanResult=nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&dVScanResult,noElemUniEdge*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	//CHECK(cudaStatus = scanV(dV,noElemUniEdge,dVScanResult));
	//if(cudaStatus!=hipSuccess){
	//	status=-1;
	//	goto Error;
	//}

	myScanV(dV,noElemUniEdge,dVScanResult);

	printf("\n ***********dVScanResult**********\n");
	displayDeviceArr(dVScanResult,noElemUniEdge);

	CHECK(cudaStatus=getSizeBaseOnScanResult(dV,dVScanResult,noElemUniEdge,noElem));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	CHECK(cudaStatus=hipMalloc((void**)&dArrUniEdgeSup,noElem*sizeof(UniEdge)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	hArrSupport = (int*)malloc(sizeof(int)*noElem);
	if (hArrSupport ==NULL){
		status =-1;
		printf("\n malloc hArrSup of hUniEdgeSatisfyMinsup failed()");
		goto Error;
	}


	int *dSup=nullptr;
	CHECK(cudaStatus=hipMalloc((void**)&dSup,noElem*sizeof(int)));
	if(cudaStatus!=hipSuccess){
		status=-1;
		goto Error;
	}

	dim3 blocka(blocksize);
	dim3 grida((noElemUniEdge + blocka.x -1)/blocka.x);
	kernelExtractUniEdgeSatifyMinsup<<<grida,blocka>>>(dArrUniEdge,dV,dVScanResult,noElemUniEdge,dArrUniEdgeSup,dSup,dResultSup);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelExtractUniEdgeSatisfyMinsup() in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		goto Error;
	}
	printf("\n ********hUniEdgeSatisfyMinsup.dUniEdge****************\n");
	FUNCHECK(status=displayArrUniEdge(dArrUniEdgeSup,noElem));
	if(status!=0){
		goto Error;
	}

	printf("\n ********hUniEdgeSatisfyMinsup.dSup****************\n");
	displayDeviceArr(dSup,noElem);

	CHECK(cudaStatus = hipMemcpy(hArrSupport,dSup,sizeof(int)*noElem,hipMemcpyDeviceToHost));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelExtractUniEdgeSatisfyMinsup() in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		goto Error;
	}

	for (int i = 0; i < noElem; i++)
	{
		printf("\n hArrSupport:%d ",hArrSupport[i]);
	}

	hipFree(dResultSup);
	hipFree(dV);
	hipFree(dVScanResult);
	hipFree(dSup);
Error:
	return status;
}


__global__ void printdArrUniEdge(UniEdge *dArrUniEdge,int pos){
	printf("\n d[%d]: (li,lij,lj):(%d %d %d)",pos,dArrUniEdge[pos].li,dArrUniEdge[pos].lij,dArrUniEdge[pos].lj);
}


__global__ void kernelFilldF(UniEdge *dArrUniEdge,int pos,EXT *dArrExt,int noElemdArrExt,int *dArrBoundaryScanResult,float *dF){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElemdArrExt){
		int li = dArrUniEdge[pos].li;
		int lij = dArrUniEdge[pos].lij;
		int lj = dArrUniEdge[pos].lj;
		int Li = dArrExt[i].li;
		int Lij = dArrExt[i].lij;
		int Lj = dArrExt[i].lj;
		if(li==Li && lij==Lij && lj==Lj){
			dF[dArrBoundaryScanResult[i]]=1;
		}
		printf("\nThread %d: UniEdge(li:%d lij:%d lj:%d) (Li:%d Lij:%d Lj:%d idxdF:%d dF:%d)",i,li,lij,lj,Li,Lij,Lj,dArrBoundaryScanResult[i],dF[dArrBoundaryScanResult[i]]);
	}
}


int PMS::findBoundary(EXT *dArrExt,int noElemdArrExt,int *&dArrBoundary){
	int status =0;
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElemdArrExt+block.x-1)/block.x);

	kernelfindBoundary<<<grid,block>>>(dArrExt,noElemdArrExt,dArrBoundary,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelfindBoundary in findBoundary() failed",cudaStatus);
		goto Error;
	}
Error:	
	return status;
}

__global__ void kernelfindBoundary(EXT *dArrExt,int noElemdArrExt,int *dArrBoundary,unsigned int maxOfVer){
	int i = blockDim.x*blockIdx.x + threadIdx.x;	
	if(i<noElemdArrExt-1){		
		unsigned int graphIdAfter=dArrExt[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=dArrExt[i].vgi/maxOfVer;
		if(graphIdAfter!=graphIdCurrent){
			dArrBoundary[i]=1;
		}
	}
}


int PMS::forwardExtension(int idxhEXTk,int *listOfVer,int noElemEmbedding,int fromRMP){
	int status = 0;
	int lastCol = hEmbedding.size() - 1;

	dim3 block(blocksize);
	dim3 grid((noElemEmbedding + block.x -1)/block.x);

	//Tìm bậc lớn nhất của các đỉnh cần mở rộng trong listOfVer
	int maxDegreeOfVer=0;
	float *dArrDegreeOfVid=nullptr; //chứa cậc của các đỉnh trong listOfVer, dùng để duyệt qua các đỉnh lân cận
	//trong csdl
	status=findMaxDegreeOfVer(listOfVer,maxDegreeOfVer,dArrDegreeOfVid,noElemEmbedding); //tìm bậc lớn nhất
	FUNCHECK(status);
	if(status==-1){
		printf("\n findMaxDegreeOfVer() in forwardExtension() failed");
		goto Error;
	}
	//Tạo mảng dArrV có số lượng phần tử bằng số lượng embedding nhân với bậc lớn nhất của các vid vừa tìm được
	//Tạo mảng d_arr_V có kích thước: maxDegree_vid_Q * |Q|
	//	Lưu ý, mảng d_arr_V phải có dạng cấu trúc đủ thể hiện cạnh mở rộng có hợp lệ hay không và là forward extension hay backward extension.
	//	struct struct_V
	//	{
	//		int valid; //default: 0, valid: 1
	//		int backward; //default: 0- forward; backward: 1
	//	}

	V *dArrV=nullptr;
	dArrV = (V*)malloc(sizeof(V));

	dArrV->noElem =maxDegreeOfVer*noElemEmbedding;
	hipError_t cudaStatus=hipMalloc((void**)&(dArrV->valid),(dArrV->noElem)*sizeof(int));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrV in  failed");
		status =-1;
		goto Error;
	}
	else
	{
		cudaStatus = hipMemset(dArrV->valid,0,(dArrV->noElem)*sizeof(int));
		CHECK(cudaStatus);
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}
	}
	cudaStatus=hipMalloc((void**)&(dArrV->backward),(dArrV->noElem)*sizeof(int));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrV in  failed");
		status =-1;
		goto Error;
	}
	else
	{
		cudaStatus=hipMemset(dArrV->backward,0,(dArrV->noElem)*sizeof(int));
		CHECK(cudaStatus);
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}
	}


	////Các mở rộng hợp lệ sẽ được ghi nhận vào mảng dArrV, đồng thời thông tin của cạnh mở rộng gồm dfscode, vgi, vgj và row pointer của nó cũng được xây dựng
	////và lưu trữ trong mảng EXT *dExtensionTemp, mảng này có số lượng phần tử bằng với mảng dArrV. Sau đó chúng ta sẽ rút trích những mở rộng hợp lệ này và lưu vào dExt. 
	////Để xây dựng dfscode (vi,vj,li,lij,lj) thì chúng ta cần:
	//// - Dựa vào giá trị của right most path để xác định vi
	//// - Dựa vào maxid để xác định vj
	//// - Dựa vào CSDL để xác định các thành phần còn lại.
	////Chúng ta có thể giải phóng bộ nhớ của dExtensionTemp sau khi đã trích các mở rộng hợp lệ thành công.


	EXT *dArrExtensionTemp= nullptr; //Nơi lưu trữ tạm thời tất cả các cạnh mở rộng. Sau đó, chúng sẽ được lọc ra các mở rộng hợp lệ sang EXTk tương ứng.
	cudaStatus = hipMalloc((void**)&dArrExtensionTemp,(dArrV->noElem)*sizeof(EXT));
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status=-1;
		fprintf(stderr,"\n hipMalloc dArrExtensionTemp forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}
	else
	{
		cudaStatus=hipMemset(dArrExtensionTemp,0,dArrV->noElem*sizeof(EXT));
		CHECK(cudaStatus);
		if(cudaStatus !=hipSuccess){
			status =-1;
			goto Error;
		}
	}

	printf("\nnoElem_dArrV:%d",dArrV->noElem );


	////Gọi kernel với các đối số: CSDL, bậc của các đỉnh, dArrV, dArrExtension,noElem_Embedding,maxDegreeOfVer,idxQ,dArrPointerEmbedding,minLabel,maxid
	dim3 blocka(blocksize);
	dim3 grida((noElemEmbedding+block.x - 1)/blocka.x);
	//hdb.at(0).dN;
	//int noElemdArrPointerEmbedding = lastCol+1;
	//kernel tìm các mở rộng forward hợp lệ	

	kernelFindValidForwardExtension<<<grida,blocka>>>(hLevelPtrEmbedding.at(idxLevel).dArrPointerEmbedding,hLevelPtrEmbedding.at(idxLevel).noElem,hLevelPtrEmbedding.at(idxLevel).noElemEmbedding,hdb.at(0).dO,hdb.at(0).dLO,hdb.at(0).dN,hdb.at(0).dLN,dArrDegreeOfVid,maxDegreeOfVer,dArrExtensionTemp,listOfVer,minLabel,maxId,fromRMP,dArrV->valid,dArrV->backward);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelFindValidForwardExtension in forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}
	//In mảng dArrV để kiểm tra thử
	printf("\n****************dArrV_valid*******************\n");
	displayDeviceArr(dArrV->valid,dArrV->noElem);
	////Chép kết quả từ dArrExtension sang dExt
	//chúng ta cần có mảng dArrV để trích các mở rộng duy nhất
	//Hàm này cũng gọi hàm trích các mở rộng duy nhất và tính độ hỗ trợ của chúng
	displayDeviceEXT(dArrExtensionTemp,dArrV->noElem);
	status = extractValidExtensionTodExt(dArrExtensionTemp,dArrV,dArrV->noElem,idxhEXTk);
	FUNCHECK(status);
	if(status!=0){
		fprintf(stderr,"\n extractValidExtensionTodExt() in forwardExtensionQ() failed");
		goto Error;
	}


	hipFree(dArrV->valid);
	hipFree(dArrV->backward);
	free(dArrV);
	hipFree(dArrDegreeOfVid);
Error:
	return status;
}

__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cache[256];


	float temp = -1.0;
	while(index + offset < n){
		temp = fmaxf(temp, array[index + offset]);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}


int PMS::findMaxDegreeOfVer(int *listOfVer,int &maxDegreeOfVer,float *&dArrDegreeOfVid,int noElem){
	int status = 0;
	FUNCHECK(status = findDegreeOfVer(listOfVer,dArrDegreeOfVid,noElem));
	if(status == -1){
		printf("\n findDegreeOfVer() in findMaxDegreeOfVer() faild");
		goto Error;
	}

	printf("\n*******dArrDegreeOfVid*************\n");
	displayDeviceArr(dArrDegreeOfVid,noElem);

	//Tìm bậc lớn nhất và lưu kết quả vào biến maxDegreeOfVer
	float *h_max;
	h_max = (float*)malloc(sizeof(float));
	if(h_max==NULL){
		printf("\nMalloc h_max failed");
		status = -1;
		FUNCHECK(status);
		goto Error;
	}

	float *d_max;
	int *d_mutex;
	hipError_t cudaStatus=hipMalloc((void**)&d_max,sizeof(float));
	CHECK(cudaStatus);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_max failed",cudaStatus);
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(d_max,0,sizeof(float)));
	}

	cudaStatus=hipMalloc((void**)&d_mutex,sizeof(int));
	CHECK(cudaStatus);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_mutex failed");
		status = -1;
		goto Error;
	}
	else
	{
		hipMemset(d_mutex,0,sizeof(int));
	}

	dim3 gridSize = 256;
	dim3 blockSize = 256;
	find_maximum_kernel<<<gridSize, blockSize>>>(dArrDegreeOfVid, d_max, d_mutex, noElem);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n find_maximum_kernel in findMaxDegreeOfVer() failed");
		status =-1;
		goto Error;
	}

	// copy from device to host
	CHECK(hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost));

	//report results
	maxDegreeOfVer = (int)(*h_max); //bậc lớn nhất của các đỉnh trong 1 cột Q
	printf("\nMax degree of vid in Q column is: %d",maxDegreeOfVer);

	hipFree(d_max);
	hipFree(d_mutex);
	free(h_max);
Error:
	return status;
}
//__global__ void kernelCalDegreeOfVid(Embedding *dArrEmbedding,int *d_O, int numberOfElementd_O,int noElem_Embedding,int numberOfElementd_N,unsigned int maxOfVer,float *dArrDegreeOfVid){
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	if(i<noElem_Embedding){
//		int vid = dArrEmbedding[i].vid;
//		float degreeOfV =0;
//		int nextVid;
//		int graphid;
//		int lastGraphId=(numberOfElementd_O-1)/maxOfVer;
//		if (vid==numberOfElementd_O-1){ //nếu như đây là đỉnh cuối cùng trong d_O
//			degreeOfV=numberOfElementd_N-d_O[vid]; //thì bậc của đỉnh vid chính bằng tổng số cạnh trừ cho giá trị của d_O[vid].
//		}
//		else
//		{
//			nextVid = vid+1; //xét đỉnh phía sau có khác 1 hay không?
//			graphid=vid/maxOfVer;
//			if(d_O[nextVid]==-1 && graphid==lastGraphId){
//				degreeOfV=numberOfElementd_N-d_O[vid];
//			}
//			else if(d_O[nextVid]==-1 && graphid!=lastGraphId){
//				nextVid=(graphid+1)*maxOfVer;
//				degreeOfV=d_O[nextVid]-d_O[vid];
//			}
//			else
//			{
//				degreeOfV=d_O[nextVid]-d_O[vid];
//			}							
//		}
//		dArrDegreeOfVid[i]=degreeOfV;
//	}
//}

__global__ void kernelCalDegreeOfVid(int *listOfVer,int *d_O, int numberOfElementd_O,int noElem_Embedding,int numberOfElementd_N,unsigned int maxOfVer,float *dArrDegreeOfVid){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_Embedding){
		int vid = listOfVer[i];
		float degreeOfV =0;
		int nextVid;
		int graphid;
		int lastGraphId=(numberOfElementd_O-1)/maxOfVer;
		if (vid==numberOfElementd_O-1){ //nếu như đây là đỉnh cuối cùng trong d_O
			degreeOfV=numberOfElementd_N-d_O[vid]; //thì bậc của đỉnh vid chính bằng tổng số cạnh trừ cho giá trị của d_O[vid].
		}
		else
		{
			nextVid = vid+1; //xét đỉnh phía sau có khác 1 hay không?
			graphid=vid/maxOfVer;
			if(d_O[nextVid]==-1 && graphid==lastGraphId){
				degreeOfV=numberOfElementd_N-d_O[vid];
			}
			else if(d_O[nextVid]==-1 && graphid!=lastGraphId){
				nextVid=(graphid+1)*maxOfVer;
				degreeOfV=d_O[nextVid]-d_O[vid];
			}
			else
			{
				degreeOfV=d_O[nextVid]-d_O[vid];
			}							
		}
		dArrDegreeOfVid[i]=degreeOfV;
	}
}


int PMS::findDegreeOfVer(int *listOfVer,float *&dArrDegreeOfVid,int noElem_Embedding){
	int status = 0;
	hipError_t cudaStatus;
	CHECK(cudaStatus =hipMalloc((void**)&dArrDegreeOfVid,noElem_Embedding*sizeof(float)));
	if(cudaStatus !=hipSuccess){
		status =-1;
		goto Error;
	}

	CHECK(cudaStatus = hipMemset(dArrDegreeOfVid,0,noElem_Embedding*sizeof(float)));
	if(cudaStatus !=hipSuccess){
		status =-1;
		goto Error;
	}

	dim3 block(blocksize);
	dim3 grid((noElem_Embedding + block.x -1)/block.x);
	//Đầu vào của kernelCalDegreeOfVid là một tập đỉnh trên RMP kèm theo Embedding Header của nó.
	kernelCalDegreeOfVid<<<grid,block>>>(listOfVer,hdb.at(0).dO, hdb.at(0).noElemdO,noElem_Embedding,hdb.at(0).noElemdN, maxOfVer,dArrDegreeOfVid);	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();	
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelCalDegreeOfVid in findDegreeOfVer() failed",cudaStatus);
		status =-1;
		goto Error;
	}

Error:
	return status;

}

