#include "hip/hip_runtime.h"
#pragma once
#include "pms.cuh"


float hTime=0.0;
float dTime=0.0;

PMS::PMS(){
	Lv=0;
	Le=0;
	maxOfVer=0;
	numberOfGraph=0;

	//std::cout<<" PMS initialized " << std::endl;
	//char* outfile;
	//outfile = "/result.graph";
	//fos.open(outfile);	
}
PMS::~PMS(){
	//std::cout<<" PMS terminated " << std::endl;
	//fos.close();

	if(hdb.size()!=0){
		for (int i = 0; i < hdb.size(); i++)
		{
			hipFree(hdb.at(i).dO);
			hipFree(hdb.at(i).dLO);
			hipFree(hdb.at(i).dN);
			hipFree(hdb.at(i).dLN);
		}
		hdb.clear();
	}
	if(hExtension.size()!=0){
		for (int i = 0; i < hExtension.size(); i++)
		{
			hipFree(hExtension.at(i).dExtension);
		}
		hExtension.clear();
	}

	if(hUniEdge.size()!=0){
		for (int i = 0; i < hUniEdge.size(); i++)
		{
			hipFree(hUniEdge.at(i).dUniEdge);
		}
		hUniEdge.clear();
	}

	if(hUniEdgeSatisfyMinsup.size()!=0){
		for (int i = 0; i < hUniEdgeSatisfyMinsup.size(); i++)
		{			
			hipFree(hUniEdgeSatisfyMinsup.at(i).dUniEdge);
			free(hUniEdgeSatisfyMinsup.at(i).hArrSup);					
		}
		hUniEdgeSatisfyMinsup.clear();
	}
	hipDeviceReset();
}

void PMS::prepareDataBase(){
		//unsigned int minsup = 34;
	unsigned int minsup = 2;
	unsigned int maxpat = 2;
	//unsigned int maxpat = 0x00000000;
	unsigned int minnodes = 0;
	bool where = true;
	bool enc = false;
	bool directed = false;

	//int opt;
	char* fname;
	//fname = "Klesscus";
	//fname = "Klessorigin";
	//fname = "KlessoriginCust1";
	fname= "G0G1G2_custom";
	

	////PMS pms;	
	ofstream fout("result.txt");

	//Chuyển dữ liệu từ fname sang TRANS
	//pms.run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);
	run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);
	//maxOfVer=pms.findMaxVertices();
	maxOfVer=findMaxVertices();	
	numberOfGraph=noGraphs();
	int sizeOfarrayO=maxOfVer*numberOfGraph;
	int* arrayO = new int[sizeOfarrayO]; //Tạo mảng arrayO có kích thước D*m
	if(arrayO==NULL){
		printf("\n!!!Memory Problem ArrayO");
		exit(1);
	}else{
		memset(arrayO, -1, sizeOfarrayO*sizeof(int)); // gán giá trị cho các phần tử mảng bằng -1
	}
	unsigned int noDeg; //Tổng bậc của tất cả các đỉnh trong csdl đồ thị TRANS
	//noDeg = pms.sumOfDeg();
	noDeg = sumOfDeg();
	//cout<<noDeg;
	unsigned int sizeOfArrayN=noDeg;
	int* arrayN = new int[sizeOfArrayN]; //Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	if(arrayN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayN");
		exit(1);
	}else
	{
		memset(arrayN, -1, noDeg*sizeof(int));
	}

//
	int* arrayLO = new int[sizeOfarrayO]; //Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	if(arrayLO==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLO");
		exit(1);
	}else
	{
		memset(arrayLO, -1, sizeOfarrayO*sizeof(int));
	}

	

	int* arrayLN = new int[noDeg]; //Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	if(arrayLN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLN");
		exit(1);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}


	//pms.importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);
	importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);
	cout<<"ArrayO:";
	displayArray(arrayO,sizeOfarrayO);
	cout<<"\nArrayLO:";
	displayArray(arrayLO,sizeOfarrayO);
	cout<<"\nArrayN:";
	displayArray(arrayN,noDeg);
	cout<<"\nArrayLN:";
	displayArray(arrayLN,noDeg);
//kích thước của dữ liệu
	size_t nBytesO = sizeOfarrayO*sizeof(int);
	size_t nBytesN = noDeg*sizeof(int);

	DB graphdb;
	graphdb.noElemdO = sizeOfarrayO;
	graphdb.noElemdN = noDeg;

	CHECK(hipMalloc((void**)&graphdb.dO,nBytesO));
	CHECK(hipMalloc((void**)&graphdb.dLO,nBytesO));
	CHECK(hipMalloc((void**)&graphdb.dN,nBytesN));
	CHECK(hipMalloc((void**)&graphdb.dLN,nBytesN));

	CHECK(hipMemcpy(graphdb.dO,arrayO,nBytesO,hipMemcpyHostToDevice));
	CHECK(hipMemcpy(graphdb.dLO,arrayLO,nBytesO,hipMemcpyHostToDevice));
	CHECK(hipMemcpy(graphdb.dN,arrayN,nBytesN,hipMemcpyHostToDevice));
	CHECK(hipMemcpy(graphdb.dLN,arrayLN,nBytesN,hipMemcpyHostToDevice));

	//pms.db.push_back(graphdb); //Đưa cơ sở dữ liệu vào vector db
	//pms.countNumberOfDifferentValue(pms.db.at(0).dLO,pms.db.at(0).noElemdO,pms.Lv);
	//pms.countNumberOfDifferentValue(pms.db.at(0).dLN,pms.db.at(0).noElemdN,pms.Le);
	hdb.push_back(graphdb); //Đưa cơ sở dữ liệu vào vector db
	countNumberOfDifferentValue(hdb.at(0).dLO,hdb.at(0).noElemdO,Lv);
	countNumberOfDifferentValue(hdb.at(0).dLN,hdb.at(0).noElemdN,Le);
	//pms.printdb();
}

bool PMS::checkArray(int *hostRef, int *gpuRef, const int N) {
	bool result=true;
	double epsilon = 1.0E-8;
	int match = 1;
	for (int i = 0; i < N; i++) {
		if ((float)(abs(hostRef[i] - gpuRef[i])) > epsilon) {
			match = 0;
			result=false;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n",
				hostRef[i], gpuRef[i], i);
			break;
		}
	}
	if (match){
		printf("Arrays match.\n\n");		
	}
	
	return result;
}


void PMS::displayArray(int *p, const unsigned int pSize=0)
{
	for(int i=0;i<pSize;i++){
		printf("P[%d]:%d ",i,p[i]);
	}
	printf("\n");
	return;
}

__global__ void kernelPrintdArr(int *dArr,unsigned int noElem){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("A[%d]:%d ",i,dArr[i]);
	}
}


void PMS::printdb(){
	printf("\n *********** Lv, Le **********\n");
	printf("\n Lv:%d",Lv);
	printf("\n Le:%d",Le);
	for (int i = 0; i < hdb.size(); i++)
	{
		unsigned int noElem =  hdb.at(i).noElemdO;	
		

		dim3 block(blocksize);
		dim3 grid((noElem + block.x -1)/block.x);
		printf("\n ********* dO *********\n");
		kernelPrintdArr<<<grid,block>>>(hdb.at(i).dO,noElem);
		hipDeviceSynchronize();
		printf("\n");

		printf("\n ********* dLO *********\n");
		kernelPrintdArr<<<grid,block>>>(hdb.at(i).dLO,noElem);
		hipDeviceSynchronize();
		printf("\n");

		unsigned int noElemdN = hdb.at(i).noElemdN;
		dim3 blocka(blocksize);
		dim3 grida((noElemdN + blocka.x -1)/blocka.x);

		printf("\n ********* dN *********\n");
		kernelPrintdArr<<<grida,blocka>>>(hdb.at(i).dN,noElemdN);
		hipDeviceSynchronize();
		printf("\n");

		printf("\n ********* dLN *********\n");
		kernelPrintdArr<<<grida,blocka>>>(hdb.at(i).dLN,noElemdN);
		hipDeviceSynchronize();
		printf("\n");
	}
}


__global__ void kernelMyScanV(int *dArrInput,int noElem,int *dResult){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		if(i==0){
			dResult[i]=0;
		}else
		{
			int temp=0;
			for (int j = 0; j <= (i-1); j++)
			{
				temp=temp + dArrInput[j];
			}
			dResult[i]=temp;
		}
	}
}


void  myScanV(int *dArrInput,int noElem,int *&dResult){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem + block.x -1)/block.x);

	CHECK(hipMalloc((void**)&dResult,noElem * sizeof(int)));

	kernelMyScanV<<<grid,block>>>(dArrInput,noElem,dResult);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	return;
}


__global__ void kernelCountNumberOfLabelVertex(int *d_LO,int *d_Lv,unsigned int sizeOfArrayLO){
	int i= blockDim.x*blockIdx.x + threadIdx.x;
	if(i<sizeOfArrayLO){
		if(d_LO[i]!=-1){
			d_Lv[d_LO[i]]=1;
		}
	}
}

void sumUntilReachZero(int *h_Lv,unsigned int n,int &result){
	for(int i=0;i<n && h_Lv[i]!=0;++i){
		++result;
	}
}

int  PMS::countNumberOfDifferentValue(int* d_LO,unsigned int sizeOfArrayLO, unsigned int &numberOfSaperateVertex){
	int status=0;
	hipError_t cudaStatus;
	numberOfSaperateVertex=0;
	size_t nBytesLv = sizeOfArrayLO*sizeof(int);
	//cấp phát mảng d_Lv trên device
	int *d_Lv;
	cudaStatus=hipMalloc((int**)&d_Lv,nBytesLv);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"hipMalloc d_Lv failed");
			goto Error;
		}
		else
		{
			hipMemset(d_Lv,0,nBytesLv);
		}

		//Cấp phát threads
		dim3 block(blocksize);
		dim3 grid((sizeOfArrayLO+block.x-1)/block.x);
		kernelCountNumberOfLabelVertex<<<grid,block>>>(d_LO,d_Lv,sizeOfArrayLO);
		
		hipDeviceSynchronize();
		printf("\nElements of d_Lv:");
		kernelPrintdArr<<<grid,block>>>(d_Lv,sizeOfArrayLO);

		int* h_Lv=NULL;
		h_Lv=(int*)malloc(nBytesLv);
		if(h_Lv==NULL){
			printf("h_Lv malloc memory fail");
			exit(1);
		}
		hipMemcpy(h_Lv,d_Lv,nBytesLv,hipMemcpyDeviceToHost);
		cudaStatus=hipDeviceSynchronize();
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"hipDeviceSynchronize fail",cudaStatus);
			status = -1;
			goto Error;
		}
		int result=0;
		sumUntilReachZero(h_Lv,sizeOfArrayLO,result);
		numberOfSaperateVertex=result;	

Error:
	hipFree(d_Lv);	
	return status;
}

__global__ void kernelGetAndStoreExtension(int *d_O,int *d_LO,unsigned int numberOfElementd_O,int *d_N,int *d_LN,unsigned int numberOfElementd_N,Extension *d_Extension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<numberOfElementd_O){
		if (d_O[i]!=-1){
			int j;
			int ek;
			//printf("\nThread:%d",i);	
			for(j=i+1;j<numberOfElementd_O;++j){					
				if(d_O[j]!=-1) {break;}				
			}			
			
			if (j==numberOfElementd_O) {
				ek=numberOfElementd_N;
			}
			else
			{
				ek=d_O[j];
			}
			//printf("\n[%d]:%d",i,ek);
			for(int k=d_O[i];k<ek;k++){
				//do something
				int index= k;
				d_Extension[index].vi=0;
				d_Extension[index].vj=0;
				d_Extension[index].li=d_LO[i];
				d_Extension[index].lij=d_LN[k];
				d_Extension[index].lj=d_LO[d_N[k]];
				d_Extension[index].vgi=i;
				d_Extension[index].vgj=d_N[k];
				//printf("\n[%d]:%d",i,index);
				/*printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",k,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,
					d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);*/
			}
		}
	}
}


int PMS::getAndStoreExtension(Extension *&d_Extension){
	int status =0;
	hipError_t cudaStatus;
	dim3 block(blocksize);
	unsigned int numberOfElementd_O = hdb.at(0).noElemdO;
	dim3 grid((numberOfElementd_O+block.x-1)/block.x);

	kernelGetAndStoreExtension<<<grid,block>>>(hdb.at(0).dO,hdb.at(0).dLO,numberOfElementd_O,hdb.at(0).dN,hdb.at(0).dLN,hdb.at(0).noElemdN,d_Extension);

	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelGetAndStoreExtension failed",cudaStatus);
		status =-1;
		goto Error;
	}


Error:
	return status;
}


int PMS::extractAllEdgeInDB(){
	int status = 0;
	arrExtension arrE;
	//cấp phát bộ nhớ cho d_Extension
	arrE.noElem =hdb.at(0).noElemdN;
	size_t nBytesOfArrayExtension = arrE.noElem*sizeof(Extension);
	
	CHECK(hipMalloc((Extension**)&arrE.dExtension,nBytesOfArrayExtension));
	//Trích tất cả các cạnh từ database rồi lưu vào d_Extension
	
	status  = getAndStoreExtension(arrE.dExtension);
	if(status ==-1){
		printf("\n getAndStoreExtension(arrE.dExtension) in extractAllEdgeInDB() failed");
		goto Error;
	}

	hExtension.push_back(arrE);
Error:
	return status;
}

__global__ void kernelPrintExtention(Extension *d_Extension,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){		
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);
	}
}

void PMS::displayArrExtension(Extension *dExtension,int noElem){

	//dim3 block(blocksize);
	//dim3 grid((noElem + block.x - 1)/block.x);

	//kernelPrintExtention<<<grid,block>>>(dExtension,noElem);
	//hipDeviceSynchronize();
	Extension *hExtension = (Extension*)malloc(sizeof(Extension)*noElem);
	if(hExtension==NULL){
		printf("\n Malloc hExtension in displayArrExtension() failed");
		exit(1);
	}
	CHECK(hipMemcpy(hExtension,dExtension,sizeof(Extension)*noElem,hipMemcpyDeviceToHost));
	for (int i = 0; i < noElem; i++)
	{
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,hExtension[i].vi,hExtension[i].vj,hExtension[i].li,hExtension[i].lij,hExtension[i].lj,hExtension[i].vgi,hExtension[i].vgj);
	}
	
	return;
}

void PMS::displayArrUniEdge(UniEdge* dUniEdge,int noElem){
	UniEdge *hUniEdge = (UniEdge*)malloc(sizeof(UniEdge)*noElem);
	if(hUniEdge==NULL){
		printf("\n malloc hUniEdge in displayArrUniEdge() failed");
		exit(1);
	}
	CHECK(hipMemcpy(hUniEdge,dUniEdge,sizeof(UniEdge)*noElem,hipMemcpyDeviceToHost));
	for (int i = 0; i < noElem; i++)
	{
		printf("\n U[%d]: (li lij lj) = (%d %d %d)",i,hUniEdge[i].li,hUniEdge[i].lij,hUniEdge[i].lj);
	}
	free(hUniEdge);
	return;
}


__global__ void	kernelValidEdge(Extension *d_Extension,int *dV,unsigned int numberElementd_Extension){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){	
		if(d_Extension[i].li<=d_Extension[i].lj){
			dV[i]=1;
		}
	}
}


hipError_t validEdge(Extension *d_Extension,int *&dV,unsigned int numberElementd_Extension){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid(numberElementd_Extension+block.x-1/block.x);

	kernelValidEdge<<<grid,block>>>(d_Extension,dV,numberElementd_Extension);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelValidEdge failed",cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

__global__ void kernelPrintdArr(float *dArr,int noElem){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("A[%d]:%d   ",i,dArr[i]);
	}
}
__global__ void kernelPrintdArr(int *dArr,int noElem){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("dArr[%d]:%d ",i,dArr[i]);
	}
}
int PMS::displayDeviceArr(int *dArr,int noElem){
	int status =0;
	//dim3 block(blocksize);
	//dim3 grid((noElem + block.x -1)/block.x);
	//kernelPrintdArr<<<grid,block>>>(dArr,noElem);
	//hipDeviceSynchronize();
	//hipError_t cudaStatus = hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n kernelDisplayDeviceArr() in displayDeviceArr() failed",cudaStatus);
	//	status = -1;
	//	goto Error;
	//}

	int *temp = (int*)malloc(sizeof(int)*noElem);
	if(temp==NULL){
		printf("\n Malloc temp in displayDeviceArr() failed");
		status=-1;
		goto Error;	
	}

	CHECK(hipMemcpy(temp,dArr,noElem*sizeof(int),hipMemcpyDeviceToHost));

	for (int i = 0; i < noElem; i++)
	{
		printf(" A[%d]:%d  ",i,temp[i]);
	}

	free(temp);
Error:

	return 0;
}

int PMS::displayDeviceArr(float *dArr,int noElem){
	int status =0;
	//dim3 block(blocksize);
	//dim3 grid((noElem + block.x -1)/block.x);
	//kernelPrintdArr<<<grid,block>>>(dArr,noElem);
	//hipDeviceSynchronize();
	//hipError_t cudaStatus = hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n kernelDisplayDeviceArr() in displayDeviceArr() failed",cudaStatus);
	//	status = -1;
	//	goto Error;
	//}
	float *temp = (float*)malloc(sizeof(float)*noElem);
	if(temp==NULL){
		printf("\n Malloc temp in displayDeviceArr() failed");
		status=-1;
		goto Error;	
	}

	CHECK(hipMemcpy(temp,dArr,noElem*sizeof(float),hipMemcpyDeviceToHost));

	for (int i = 0; i < noElem; i++)
	{
		int a = (int)temp[i];
		printf(" A[%d]:%d  ",i,a);
	}

	free(temp);
Error:
	return status;
}



__global__ void kernelGetSize(int *dV,int *dVScanResult,int noElem,int *size){
	*size = dVScanResult[noElem-1];
	if(dV[noElem-1]==1){
		*size = *size + 1;
	}
}


hipError_t getSizeBaseOnScanResult(int *dV,int *dVScanResult,int noElem,int &output){
	hipError_t cudaStatus;
	int temp=0;
	int *size;
	CHECK(hipMalloc((void**)&size,sizeof(int)));
	CHECK(hipMemset(size,0,sizeof(int)));
	kernelGetSize<<<1,1>>>(dV,dVScanResult,noElem,size);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\n kernelGetSize() in getSizeBaseOnResult() failed",cudaStatus);
		goto Error;
	}

	CHECK(hipMemcpy(&temp,size,sizeof(int),hipMemcpyDeviceToHost));
	output = (int)temp;

	hipFree(size);
Error:
	
	return cudaStatus;
}

__global__ void kernelExtractValidExtension(Extension *d_Extension,int *dV,int *dVScanResult,int numberElementd_Extension,Extension *d_ValidExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){
		if(dV[i]==1){
			int index = dVScanResult[i];
			//printf("\nV[%d]:%d, index[%d]:%d,d_Extension[%d], d_Extension[%d]:%d\n",i,V[i],i,index[i],i,i,d_Extension[i].vgi);
			d_ValidExtension[index].li=d_Extension[i].li;
			d_ValidExtension[index].lj=d_Extension[i].lj;
			d_ValidExtension[index].lij=d_Extension[i].lij;
			d_ValidExtension[index].vgi=d_Extension[i].vgi;
			d_ValidExtension[index].vgj=d_Extension[i].vgj;
			d_ValidExtension[index].vi=d_Extension[i].vi;
			d_ValidExtension[index].vj=d_Extension[i].vj;
		}
	}
}

hipError_t extractValidExtension(Extension *d_Extension,int *dV,int *dVScanResult, int numberElementd_Extension,Extension *&d_ValidExtension){
	hipError_t cudaStatus;
	
	//printfExtension(d_Extension,numberElementd_Extension);

	dim3 block(blocksize);
	dim3 grid((numberElementd_Extension+block.x)/block.x);

	kernelExtractValidExtension<<<grid,block>>>(d_Extension,dV,dVScanResult,numberElementd_Extension,d_ValidExtension);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelGetValidExtension failed");
		goto Error;
	}

Error:
	return cudaStatus;
}

int PMS::getValidExtension(){
	int status = 0;


	//Phase 1: đánh dấu vị trí những cạnh hợp lệ (li<=lj)

	int numberElementd_Extension = hExtension.at(0).noElem;
	int *dV;
	size_t nBytesdV= numberElementd_Extension*sizeof(int);

	hipError_t cudaStatus=hipMalloc((void**)&dV,nBytesdV);
	if (cudaStatus!= hipSuccess){
		fprintf(stderr,"hipMalloc array V failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dV,0,nBytesdV));
	}
	
	cudaStatus=validEdge(hExtension.at(0).dExtension,dV,hExtension.at(0).noElem);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize validEdge failed",cudaStatus);
		status = -1;
		goto Error;
	}
	//
	int *hV = (int*)malloc(sizeof(int)*numberElementd_Extension);
	hipMemcpy(hV,dV,sizeof(int)*numberElementd_Extension,hipMemcpyDeviceToHost);
	printf("\n ************ dV **************\n");
	for (int i = 0; i < numberElementd_Extension; i++)
	{
		int temp = hV[i];
		printf("[%d]:%d ",i,temp);
	}

	int* dVScanResult;
	cudaStatus=hipMalloc((void**)&dVScanResult,numberElementd_Extension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"Cuda Malloc failed",cudaStatus);
		goto Error;
	}	
	else
	{
		hipMemset(dVScanResult,0,numberElementd_Extension*sizeof(int));
	}
	//Exclusive scan mảng V và lưu kết quả scan vào mảng index
	 scanV(dV,numberElementd_Extension,dVScanResult);
	
		
	printf("\n ************ dVScanResult **************\n");
	hipMemcpy(hV,dVScanResult,sizeof(int)*numberElementd_Extension,hipMemcpyDeviceToHost);
	for (int i = 0; i < numberElementd_Extension; i++)
	{
		int temp = hV[i]; 
		printf("[%d]:%d ",i,temp);
	}

	//Phase 2: trích những cạnh hợp lệ sang một mảng khác dValidExtension
		//arrExtension arrValidExtension;
		hValidExtension.resize(1);
		getSizeBaseOnScanResult(dV,dVScanResult,numberElementd_Extension,hValidExtension.at(0).noElem);
		printf("\n arrValidExtension.noElem:%d",hValidExtension.at(0).noElem);
		CHECK(hipMalloc((void**)&(hValidExtension.at(0).dExtension),sizeof(Extension)*hValidExtension.at(0).noElem));
		CHECK(extractValidExtension(hExtension.at(0).dExtension,dV,dVScanResult,numberElementd_Extension,hValidExtension.at(0).dExtension));
		//displayArrExtension(arrValidExtension.dExtension,arrValidExtension.noElem);

		//hValidExtension.push_back(arrValidExtension);
		printf("\n************hValidExtension***********\n");
		displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

		hipFree(dV);
		hipFree(dVScanResult);
		free(hV);
Error:	
	return status;
}

__global__ void kernelMarkLabelEdge(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *d_allPossibleExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		int index=	d_ValidExtension[i].li*Lv*Le + d_ValidExtension[i].lij*Lv + d_ValidExtension[i].lj;
		d_allPossibleExtension[index]=1;
	}

}

hipError_t markLabelEdge(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *&d_allPossibleExtension){
	hipError_t cudaStatus;
	
	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	kernelMarkLabelEdge<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,Lv,Le,d_allPossibleExtension);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"kernelMarkLabelEdge in markLabelEdge() failed");
		goto Error;
	}

Error:
	return cudaStatus;
}

__global__ void kernelCalcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,UniEdge *d_UniqueExtension,unsigned int Le,unsigned int Lv){
	int i=blockIdx.x*blockDim.x + threadIdx.x;	
	if(i<noElem_allPossibleExtension && d_allPossibleExtension[i]==1){
		int li,lj,lij;
		li=i/(Le*Lv);
		lij=(i%(Le*Lv))/Lv;
		lj=(i%(Le*Lv))-((i%(Le*Lv))/Lv)*Lv;
		int index = d_allPossibleExtensionScanResult[i];
		//printf("\n[%d]:%d li:%d lij:%d lj:%d",i,d_allPossibleExtensionScanResult[i],li,lij,lj);
		d_UniqueExtension[index].li=li;
		d_UniqueExtension[index].lij=lij;
		d_UniqueExtension[index].lj=lj;
	}
}

hipError_t calcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,UniEdge *&d_UniqueExtension,unsigned int noElem_d_UniqueExtension,unsigned int Le,unsigned int Lv){
	hipError_t cudaStatus;
	
	dim3 block(blocksize);
	dim3 grid((noElem_allPossibleExtension+block.x-1)/block.x);
	kernelCalcLabelAndStoreUniqueExtension<<<grid,block>>>(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,Le,Lv);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"kernelCalcLabelAndStoreUniqueExtension in CalcLabelAndStoreUniqueExtension() failed");
		goto Error;
	}

Error:
	return cudaStatus;
}

int PMS::extractUniEdge(){
	int status=0;


	//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
	unsigned int noElem_dallPossibleExtension=Le*Lv*Lv;
	int *d_allPossibleExtension;

	//cấp phát bộ nhớ cho mảng d_allPossibleExtension
	hipError_t	cudaStatus=hipMalloc((void**)&d_allPossibleExtension,noElem_dallPossibleExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtension failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(d_allPossibleExtension,0,noElem_dallPossibleExtension*sizeof(int)));
	}

	cudaStatus=markLabelEdge(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,Lv,Le,d_allPossibleExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"getUniqueExtension failed",cudaStatus);
		status=-1;
		goto Error;
	}


	int *d_allPossibleExtensionScanResult;
	cudaStatus=hipMalloc((void**)&d_allPossibleExtensionScanResult,noElem_dallPossibleExtension*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtensionScanResult failed");
		status = -1;
		goto Error;
	}
	// printf("\n **************** hValidExtension ****************\n");
	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

	//Exclusive scan mảng d_allPossibleExtension và lưu kết quả vào mảng d_allPossibleExtensionScanResult
	cudaStatus = scanV(d_allPossibleExtension,noElem_dallPossibleExtension,d_allPossibleExtensionScanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n ScanV() in computeSupport() failed");
		status = -1;
		goto Error;
	}
	//	 printf("\n **************** hValidExtension ****************\n");
	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);


	//printf("\n**********d_allPossibleExtension************\n");
	//displayDeviceArr(d_allPossibleExtension,noElem_dallPossibleExtension);


	arrUniEdge strUniEdge;
	int noElem_d_UniqueExtension=0;
	//Tính kích thước của mảng d_UniqueExtension dựa vào kết quả exclusive scan
	cudaStatus=getSizeBaseOnScanResult(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_dallPossibleExtension,noElem_d_UniqueExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getLastElement() in extractUniEdge() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	 //printf("\n\nnoElem_d_UniqueExtension:%d",noElem_d_UniqueExtension);
	 strUniEdge.noElem = noElem_d_UniqueExtension;

	

	//Tạo mảng d_UniqueExtension với kích thước mảng vừa tính được
	cudaStatus=hipMalloc((void**)&strUniEdge.dUniEdge,noElem_d_UniqueExtension*sizeof(UniEdge));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_UniqueExtension in extractUniEdge() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(strUniEdge.dUniEdge,0,noElem_d_UniqueExtension*sizeof(UniEdge)));
	}

	


	//Ánh xạ ngược lại từ vị trí trong d_allPossibleExtension thành cạnh và lưu kết quả vào d_UniqueExtension
	cudaStatus=calcLabelAndStoreUniqueExtension(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_dallPossibleExtension,strUniEdge.dUniEdge,noElem_d_UniqueExtension,Le,Lv);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n\ncalcLabelAndStoreUniqueExtension() in extractUniEdge() failed",cudaStatus);
		status = -1;
		goto Error;
	}



	hUniEdge.push_back(strUniEdge);
	printf("\n **************** hUniEdge ****************\n");
	displayArrUniEdge(hUniEdge.at(0).dUniEdge,hUniEdge.at(0).noElem);


	hipFree(d_allPossibleExtension);
	hipFree(d_allPossibleExtensionScanResult);
Error:

	return status;
}

__global__ void kernelCalcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *dB,unsigned int maxOfVer){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension-1){
		unsigned int graphIdAfter=d_ValidExtension[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=d_ValidExtension[i].vgi/maxOfVer;
		unsigned int resultDiff=graphIdAfter-graphIdCurrent;
		dB[i]=resultDiff;
	}
}

hipError_t calcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *&dB,unsigned int maxOfVer){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x)/block.x);

	kernelCalcBoundary<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,dB,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\kernelCalcBoundary in calcBoundary() failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
__global__ void kernelGetLastElement(int *dScanResult,unsigned int noElem,int *output){
	output[0]=dScanResult[noElem-1];
}


hipError_t getLastElement(int *dScanResult,unsigned int noElem,int &output){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem+block.x-1)/block.x);

	int *value;
	hipMalloc((int**)&value,sizeof(int));
	
	kernelGetLastElement<<<1,1>>>(dScanResult,noElem,value);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	CHECK(hipMemcpy(&output,value,sizeof(int),hipMemcpyDeviceToHost));
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	
	hipFree(value);
Error:
	return cudaStatus;	
}

__global__ void kernelSetValuedF(UniEdge *dUniEdge,int noElemdUniEdge,Extension *dValidExtension,int noElemdValidExtension,int *dBScanResult,int *dF,int noElemF){
	int i = blockDim.x * blockIdx.x +threadIdx.x;
	if(i<noElemdValidExtension){
		for (int j = 0; j < noElemdUniEdge; j++)
		{
			if(dUniEdge[j].li==dValidExtension[i].li && dUniEdge[j].lij==dValidExtension[i].lij &&	dUniEdge[j].lj==dValidExtension[i].lj){
				dF[dBScanResult[i]+j*noElemF]=1;
			}
		}
	}
}

__global__ void kernelCopyFromdFtoTempF(int *d_F,int *tempF,int from,int noElemNeedToCopy){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemNeedToCopy){
		int index = from*noElemNeedToCopy + i;
		tempF[i]=d_F[index];
	}
}

hipError_t calcSupport(UniEdge *dUniEdge,int noElemdUniEdge,Extension *dValidExtension,int noElemdValidExtension,int *dBScanResult,int *dF,int noElemF,int *&hResultSup){
	hipError_t cudaStatus;

	//Đánh dấu những đồ thị chứa embedding trong mảng d_F
	dim3 block(blocksize);
	dim3 grid((noElemdValidExtension+block.x - 1)/block.x);
	kernelSetValuedF<<<grid,block>>>(dUniEdge,noElemdUniEdge,dValidExtension,noElemdValidExtension,dBScanResult,dF,noElemF);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() of kernelComputeSupport in computeSupport failed",cudaStatus);
		goto Error;
	}

	

	//Duyệt qua mảng d_UniqueExtension, tính reduction cho mỗi segment i*noElemF, kết quả của reduction là độ support của cạnh i trong d_UniqueExtension
	int *tempF;
	cudaStatus = hipMalloc((void**)&tempF,noElemF*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n CudaMalloc tempF in calcSupport() failed",cudaStatus);
		goto Error;
	}
	else
	{
		CHECK(hipMemset(tempF,0,noElemF*sizeof(int)));
	}

	//float *resultSup; /* Lưu kết quả reduction */
	hResultSup = (int*)malloc(noElemdUniEdge*sizeof(int));
	if (hResultSup==NULL){
		printf("\n Malloc hResultSup in calcSupport() failed");
		exit(1);
	}

	dim3 blocka(blocksize);
	dim3 grida((noElemF+blocka.x-1)/blocka.x);
	/*int from =0;*/	
	for (int i = 0; i < noElemdUniEdge; i++)
	{		
		//chép dữ liệu d_F sang tempF ứng theo các phần tử lần lược là i*noElemF, copy đúng noElemF
		/*from =i;*/				
		kernelCopyFromdFtoTempF<<<grid,block>>>(dF,tempF,i,noElemF);
		hipDeviceSynchronize();
		reduction(tempF,noElemF,hResultSup[i]);		
	}
	////In độ hỗ trợ cho các cạnh tương ứng trong mảng kết quả resultSup
	//for (int i = 0; i < noElemdUniEdge; i++)
	//{
	//	printf("\n resultSup[%d]:%d",i,hResultSup[i]);
	//}

	hipFree(tempF);

Error:
	return cudaStatus;

}

__global__ void	kernelMarkUniEdgeSatisfyMinsup(int *dResultSup,int noElemUniEdge,int *dV,unsigned int minsup){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge){
		int temp = dResultSup[i];
		if(temp >= minsup){
			dV[i]=1;
		}
	}
}

__global__ void	kernelExtractUniEdgeSatifyMinsup(UniEdge *dUniEdge,int *dV,int *dVScanResult,int noElemUniEdge,UniEdge *dUniEdgeSatisfyMinsup,int *dSup,int *dResultSup){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemUniEdge){
		if(dV[i]==1){
			dUniEdgeSatisfyMinsup[dVScanResult[i]].li = dUniEdge[i].li;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lij = dUniEdge[i].lij;
			dUniEdgeSatisfyMinsup[dVScanResult[i]].lj=dUniEdge[i].lj;
			dSup[dVScanResult[i]]=dResultSup[i];
		}
	}
}



int PMS::extractUniEdgeSatisfyMinsup(int *hResultSup,int noElemUniEdge,unsigned int minsup){
	int status=0;
	hipError_t cudaStatus;
	//1. Cấp phát mảng trên device có kích thước bằng noElemUniEdge
	int *dResultSup=nullptr;
	CHECK(hipMalloc((void**)&dResultSup,noElemUniEdge*sizeof(int)));
	CHECK(hipMemcpy(dResultSup,hResultSup,noElemUniEdge*sizeof(int),hipMemcpyHostToDevice));

	printf("\n *******dResultSup********\n");
	displayDeviceArr(dResultSup,noElemUniEdge);

	//2. Đánh dấu 1 trên dV cho những phần tử thoả minsup
	int *dV=nullptr;
	CHECK(hipMalloc((void**)&dV,noElemUniEdge*sizeof(int)));
	CHECK(hipMemset(dV,0,sizeof(int)*noElemUniEdge));

	dim3 block(blocksize);
	dim3 grid((noElemUniEdge + block.x - 1)/block.x);
	kernelMarkUniEdgeSatisfyMinsup<<<grid,block>>>(dResultSup,noElemUniEdge,dV,minsup);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelMarkUniEdgeSatisfyMinsup in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	
	printf("\n ***********dV**********\n");
	displayDeviceArr(dV,noElemUniEdge);

	int *dVScanResult=nullptr;
	CHECK(hipMalloc((void**)&dVScanResult,noElemUniEdge*sizeof(int)));
	CHECK(scanV(dV,noElemUniEdge,dVScanResult));
	printf("\n ***********dVScanResult**********\n");
	displayDeviceArr(dVScanResult,noElemUniEdge);

	hUniEdgeSatisfyMinsup.resize(1);
	CHECK(getSizeBaseOnScanResult(dV,dVScanResult,noElemUniEdge,hUniEdgeSatisfyMinsup.at(0).noElem));
	CHECK(hipMalloc((void**)&hUniEdgeSatisfyMinsup.at(0).dUniEdge,hUniEdgeSatisfyMinsup.at(0).noElem*sizeof(UniEdge)));
	hUniEdgeSatisfyMinsup.at(0).hArrSup = (int*)malloc(sizeof(int)*hUniEdgeSatisfyMinsup.at(0).noElem);
	if (hUniEdgeSatisfyMinsup.at(0).hArrSup ==NULL){
		printf("\n malloc hArrSup of hUniEdgeSatisfyMinsup failed()");
		exit(1);
	}


	int *dSup=nullptr;
	CHECK(hipMalloc((void**)&dSup,hUniEdgeSatisfyMinsup.at(0).noElem*sizeof(int)));
	

	dim3 blocka(blocksize);
	dim3 grida((noElemUniEdge + blocka.x -1)/blocka.x);
	kernelExtractUniEdgeSatifyMinsup<<<grida,blocka>>>(hUniEdge.at(0).dUniEdge,dV,dVScanResult,noElemUniEdge,hUniEdgeSatisfyMinsup.at(0).dUniEdge,dSup,dResultSup);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelExtractUniEdgeSatisfyMinsup() in extractUniEdgeSatisfyMinsup() failed",cudaStatus);
		goto Error;
	}
	printf("\n ********hUniEdgeSatisfyMinsup.dUniEdge****************\n");
	displayArrUniEdge(hUniEdgeSatisfyMinsup.at(0).dUniEdge,hUniEdgeSatisfyMinsup.at(0).noElem);
	printf("\n ********hUniEdgeSatisfyMinsup.dSup****************\n");
	displayDeviceArr(dSup,hUniEdgeSatisfyMinsup.at(0).noElem);

	CHECK(hipMemcpy(hUniEdgeSatisfyMinsup.at(0).hArrSup,dSup,sizeof(int)*hUniEdgeSatisfyMinsup.at(0).noElem,hipMemcpyDeviceToHost));

	for (int i = 0; i < hUniEdgeSatisfyMinsup.at(0).noElem; i++)
	{
		printf("\n hArrSup:%d ",hUniEdgeSatisfyMinsup.at(0).hArrSup[i]);
	}

	hipFree(dResultSup);
	hipFree(dV);
	hipFree(dVScanResult);
	hipFree(dSup);
Error:
	return status;
}


int PMS::computeSupport(){
	int status=0;
	/* Xây dựng Boundary cho mảng d_ValidExtension */
	//1. Cấp phát một mảng d_B và gán các giá trị 0 cho mọi phần tử của d_B
	unsigned int noElement_dB=hValidExtension.at(0).noElem;
	int* dB;
	hipError_t cudaStatus=hipMalloc((int**)&dB,noElement_dB*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc dB in computeSupport() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dB,0,noElement_dB*sizeof(int)));
	}
	//printf("\n**********dValidExtension*************\n");
	//displayArrExtension(hValidExtension.at(0).dExtension,noElement_dB);
	//printf("\n*********dB********\n");
	//displayDeviceArr(dB,noElement_dB);


	//Gián giá trị boundary cho d_B
	cudaStatus=calcBoundary(hValidExtension.at(0).dExtension,noElement_dB,dB,maxOfVer);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"calcBoundary() in computeSupport() failed",cudaStatus);
		return 1;
	}

	printf("\n**********dValidExtension*************\n");
	displayArrExtension(hValidExtension.at(0).dExtension,noElement_dB);
	printf("\n*********dB********\n");
	displayDeviceArr(dB,noElement_dB);


	//2. Exclusive Scan mảng d_B
	int* dBScanResult;
	cudaStatus=hipMalloc((int**)&dBScanResult,noElement_dB*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc dBScanResult in computeSupport() failed",cudaStatus);
		status = -1;
		goto Error;
	}
	else
	{
		hipMemset(dBScanResult,0,noElement_dB*sizeof(int));
	}

	cudaStatus=scanV(dB,noElement_dB,dBScanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanB function failed",cudaStatus);
		status =-1;
		goto Error;
	}

	printf("\n\n*******dBScanResult***********\n");
	displayDeviceArr(dBScanResult,noElement_dB);

	//3. Tính độ hỗ trợ cho các mở rộng trong d_UniqueExtension
	//3.1 Tạo mảng d_F có số lượng phần tử bằng với giá trị cuối cùng của mảng d_scanB_Result cộng 1 và gán giá trị 0 cho các phần tử.
	int noElemF=0;
	cudaStatus=getLastElement(dBScanResult,noElement_dB,noElemF);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ngetLastElement function failed",cudaStatus);
		return 1;
	}

	noElemF++;
	/*noElemGraphInExt=noElemF;*/

	printf("\n\n noElement_F:%d",noElemF);
	int noElem_d_UniqueExtension= hUniEdge.at(0).noElem;
	int *dF;
	cudaStatus=hipMalloc((int**)&dF,noElem_d_UniqueExtension*noElemF*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc dF in computeSupport() failed",cudaStatus);
		status =-1;
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dF,0,noElem_d_UniqueExtension*noElemF*sizeof(int)));
	}
	int *hResultSup=nullptr;
	cudaStatus=calcSupport(hUniEdge.at(0).dUniEdge,hUniEdge.at(0).noElem,hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,dBScanResult,dF,noElemF,hResultSup);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n calcSupport() in computeSupport() failed",cudaStatus);
		status =-1;
		goto Error;
	}

	////In độ hỗ trợ cho các cạnh tương ứng trong mảng kết quả h_resultSup
	//for (int i = 0; i < noElem_d_UniqueExtension; i++)
	//{
	//	printf("\n resultSup[%d]:%d",i,hResultSup[i]);
	//}
	//
	extractUniEdgeSatisfyMinsup(hResultSup,noElem_d_UniqueExtension,minsup);

	hipFree(dBScanResult);
	hipFree(dB);
Error:
	return status;
}

__global__ void kernelGetGraphIdContainEmbedding(int li,int lij,int lj,Extension *d_ValidExtension,int noElem_d_ValidExtension,int *dV,unsigned int maxOfVer){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension){
		if(	d_ValidExtension[i].li == li && d_ValidExtension[i].lij == lij && 	d_ValidExtension[i].lj == lj){
			int graphid = (d_ValidExtension[i].vgi/maxOfVer);
			dV[graphid]=1;
		}
	}
}

__global__ void kernelGetLastElementExtension(Extension *inputArray,unsigned int noEleInputArray,int *value,unsigned int maxOfVer){
	value[0] = inputArray[noEleInputArray-1].vgi/maxOfVer; /*Lấy global vertex id chia cho tổng số đỉnh của đồ thị (maxOfVer). Ở đây các đồ thị luôn có số lượng đỉnh bằng nhau (maxOfVer) */
}

hipError_t getLastElementExtension(Extension* inputArray,unsigned int numberElementOfInputArray,int &outputValue,unsigned int maxOfVer){
	hipError_t cudaStatus;

	int *temp=nullptr;
	CHECK(hipMalloc((int**)&temp,sizeof(int)));
	//kernelPrintExtention<<<1,512>>>(inputArray,numberElementOfInputArray);
	//hipDeviceSynchronize();
	//cudaStatus= hipGetLastError();
	//if(cudaStatus != hipSuccess){
	//	fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
	//	goto Error;
	//}

	/* Lấy graphId chứa embedding cuối cùng */
	kernelGetLastElementExtension<<<1,1>>>(inputArray,numberElementOfInputArray,temp,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	CHECK(hipMemcpy(&outputValue,temp,sizeof(int),hipMemcpyDeviceToHost));
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	
	hipFree(temp);
Error:	
	return cudaStatus;	
}

__global__ void kernelGetGraph(int *dV,int noElemdV,int *d_kq,int *dVScanResult){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdV){
		if(dV[i]!=0){
			d_kq[dVScanResult[i]]=i;
		}
	}
}

int PMS::getGraphIdContainEmbedding(UniEdge edge,int *&hArrGraphId,int &noElemhArrGraphId){
	int status =0;
	int noElemdValidExtension = hExtension.at(0).noElem;

	int li,lij,lj;
	li = edge.li;
	lij = edge.lij;
	lj = edge.lj;
	dim3 block(blocksize);
	dim3 grid((noElemdValidExtension+block.x-1)/block.x);

	int *dV=nullptr;
	int noElemdV=0;

	//displayArrExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);

	CHECK(getLastElementExtension(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,noElemdV,maxOfVer));
	noElemdV++;
	
	hipError_t cudaStatus=hipMalloc((void**)&dV,noElemdV*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dV in getGraphIdContainEmbedding() failed");
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dV,0,noElemdV*sizeof(int)));
	}

	kernelGetGraphIdContainEmbedding<<<grid,block>>>(li,lij,lj,hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,dV,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		status =-1;
		fprintf(stderr,"\n kernelGetGraphIdContainEmbedding() in getGraphIdContainEmbedding() failed",cudaStatus);
		goto Error;
	}

	int *dVScanResult=nullptr;
	cudaStatus=hipMalloc((void**)&dVScanResult,noElemdV*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dVScanResult in getGraphIdContainEmbedding() failed");
		goto Error;
	}
	else
	{
		CHECK(hipMemset(dVScanResult,0,noElemdV*sizeof(int)));
	}


	scanV(dV,noElemdV,dVScanResult);

	printf("\n ************* dVScanResult *************\n");
	displayDeviceArr(dVScanResult,noElemdV);
	int noElem_kq;	
	CHECK(getLastElement(dVScanResult,noElemdV,noElem_kq));
	noElem_kq++;

	int *d_kq;
	hipMalloc((void**)&d_kq,sizeof(int)*noElem_kq);
	
	dim3 blocka(blocksize);
	dim3 grida((noElemdV + blocka.x -1)/blocka.x);

	kernelGetGraph<<<grida,blocka>>>(dV,noElemdV,d_kq,dVScanResult);
	hipDeviceSynchronize();

	printf("\n*********** d_kq ***********\n");
	displayDeviceArr(d_kq,noElem_kq);

	hArrGraphId=(int*)malloc(sizeof(int)*noElem_kq);
	if(hArrGraphId==NULL){
		printf("\nMalloc hArrGraphId in getGraphIdContainEmbedding() failed");
		exit(1);
	}
	noElemhArrGraphId=noElem_kq;

	CHECK(hipMemcpy(hArrGraphId,d_kq,sizeof(int)*noElem_kq,hipMemcpyDeviceToHost));
	
	hipFree(d_kq);
	hipFree(dV);
	hipFree(dVScanResult);
Error:
	return status;
}


int PMS::Mining(){
	int status = 0;

	int noElemtemp = hUniEdgeSatisfyMinsup.at(0).noElem;
	UniEdge *temp=(UniEdge*)malloc(sizeof(UniEdge)*noElemtemp);
	if(temp==NULL){
		printf("\n malloc temp failed");
		status =-1;
		goto Error;
	}

	CHECK(hipMemcpy(temp,hUniEdgeSatisfyMinsup.at(0).dUniEdge,noElemtemp*sizeof(UniEdge),hipMemcpyDeviceToHost));
	
	for (int i = 0; i < noElemtemp; i++) //Duyệt qua các UniEdge thoả minSup
	{
			int li,lij,lj;
			li = temp[i].li;
			lij= temp[i].lij;
			lj=temp[i].lj;

			DFS_CODE.push(0,1,temp[i].li,temp[i].lij,temp[i].lj);//xây dựng DFS_CODE
			int minLabel = temp[i].li;
			int maxid = 1;
			
			if(is_min()){ //Nếu DFS_CODE là min thì tìm các graphid chứa embedding của DFS_CODE
				printf("\n This is minDFSCODE\n");

				int *hArrGraphId; //Mảng chứa các graphID có embedding của DFS_Code.
				int noElemhArrGraphId=0;
				/* Trước khi ghi kết quả thì phải biết đồ thị phổ biến đó tồn tại ở những graphId nào. Hàm getGraphIdContainEmbedding dùng để làm việc này
				* 3 tham số đầu tiên của hàm là nhãn cạnh của phần tử d_UniqueExtension đang xét */
				status =getGraphIdContainEmbedding(temp[i],hArrGraphId,noElemhArrGraphId);
				if (status!=0){
					printf("\n\n getGraphIdContainEmbedding() in Mining() failed");
					goto Error;
				}
				
				//In nội dung mảng hArrGraphId
				 
				printf("\n ************** hArrGraphId ****************\n");
				for (int j = 0; j < noElemhArrGraphId; j++)
				{
					printf("%d ",hArrGraphId[j]);
				}

				/*	Ghi kết quả DFS_CODE vào file result.txt ************************************************************
				 *	Hàm report sẽ chuyển DFS_CODE pattern sang dạng đồ thị, sau đó sẽ ghi đồ thị đó xuống file result.txt
				 *	Hàm report gồm 3 tham số:
				 *	Tham số thứ 1: mảng chứa danh sách các graphID chứa DFS_CODE pattern
				 *	Tham số thứ 2: số lượng mảng
				 *	Tham số thứ 3: độ hỗ trợ của DFS_CODE pattern *******************************************************/
				
				report(hArrGraphId,noElemhArrGraphId,hUniEdgeSatisfyMinsup.at(0).hArrSup[i]);

				//Xây dựng Embedding cho DFS_Code rồi gọi hàm GraphMining để khai thác
				//Trong GraphMining sẽ gọi GraphMining khác để thực hiện khai thác đệ quy

				FUNCHECK(buildFirstEmbedding(temp[i])); //Xây dựng 2 cột embedding ban đầu.

				//FSMining(DFS_CODE,EmbeddingColumn,hValidExtension(EXTk),hUniEdge,RMP,FwEXt,BwEXt)

				//Giải phóng bộ nhớ 
				free(hArrGraphId);
				DFS_CODE.pop();
				if(hEmbedding.size()!=0){
					for (int j = 0; j < hEmbedding.size(); j++)
					{
						hipFree(hEmbedding.at(j).dArrEmbedding);
					}
					hEmbedding.clear();
				}
			}
	}
	
	free(temp);
Error:
	return status;
}


__global__ void kernelMarkExtension(const Extension *d_ValidExtension,int noElem_d_ValidExtension,int *dV,int li,int lij,int lj){
	int i= blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			dV[i]=1;
		}		
	}
}

__global__ void kernelSetValueForFirstTwoEmbeddingColumn(const Extension *d_ValidExtension,int noElem_d_ValidExtension,Embedding *dQ1,Embedding *dQ2,int *d_scanResult,int li,int lij,int lj){
	int i = blockDim.x *blockIdx.x +threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij == lij && d_ValidExtension[i].lj==lj){
			dQ1[d_scanResult[i]].idx=-1;
			dQ1[d_scanResult[i]].vid=d_ValidExtension[i].vgi;
			

			dQ2[d_scanResult[i]].idx=d_scanResult[i];
			dQ2[d_scanResult[i]].vid=d_ValidExtension[i].vgj;
		}
	}
}

__global__ void	kernelPrintEmbedding(Embedding *dArrEmbedding,int noElem){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem){
		printf("\n Thread:%d (idx vid):(%d %d)",i,dArrEmbedding[i].idx,dArrEmbedding[i].vid);
	}
}


int PMS::buildFirstEmbedding(UniEdge ue){
	int li,lij,lj;
	li=ue.li;
	lij=ue.lij;
	lj=ue.lj;
	int status =0;
	hipError_t cudaStatus;
	hEmbedding.resize(2);
	hEmbedding.at(0).noElem;

	int *dV=nullptr;
	int noElemdV = hValidExtension.at(0).noElem;
	CHECK(hipMalloc((void**)&dV, sizeof(int)*noElemdV));
	CHECK(hipMemset(dV,0,sizeof(int)*noElemdV));
	dim3 block(blocksize);
	dim3 grid((noElemdV+block.x-1)/block.x);

	//kernelPrintExtention<<<1,512>>>(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem);
	//hipDeviceSynchronize();
	//CHECK(hipGetLastError());
	//if(hipGetLastError() !=hipSuccess){
	//	printf("Error here");
	//	goto Error;
	//}

	kernelMarkExtension<<<grid,block>>>(hValidExtension.at(0).dExtension,noElemdV,dV,li,lij,lj);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus!=hipSuccess){
		status = -1;
		fprintf(stderr,"\n kernelMarkExtension failed",cudaStatus);
		goto Error;
	}

	int* dVScanResult;
	CHECK(hipMalloc((int**)&dVScanResult,noElemdV*sizeof(int)));
	CHECK(hipMemset(dVScanResult,0,noElemdV*sizeof(int)));

	CHECK(scanV(dV,noElemdV,dVScanResult));
	//myScanV(dV,noElemdV,dVScanResult);


	int noElemOfdArEmbedding=0;
	CHECK(getSizeBaseOnScanResult(dV,dVScanResult,noElemdV,noElemOfdArEmbedding));
	hEmbedding.at(0).noElem=hEmbedding.at(1).noElem=noElemOfdArEmbedding;

	CHECK(hipMalloc((void**)&hEmbedding.at(0).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));
	CHECK(hipMalloc((void**)&hEmbedding.at(1).dArrEmbedding,noElemOfdArEmbedding*sizeof(Embedding)));

	
	kernelSetValueForFirstTwoEmbeddingColumn<<<grid,block>>>(hValidExtension.at(0).dExtension,hValidExtension.at(0).noElem,hEmbedding.at(0).dArrEmbedding,hEmbedding.at(1).dArrEmbedding,dVScanResult,li,lij,lj);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);
	if(cudaStatus !=hipSuccess){
		fprintf(stderr,"\n kernelSetValueForFirstTwoEmbeddingColumn in failed",cudaStatus);
		status = -1;
		goto Error;
	}
	
	hEmbedding.at(0).prevCol=-1;
	hEmbedding.at(1).prevCol=0;

	for (int i = 0; i < hEmbedding.size(); i++)
	{
		printf("\n\n Q[%d] prevCol:%d ",i,hEmbedding.at(i).prevCol);		
		kernelPrintEmbedding<<<1,512>>>(hEmbedding.at(i).dArrEmbedding,hEmbedding.at(i).noElem);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		CHECK(cudaStatus);
		if(cudaStatus!=hipSuccess){
			status =-1;
			printf("kernelPrintEmbedding failed");
			goto Error;
		}
	}


Error:
	return status;
}



//Why do this snippet face the error: Invalid device pointer

//void PMS::displayEmbeddingColumn(EmbeddingColumn ec){
//	printf("\n noElem:%d prevCol:%d",ec.noElem,ec.prevCol);
//	
//	Embedding *hArrEmbeddingt = (Embedding*)malloc(sizeof(Embedding)*ec.noElem);
//	if(hArrEmbeddingt==NULL){
//		printf("\n malloc hArrEmbeddingt in displayEmbeddingColumn() failed");
//		exit(1);
//	}
//
//	CHECK(hipMemcpy(hArrEmbeddingt,ec.dArrEmbedding,sizeof(Embedding)*ec.noElem,hipMemcpyDeviceToHost));
//	for (int i = 0; i < ec.noElem; i++)
//	{
//		printf("\n A[%d]: (idx, vid):(%d, %d)",i,hArrEmbeddingt[i].idx,hArrEmbeddingt[i].vid);
//	}
//
//	hipFree(hArrEmbeddingt);
//}

