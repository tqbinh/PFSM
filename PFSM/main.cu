#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <iostream>
#include <string>
#include <map>
#include "conio.h"
#include <fstream>
#include "pms.cuh"
//#include "kernelPrintf.h"
//#include "kernelCountLabelInGraphDB.h"
//#include "kernelMarkInvalidVertex.h"
//#include "markInvalidVertex.h"
//#include "checkArray.h"
//#include "displayArray.h"
//#include "checkDataBetweenHostAndGPU.h"
//#include "access_d_LO_from_idx_of_d_O.h"
//#include "countNumberOfLabelVetex.h"
//#include "countNumberOfEdgeLabel.h"
//#include "extractUniqueEdge.h"
//#include "ExtensionStructure.h"
//#include "getAndStoreExtension.h"
//#include "validEdge.h"
//#include "scanV.h"
//#include "getLastElement.h"
//#include "getValidExtension.h"
//#include "getUniqueExtension.h"
//#include "calcLabelAndStoreUniqueExtension.h"
//#include "calcBoundary.h"
//#include "calcSupport.h"
//#include "getSatisfyEdge.h"
//#include "header.h"
//
//
//#include "helper_timer.h"
using namespace std;


//
//#define CHECK(call) \
//{ \
//const hipError_t error = call; \
//if (error != hipSuccess) \
//{ \
//printf("Error: %s:%d, ", __FILE__, __LINE__); \
//printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
//exit(1); \
//} \
//}



int main(int argc, char * const  argv[]){	
	StopWatchWin timer;

#pragma region "load database"

	std::ofstream fout("result.txt", std::ios_base::app | std::ios_base::out);
	
	timer.start();
	PMS pms;
	pms.os=&fout;
	pms.prepareDataBase(); //chuẩn bị dữ liệu
	timer.stop();
	pms.printdb(); //hiển thị dữ liệu
	
	printf("\n\n**===-------------------------------------------------===**\n");
    printf("Loading data...\n");
	printf("Processing time: %f (ms)\n", timer.getTime());
	hTime=timer.getTime();
	timer.reset();

#pragma endregion "end load database"

	FUNCHECK(pms.extractAllEdgeInDB()); //Từ CSDL đã nạp vào device, trích tất cả các cạnh trong CSDL song song
	pms.displayArrExtension(pms.hExtension.at(0).dExtension,pms.hExtension.at(0).noElem); //Những cạnh này được xem như là một mở rộng của pattern P

	FUNCHECK(pms.getValidExtension()); //Trích các mở rộng hợp lệ (li<lj: chỉ xét cho đơn đồ thị vô hướng)
	
	FUNCHECK(pms.extractUniEdge());

	FUNCHECK(pms.computeSupport()); //Tính độ hộ trợ của cả cạnh trong UniEdge và loại bỏ những mở rộng không thoả minsup

	FUNCHECK(pms.Mining()); //kiểm tra DFS_CODE có phải là min hay không, nếu là min thì ghi kết quả vào file result.txt, và xây dựng Embedding Columns


	system("pause");

	return 0;
}

