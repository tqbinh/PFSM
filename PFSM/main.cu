#include "hip/hip_runtime.h"
#pragma once
// Include all MGPU kernels.
#include "moderngpu.cuh"
#include <typeinfo>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <iostream>
#include <string>
#include <map>
#include "conio.h"
#include <fstream>
#include "pms.cuh"

using namespace std;
using namespace mgpu;

ContextPtr ctx;

void device_info()
{
	int devCount = 0;
	hipGetDeviceCount(&devCount);
	cout<<endl<<"So luong device:"<<devCount<<endl;
	hipDeviceProp_t devProp;
	for (int i = 0; i < devCount; i++)
	{
		hipGetDeviceProperties(&devProp,0);
		cout<<endl<<"name: "<<devProp.name<<endl;
		cout<<endl<<"major: "<<devProp.major<<endl;
		cout<<endl<<"minor: "<<devProp.minor<<endl;
		cout<<endl<<"totalGlobalMem: "<<devProp.totalGlobalMem<<endl;
		cout<<endl<<"totalConstMem: "<<devProp.totalConstMem<<endl;
		cout<<endl<<"maxGridSize x,y,z,all: "<<devProp.maxGridSize[0]<<","<< \
			devProp.maxGridSize[1]<<","<<devProp.maxGridSize[2]<<","<<devProp.maxGridSize[3]<<endl;
		cout<<endl<<"maxThreadsDim x,y,z,all: "<<devProp.maxThreadsDim[0]<<","<< \
			devProp.maxThreadsDim[1]<<","<<devProp.maxThreadsDim[2]<<","<<devProp.maxThreadsDim[3]<<endl;
		cout<<endl<<"maxThreadsPerBlock(so luong tieu trinh toi da 1 block): "<<devProp.maxThreadsPerBlock<<endl;
		cout<<endl<<"devProp.maxThreadsPerMultiProcessor(so luong tieu trinh toi da 1 SM):"<<devProp.maxThreadsPerMultiProcessor<<endl;
		cout<<endl<<"sharedMemPerBlock (Dung luong shareMem cua 1 Block) (KB): "<<devProp.sharedMemPerBlock<<endl;
		cout<<endl<<"multiProcessorCount(so luong SM): "<<devProp.multiProcessorCount<<endl;
		cout<<endl<<"regsPerBlock: "<<devProp.regsPerBlock<<endl;
		cout<<endl<<"warpSize: "<<devProp.warpSize<<endl;
		cout<<endl<<"concurrentKernels: "<<devProp.concurrentKernels<<endl;
	}
	system("pause");
	exit(0);
}
int main(int argc, char** argv){
	//int status=0;
	hipDeviceReset();
	ctx = CreateCudaDevice(argc, argv, true);
	cout << typeid(ctx).name() << endl;

	//device_info();
	//cdactx=*ctx;
	StopWatchWin timer;
	//exit(0);
	//system("pause");
#pragma region "load database"
	//Open file result.txt to write append
	std::ofstream fout("result.txt", std::ios_base::app | std::ios_base::out);
	
	timer.start();
	PMS pms; //Tạo đối tượng PMS.
	pms.os=&fout;
	pms.prepareDataBase(); //chuẩn bị dữ liệu
	timer.stop();
	//pms.printdb(); //hiển thị dữ liệu
	
	std::printf("\n\n**===-------------------------------------------------===**\n");
	std::printf("Loading data...\n");
	std::printf("Processing time: %f (ms)\n", timer.getTime());//Processing time: 6595
	hTime=timer.getTime();
	timer.reset();

#pragma endregion "end load database"

	pms.extractAllEdgeInDB(); //Từ CSDL đã nạp vào device, trích tất cả các cạnh trong CSDL song song
	timer.start();
	//Trích các mở rộng hợp lệ (li<lj: chỉ xét cho đơn đồ thị vô hướng) \
	//==> Notes: Cần phải xét cho trường hợp đa đồ thị vô hướng và có hướng
	pms.getValidExtension_pure(); 
	timer.stop();
	std::printf("\n\n**===-------------------------------------------------===**\n");
	std::printf("getValidExtension_pure\n");
	std::printf("Processing time: %f (ms)\n", timer.getTime());//Processing time: 8.730469 (ms)
	hTime=timer.getTime();
	timer.reset();
	timer.start();
	pms.extractUniEdge();
	timer.stop();
	std::printf("\n\n**===-------------------------------------------------===**\n");
	std::printf("extractUniEdge\n");
	std::printf("Processing time: %f (ms)\n", timer.getTime());//Processing time: 1.730469 (ms)
	hTime=timer.getTime();
	timer.reset();
	timer.start();
	pms.computeSupport(); //Tính độ hộ trợ của cả cạnh trong UniEdge và loại bỏ những mở rộng không thoả minsup
	//Đến đây, chúng ta đã thu thập được các mở rộng một cạnh thoả minsup (hUniEdgeSatisfyMinSup)
	//
	//FUNCHECK(pms.Mining()); //kiểm tra DFS_CODE có phải là min hay không, nếu là min thì ghi kết quả vào file result.txt, và xây dựng Embedding Columns
	timer.stop();
	std::printf("\n\n**===-------------------------------------------------===**\n");
	std::printf("computeSupport\n");
	std::printf("Processing time: %f (ms)\n", timer.getTime()/1000);//Processing time: 15.730469 (s)
	hTime=timer.getTime();
	timer.reset();
	
	timer.start();
	//Duyệt qua các cạnh thoả minsup để xây dựng:
	//DFSCODE, hEmbedding, hLevelPtrEmbedding, hLevelListVerRMP và hLevelRMP để chuẩn bị khai thác.
	//FUNCHECK(pms.initialize());
	//Trích các mở rộng thoả minDFS_CODE ban đầu
	pms.MiningDeeper(pms.hLevelEXT.at(0).vE.at(0), pms.hLevelUniEdgeSatisfyMinsup.at(0).vecUES.at(0));
	timer.stop();
	std::printf("\n\n**===-------------------------------------------------===**\n");
	std::printf("MiningDeeper()\n");
	std::printf("Processing time: %f (ms)\n", timer.getTime());//Processing time:  (ms)
	hTime=timer.getTime();
	system("pause");

	return 0;
}