#include "hip/hip_runtime.h"
#pragma once

#include "moderngpu.cuh"		// Include all MGPU kernels.

using namespace mgpu;


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <iostream>
#include <string>
#include <map>
#include "conio.h"
#include <fstream>
#include "pms.cuh"
//#include "kernelPrintf.h"
//#include "kernelCountLabelInGraphDB.h"
//#include "kernelMarkInvalidVertex.h"
//#include "markInvalidVertex.h"
//#include "checkArray.h"
//#include "displayArray.h"
//#include "checkDataBetweenHostAndGPU.h"
//#include "access_d_LO_from_idx_of_d_O.h"
//#include "countNumberOfLabelVetex.h"
//#include "countNumberOfEdgeLabel.h"
//#include "extractUniqueEdge.h"
//#include "ExtensionStructure.h"
//#include "getAndStoreExtension.h"
//#include "validEdge.h"
//#include "scanV.h"
//#include "getLastElement.h"
//#include "getValidExtension.h"
//#include "getUniqueExtension.h"
//#include "calcLabelAndStoreUniqueExtension.h"
//#include "calcBoundary.h"
//#include "calcSupport.h"
//#include "getSatisfyEdge.h"
//#include "header.h"
//
//
//#include "helper_timer.h"
using namespace std;


//
//#define CHECK(call) \
//{ \
//const hipError_t error = call; \
//if (error != hipSuccess) \
//{ \
//printf("Error: %s:%d, ", __FILE__, __LINE__); \
//printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
//exit(1); \
//} \
//}



int main(int argc, char** argv){	
	int status=0;
	ContextPtr ctx = CreateCudaDevice(argc, argv, true);
	StopWatchWin timer;
	
	system("pause");
#pragma region "load database"

	std::ofstream fout("result.txt", std::ios_base::app | std::ios_base::out);
	
	timer.start();
	PMS pms;
	pms.os=&fout;
	FUNCHECK(status=pms.prepareDataBase()); //chuẩn bị dữ liệu
	if(status!=0){
		cout<<endl<<"prepareDataBase function failed"<<endl;
		exit(1);
	}

	timer.stop();
	pms.printdb(); //hiển thị dữ liệu
	
	std::printf("\n\n**===-------------------------------------------------===**\n");
    std::printf("Loading data...\n");
	std::printf("Processing time: %f (ms)\n", timer.getTime());
	hTime=timer.getTime();
	timer.reset();

#pragma endregion "end load database"

	FUNCHECK(pms.extractAllEdgeInDB()); //Từ CSDL đã nạp vào device, trích tất cả các cạnh trong CSDL song song
	pms.displayArrExtension(pms.hExtension.at(0).dExtension,pms.hExtension.at(0).noElem); //Những cạnh này được xem như là một mở rộng của pattern P

	FUNCHECK(pms.getValidExtension_pure()); //Trích các mở rộng hợp lệ (li<lj: chỉ xét cho đơn đồ thị vô hướng)
	
	FUNCHECK(pms.extractUniEdge());

	FUNCHECK(pms.computeSupport()); //Tính độ hộ trợ của cả cạnh trong UniEdge và loại bỏ những mở rộng không thoả minsup
	//Đến đây, chúng ta đã thu thập được các mở rộng một cạnh thoả minsup (hUniEdgeSatisfyMinSup)
	//
	//FUNCHECK(pms.Mining()); //kiểm tra DFS_CODE có phải là min hay không, nếu là min thì ghi kết quả vào file result.txt, và xây dựng Embedding Columns
	FUNCHECK(pms.initialize()); //Duyệt qua các cạnh thoả minsup để xây dựng DFSCODE, hEmbedding, hLevelPtrEmbedding, hLevelListVerRMP và hLevelRMP để chuẩn bị khai thác.

	system("pause");

	return 0;
}




//int main(int argc, char** argv) 
//{
//    ContextPtr context = CreateCudaDevice(argc, argv, true);
//
//   int noElem = 5;
//   int* ptr = (int*)malloc(sizeof(int)*noElem);
//   for (int i = 0; i < noElem; i++)
//   {
//	   ptr[i]=i;
//	   cout<<ptr[i]<<" ";
//   }
//   cout<<endl;
//   int *p=nullptr;
//   hipMalloc((void**)&p,sizeof(int)*noElem);
//   hipMemcpy(p,ptr,noElem*sizeof(int),hipMemcpyHostToDevice);
//   cout<<"Input data"<<endl;
//   kernelPrintdArr<<<1,100>>>(p,noElem);
//   hipDeviceSynchronize();
//   cout<<endl;
//  //// int result = Reduce(p, noElem, *context);
//  //// printf("Reduction total: %d\n\n", result);
//   int result=0;
//   //ScanExc(p, noElem, &result, *context);
//   ScanExc(p, noElem, *context);
////   PrintArray(*data, "%4d", 10);
//    kernelPrintdArr<<<1,100>>>(p,noElem);
//    hipDeviceSynchronize();
//    //printf("Exclusive scan:\n");
//    //printf("Scan total: %d\n", result);
//
//	hipFree(p);
//
//    //// Run an exclusive scan.
//    //ScanExc(data->get(), N, &total, context);
//    //printf("Exclusive scan:\n");
//    //PrintArray(*data, "%4d", 10);
//    //printf("Scan total: %d\n", total);
//
//	_getch();
//    return 0;
//}