#include "hip/hip_runtime.h"
#include "reduction.h"


#define funcCheck(stmt) {                                            \
    hipError_t err = stmt;                                          \
    if (err != hipSuccess)                                          \
    {                                                                \
        printf( "Failed to run stmt %d ", __LINE__);                 \
        printf( "Got CUDA error ...  %s ", hipGetErrorString(err)); \
        return cudaStatus;                                                   \
    }                                                                \
}

__global__  void total(float * input, float * output, int len) 
{
	// Load a segment of the input vector into shared memory
	__shared__ float partialSum[2*BLOCK_SIZE];
	int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;

	if ((start + t) < len)
	{
		partialSum[t] = input[start + t];      
	}
	else
	{       
		partialSum[t] = 0.0;
	}
	if ((start + blockDim.x + t) < len)
	{   
		partialSum[blockDim.x + t] = input[start + blockDim.x + t];
	}
	else
	{
		partialSum[blockDim.x + t] = 0.0;
	}

	// Traverse reduction tree
	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t + stride];
	}
	__syncthreads();

	// Write the computed sum of the block to the output vector at correct index
	if (t == 0 && (globalThreadId*2) < len)
	{
		output[blockIdx.x] = partialSum[t];
	}
}


hipError_t reduction(float *deviceInput,int len,float &support){
	hipError_t cudaStatus;	
	
    float * deviceOutput;

	int numInputElements = len; // number of elements in the input list
	int numOutputElements; // number of elements in the output list

	numOutputElements = numInputElements / (BLOCK_SIZE<<1);
	if (numInputElements % (BLOCK_SIZE<<1)) 
	{
		numOutputElements++;
	}
		
    funcCheck(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));

	// Initialize the grid and block dimensions here
    dim3 DimGrid( numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // Launch the GPU Kernel here
    total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize() reduction failed");
		goto Error;
	}
	//printf("\n");
	//printFloat(deviceOutput,numOutputElements);
	hipMemcpy(&support,deviceOutput,numOutputElements*sizeof(float),hipMemcpyDeviceToHost);

//	hipFree(deviceInput);
//	hipFree(deviceOutput);
Error:
	return cudaStatus;
}


inline hipError_t reduction(int *input,int len,int &support){
		hipError_t cudaStatus;	
	
    float * deviceOutput;
	float *deviceInput;


	int numInputElements = len; // number of elements in the input list
	int numOutputElements; // number of elements in the output list

	numOutputElements = numInputElements / (BLOCK_SIZE<<1);
	if (numInputElements % (BLOCK_SIZE<<1)) 
	{
		numOutputElements++;
	}
		
    funcCheck(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));		
    funcCheck(hipMalloc((void **)&deviceInput, numInputElements * sizeof(float)));

	dim3 block(512);
	dim3 grid((numInputElements+block.x-1)/block.x);
	kernelCastingInt2Float<<<grid,block>>>(deviceInput,input,numInputElements);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelCastingInt2Float() reduction() failed");
		goto Error;
	}

	// Initialize the grid and block dimensions here
    dim3 DimGrid( numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // Launch the GPU Kernel here
    total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n total kernel in reduction() failed");
		goto Error;
	}

	int *deviceOutputInt=nullptr;
	cudaStatus = hipMalloc((void**)&deviceOutputInt,numOutputElements*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc((void**)&deviceOutputInt in reduction() failed");
		goto Error;
	}

	kernelCastingFloat2Int<<<grid,block>>>(deviceOutputInt,deviceOutput,numOutputElements);
	hipDeviceSynchronize();

	funcCheck(hipMemcpy(&support,deviceOutputInt,numOutputElements*sizeof(int),hipMemcpyDeviceToHost));

	hipFree(deviceInput);
	hipFree(deviceOutput);
	hipFree(deviceOutputInt);
Error:
	return cudaStatus;
}
