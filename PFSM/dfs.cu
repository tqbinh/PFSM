#include "hip/hip_runtime.h"
/*
 *  dfs.cpp
 *  GSPAN
 *
 *  Created by Jinseung KIM on 09. 07. 19.
 *  Copyright 2009 KyungHee. All rights reserved.
 *
 */
#pragma once
#include "gspan.cuh"
#include "pms.cuh"
#include <cstring>
#include <string>
#include <iterator>
#include <set>
using namespace std;

#define CUCHECK(call) \
{ \
const hipError_t error = call; \
if (error != hipSuccess) \
{ \
std::printf("Error: %s:%d, ", __FILE__, __LINE__); \
std::printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
system("pause"); \
exit(0); \
} \
}
void DFSCode::add(int vi,int vj,int li,int lij,int lj)
{
	if (nodeCount()==0)
	{
		push(vi,vj,li,lij,lj); //Push 1st edge to empty DFS_CODE
		minLabel = vi;
		maxId = vj;
		return;
	}
	if(vi<vj)
	{
		push(vi,vj,-1,lij,lj);//build DFS_CODE forward
		maxId=vj;
	}
	else
	{
		push(vi,vj,-1,lij,-1);//xây dựng DFS_CODE backward
	}
}
//use
void DFSCode::remove(int vi,int vj)
{
	pop();
	if (vi<vj)
	{
		--maxId;
	}
}

void DFSCode::fromGraph(Graph& g){
	clear();
	EdgeList edges;
	for(unsigned int from=0;from<g.size();++from)
	{
		if(get_forward_root(g,g[from],edges)==false)
			continue;
		for(EdgeList::iterator it = edges.begin();it!=edges.end();++it)
			push(from,(*it)->to,g[(*it)->from].label,(*it)->elabel,g[(*it)->to].label);
	}
}

bool DFSCode::toGraph(Graph& g) //Convert DFSCode sang đồ thị.
{
	g.clear(); //g là một graph hay là một vector<vertex>, mỗi một phần tử của vector là một vertex và kèm theo các cạnh gắn liền với đỉnh đó.
	
	for(DFSCode::iterator it = begin();it != end(); ++it){ //Duyệt qua DFSCODE
		g.resize(std::max (it->from,it->to) +1); //khởi tạo kích thước cho đồ thị g chính bằng số lượng đỉnh của DFSCode
		
		if(it->fromlabel != -1) //nếu như nhãn của đỉnh là hợp lệ
			g[it->from].label = it->fromlabel; //
		if(it->tolabel != -1)
			g[it->to].label = it->tolabel;
		g[it->from].push (it->from,it->to,it->elabel);
		if(g.directed == false)
			g[it->to].push (it->to,it->from,it->elabel);
	}
	
	g.buildEdge();
	
	return (true);
}

void importDataToArray(int*& _arrayO,int*& _arrayLO,int*& _arrayN,int*& _arrayLN, \
					   const unsigned int _sizeOfarrayO,const unsigned int _noDeg,Graph& g) //return -1 if error
{
	int i=0;
	int numberOfEdges=0;
	int j=0;
	_arrayO[i]=0;
	for(Graph::vertex_iterator v = g.begin(); v !=g.end(); ++v)
	{	//Duyệt qua các cạnh của đỉnh
		for(Vertex::edge_iterator it = v->edge.begin();it!=v->edge.end();++it)
		{	//Gán nhãn cho đỉnh From trong mảng LO, bị gán nhiều lần trong mỗi lần lặp cạnh không tốt
			_arrayLO[i]=g[it->from].label; 
			_arrayN[j]=it->to; //gán id cho đỉnh to trong mảng N
			_arrayLN[j]=it->elabel; //gán nhãn cho cạnh
			j=j+1;	//tăng chỉ số trong mảng N và mảng LN
			++numberOfEdges; //số cạnh đã duyệt
		}
		if (i>=(_sizeOfarrayO-1)) return;
		_arrayO[i+1]=numberOfEdges;
		++i;
	}
}


//use
//Build DFS_Code on Device for checking minDFSCODE
//This action convert DFS_CODE to graph and store graph db on device.
void DFSCode::buildDFSCodeOnDevice()
{
	Graph tempGraph;
	int noOfVer=0;
	int noDeg=0;
	toGraph(tempGraph);
	//Get total of vertex in graph
	noOfVer = tempGraph.vertex_size();
	int* hArrO = new int[noOfVer];
	if (hArrO==NULL){exit(-1);}
	else {memset(hArrO, -1, noOfVer*sizeof(int));}
	//Get total of degree of all vertex in graph.
	Graph& g = tempGraph; 
	for(Graph::vertex_iterator v = g.begin(); v !=g.end(); ++v)
	{noDeg +=v->edge.size();}
	unsigned int sizeOfArrayN=noDeg;
	//Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	int* hArrN = new int[sizeOfArrayN];

	if(hArrN==NULL){exit(-1);}
	else {memset(hArrN, -1, noDeg*sizeof(int));}

	//Prepare dataset on host
	//Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	int* arrayLO = new int[noOfVer];
	if(arrayLO==NULL)
	{
		exit(-1);
	}else
	{
		memset(arrayLO, -1, noOfVer*sizeof(int));
	}


	//Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	int* arrayLN = new int[noDeg];
	if(arrayLN==NULL){
		exit(0);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}

	importDataToArray(hArrO,arrayLO,hArrN,arrayLN,noOfVer,noDeg,g);
	for(int i = 0; i<noOfVer;i++)
	{
		cout<<hArrO[i]<<":"<<arrayLO[i]<<" ";
	}
	cout<<endl;
	for(int i = 0; i<noDeg;i++)
	{
		cout<<hArrN[i]<<":"<<arrayLN[i]<<" ";
	}
	cout<<endl;

	//Copy data from host to device
	graphdfscode.noElemdO = noOfVer;
	graphdfscode.noElemdN = noDeg;
	size_t  nBytesO = noOfVer * sizeof(int);
	size_t nBytesN = noDeg * sizeof(int);
	CUCHECK(hipMalloc((void**)&graphdfscode.dO,nBytesO));
	CUCHECK(hipMalloc((void**)&graphdfscode.dLO,nBytesO));
	CUCHECK(hipMalloc((void**)&graphdfscode.dN,nBytesN));
	CUCHECK(hipMalloc((void**)&graphdfscode.dLN,nBytesN));

	//Chép dữ liệu từ mảng arrayO trên CPU sang GPU được quản lý bởi pointer dO
	CUCHECK(hipMemcpy(graphdfscode.dO,hArrO,nBytesO,hipMemcpyHostToDevice));
	CUCHECK(hipMemcpy(graphdfscode.dLO,arrayLO,nBytesO,hipMemcpyHostToDevice));
	CUCHECK(hipMemcpy(graphdfscode.dN,hArrN,nBytesN,hipMemcpyHostToDevice));
	CUCHECK(hipMemcpy(graphdfscode.dLN,arrayLN,nBytesN,hipMemcpyHostToDevice));

	//Release host memory
	delete[] hArrN;
	delete[] hArrO;
	delete[] arrayLO;
	delete[] arrayLN;
	return;
}

void extractAllEdgeInDB(DB &graphdfscode,arrExtension &arrE)
{
	
	//cấp phát bộ nhớ cho d_Extension
	//Lấy số lượng cạnh của tất cả các đồ thị
	arrE.noElem =graphdfscode.noElemdN;
	//Cấp phát bộ nhớ để lưu trữ tất cả các mở rộng ban đầu tương ứng với số lượng cạnh thu được;
	size_t nBytesOfArrayExtension = arrE.noElem*sizeof(Extension); 

	CUCHECK(hipMalloc((Extension**)&arrE.dExtension,nBytesOfArrayExtension));

	//Trích tất cả các cạnh từ database rồi lưu vào d_Extension
	
	dim3 block(blocksize);
	unsigned int numberOfElementd_O = graphdfscode.noElemdO;
	dim3 grid((numberOfElementd_O+block.x-1)/block.x);
	
	
	kernelGetAndStoreExtension<<<grid,block>>>( \
		graphdfscode.dO,graphdfscode.dLO, \
		graphdfscode.noElemdO, \
		graphdfscode.dN,graphdfscode.dLN,graphdfscode.noElemdN, \
		arrE.dExtension);
	CUCHECK(hipDeviceSynchronize());
	CUCHECK(hipGetLastError());
	return;
}
bool DFSCode::check_min()
{
	if (this->size() == 1) return true;
	//1. Xây dựng database của đồ thị trên GPU: dfscode_LO, dfscode_O, dfscode_N, dfscode_LN
	//Convert DFS_CODE sang Graph
	arrExtension pureExtension;
	this->buildDFSCodeOnDevice();
	//2. Tìm tất cả mở rộng 1 cạnh ban đầu hợp lệ (GPU step)
	extractAllEdgeInDB(this->graphdfscode,pureExtension);
	//3. So sánh chúng với cạnh đầu tiên của DFS_CODE (GPU step)
		//Nếu có cạnh nhỏ hơn DFS_CODE thì return False
		//Xây dựng embeddings Colum cho cạnh bằng với cạnh đầu tiên của DFS_CODE
	//4. Duyệt qua các cạnh còn lại của DFS_CODE theo thứ tự (tạm gọi là cạnh i). Từ cạnh i ta biết được mở rộng tiếp theo
			//là từ đỉnh nào của RMP.
				//Nếu cạnh i là cho biết mở rộng backward từ đỉnh cuối của RMP. Thì phải tìm đúng mở rộng backward)
				//Nếu cạnh i là mở rộng forward từ đỉnh cuối của RMP thì phải xét cả backward và forward
				//Nếu cạnh i không là mở rộng từ đỉnh không thuộc đỉnh cuối của RMP thì phải xét backward, forward của đỉnh cuối
					//và xét các forward của các đỉnh kế cuối đến i. 
	//5. Tìm RMP của DFS_CODE hiện tại
	//6. Duyệt qua RMP từ đỉnh phải cùng.
		//6.1 Nếu cạnh i là backward và Nếu là các đỉnh phải cùng của RMP thì tìm các mở rộng backward trước.
				//6.1.1 Nếu có mở rộng nào nhỏ hơn cạnh i thì return false.
				//6.1.2 Ngược lại, xây dựng embeddings columns cho các mở rộng bằng với cạnh i,
						//rồi quay lên bước 6. (Nếu không có cạnh nào bằng với cạnh i thì sao?
												//Điều này có thể xảy ra hay không?)
		//6.2 Tìm các mở rộng forward
			//6.2.1 Nếu có mở rộng nào nhỏ hơn cạnh i thì return false (làm sao dọn dẹp bộ nhớ trên device trước khi return false)?
			//6.2.2 Ngược lại, xây dựng embedding columns cho các mở rộng bằng với cạnh i, rồi quay lên bước 6.
	//7. return true (thoả min)
}

unsigned int DFSCode::nodeCount(void) //giải thuật đếm node trên cây
{
	unsigned int nodecount = 0;
	for(DFSCode::iterator it = begin();it != end(); ++it)
		nodecount = std::max(nodecount,(unsigned int) (std::max(it->from,it->to) + 1)); 
	return (nodecount);
}

std::ostream& DFSCode::write(std::ostream& os)
{
	if(size()==0) return os;
	
	os<<"("<<(*this)[0].fromlabel<<") "<<(*this)[0].elabel<<" (of"<<(*this)[0].tolabel<<")";
	
	for(unsigned int i=1;i<size();++i){
		if((*this)[i].from < (*this)[i].to){
			os<<" "<<(*this)[i].elabel<<" ("<<(*this)[i].from<<"f"<<(*this)[i].tolabel<<")";
		}else{
			os<<" "<<(*this)[i].elabel<<" (b"<<(*this)[i].to<<")";
		}
	}
	return os;
}
